//#include "common.h"
#include "user_function.h"

void release_MultiGraph(struct HYB *m) 
{
	hipFree(m->_ces); hipFree(m->_cey); 
        hipFree(m->_dx); hipFree(m->_dy); hipFree(m->_dccindex);
        hipFree(m->_sp1); hipFree(m->_sp2); hipFree(m->_mapper);
        hipFree(m->_ncsc_v); hipFree(m->_ncsc_e);
        hipFree(m->_itable);
        hipFree(m->_i1); 
	if((m->dcnt) > 0) {
		hipFree(m->_dp1);	
	}

	//m->vv release
	
	hipFree(m->_i1); //dep
}
