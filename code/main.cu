#ifndef _KERNEL_SUPPORT_
#define _KERNEL_SUPPORT_
#endif

#include "user_function.h"
#include <stdlib.h>
//#include "common.h"


struct csc_package inp;
struct HYB m;
struct MULTI_SPARSE m_sparse;
//struct MG status; // MG = multigraph
struct OUTPUT out;
struct vector_data vd;


int main(int argc, char **argv)
{

int *tt; short *tt2;
int u_loop = atoi(argv[3]);
double total_time=0.0f;
double total_ms=0.0f;

#if defined BFS || defined SSSP 

	generate_CSC(&inp, argv);

	generate_MultiGraph(&m, &inp);

	initValue(inp.start_p, &m, &vd);


    for(int loop=0;loop<u_loop;loop++) {
	int iter=1;
	resetValue(inp.start_p, &m, &vd);
	hipDeviceSynchronize();

	double s1 = rtclock();
	while(MultiGraph_V_E_V(&m, &inp, &vd, &out, &iter)); 
	double s2 = rtclock();
	total_time += s2-s1;
    }
	total_ms = total_time*1000/(float)u_loop;	
	fprintf(stdout, "processing : %f ms,%f MTEPS,", total_ms, (float)inp.ne / (total_ms * 1000));

	verifyResults(&inp, &m, &out, inp.start_p);

	release_MultiGraph(&m);
#endif
#if defined BC

	generate_CSC(&inp, argv);

	generate_MultiGraph(&m, &inp);

	initValue(inp.start_p, &m, &vd);

//INT_PRINT(vd._vv2, m.nv);
//SHORT_PRINT(m._sp1, 10);
//SHORT_PRINT(m._sp2, 10);
//exit(0);

    for(int loop=0;loop<u_loop;loop++) {
 	int iter=1;
	resetValue(inp.start_p, &m, &vd);
	hipDeviceSynchronize();
	double s1 = rtclock();
	while(MultiGraph_V_E_V(&m, &inp, &vd, &out, &iter)); 
	MultiGraph_V_E_V_pull(&m, &inp, &vd, &out, &iter); 
	double s2 = rtclock();
	total_time += s2-s1;
    }
	total_ms = total_time*1000/(float)u_loop;	
	fprintf(stdout, "processing : %f ms,%f MTEPS,", total_ms, (float)inp.ne * 2 / (total_ms * 1000));

//INT_PRINT(vd._vv, 100);

	verifyResults(&inp, &m, &out, inp.start_p);

//	verifyResults(&inp, &m, &out, 0);

//	release_MultiGraph(&m);

#endif
#if defined CC
	int iter=1, dummy;

	struct vector_data vvd;

	int *lst, *_lst;

	generate_CSC(&inp, argv);

	lst=(int *)malloc(sizeof(int)*(inp.nv));
	for(int i=0;i<inp.nv;i++) {
		lst[i] = i;
	}	
	hipMalloc((void **) &_lst, sizeof(int)*(inp.nv));
	hipMemcpyAsync(_lst, lst, sizeof(int)*(inp.nv), hipMemcpyHostToDevice);

	generate_MultiGraphS(&m_sparse, &inp);

	initValue(0, &m_sparse, &vd);
	initValue2(0, &m_sparse, &vvd);

   for(int loop=0;loop<u_loop;loop++) {
	resetValue(inp.start_p, &m_sparse, &vd);
	hipDeviceSynchronize();
	double s1 = rtclock();
	iter=1;
	while(1) {
		if(!MultiGraph_E_E(&m_sparse, &vd, &out, &iter)) break;
		vvd.h_fs[0] = (m_sparse.nv); vvd.h_fs[1] = 0; vvd.h_fs[2] = 0; vvd.h_fs[3] = 0;
		vvd.htot_size = vvd.h_fs[0]; 
		hipMemcpyAsync(vvd._curr_f, _lst, sizeof(int)*(m_sparse.nv), hipMemcpyDeviceToDevice);
		while(MultiGraph_V_V(&m_sparse, &vd, &vvd, &out, &dummy));
	}
	double s2 = rtclock();
	total_time += s2-s1;
   }
	total_ms = total_time*1000/(float)u_loop;	
	fprintf(stdout, "processing : %f ms,%f MTEPS,", total_ms, (float)inp.ne / (total_ms * 1000));
	verifyResults(&inp, &m_sparse, &out, 0);
#endif


#if defined PR_D || defined PR_T
	int iter;
	generate_CSC(&inp, argv);


//fprintf(stderr, "099990900\n");

	generate_MultiGraph(&m, &inp);

//fprintf(stderr, "099990900\n");

	initValue(0, &m, &vd);	


   for(int loop=0;loop<u_loop;loop++) {
	resetValue(inp.start_p, &m, &vd);
	hipDeviceSynchronize();
	double s1 = rtclock();
	while(MultiGraph_V_E_V_NOTIDEM(&m, &inp, &vd, &out, &iter)) {
	}; 
	double s2 = rtclock();
	total_time += s2-s1;
    }

	total_ms = total_time*1000/(float)u_loop;	
	fprintf(stdout, "processing : %f ms,%f MTEPS,", total_ms, (float)inp.ne / (total_ms * 1000));

	verifyResults(&inp, &m, &out, 0);
#endif

}
