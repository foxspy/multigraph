#include "hip/hip_runtime.h"
#include "user_function.h"
//#include "common.h"
#ifdef BC

__global__ void process_fbc(int nv, int iter, int tf_size, int *f1, int *vvc, int *v_count, float *v_dep, int *csr_v, int *csr_e)
{

        int i,j;
        int base_addr = blockIdx.x*64 + (threadIdx.x>>2); // blocksize = 128, 4thread=1vertex
        int warp_id, index, index_size, bias;

        __shared__ int buffer1[256], buffer2[256], buffer3[256], buffer4[256];
        __shared__ int buffer_p[2];
        __shared__ float s_buffer[8];
        if(threadIdx.x < 2) {
                buffer_p[threadIdx.x] = 0;
        }
        __syncthreads();

        warp_id = ((threadIdx.x>>5)&3);
if(base_addr < tf_size) {
        bias=0;
        index = f1[base_addr];

        index_size = csr_v[index+1] - csr_v[index];

        if(index_size >= 32) {
                bias = index_size - (index_size&31);
                if((threadIdx.x&3) == 0) {
                        int p = atomicAggInc(&buffer_p[0]);
                        buffer1[p] = index;
                        buffer2[p] = bias;
                }
                if(index_size >= 256) {
                        if((threadIdx.x&3) == 0) {
                                int p2 = atomicAggInc(&buffer_p[1]);
                                buffer3[p2] = index;
                                buffer4[p2] = index_size - (index_size&255);
                        }
                }
        }


        float v_center = (float)v_count[index];
        //float dep_s=v_dep[index]; int numpath = v_count[index];
        float v_dep_partial = 0.0;
        for(i=bias+(threadIdx.x&3); i<index_size; i+=4) {
                int index_dst = csr_e[csr_v[index]+i];
                if(vvc[index_dst] == iter+1) {
                        //atomicAdd(&v_dep[index_dst], ((float)v_count[index_dst]/numpath)*(1+dep_s));
                        //v_dep_partial += (((float)v_count[index_dst]/numpath)*(1+dep_s));
                        v_dep_partial += ((v_center/v_count[index_dst])*(1+v_dep[index_dst]));
                }
        }
        v_dep_partial += __shfl_down(v_dep_partial, 2);
        v_dep_partial += __shfl_down(v_dep_partial, 1);
        if((threadIdx.x&3) == 0) {
                v_dep[index] += v_dep_partial;
        }
}

        __syncthreads();

        for(i=(threadIdx.x>>5);i<buffer_p[0];i+=8) {
                index = buffer1[i];
                int bf2 = buffer2[i];
                int bf22 = bf2 - (bf2&255);

//              float dep_s=v_dep[index]; int numpath = v_count[index];
                float v_center = (float)v_count[index];
                float v_dep_partial = 0.0;
                for(j=bf22+(threadIdx.x&31);j<bf2;j+=32) {
                        int index_dst = csr_e[csr_v[index]+j];
                        if(vvc[index_dst] == iter+1) {
//                              atomicAdd(&v_dep[index_dst], ((float)v_count[index_dst]/numpath)*(1+dep_s));
                        //      v_dep_partial += (((float)v_count[index_dst]/numpath)*(1+dep_s));
                                v_dep_partial += ((v_center/v_count[index_dst])*(1+v_dep[index_dst]));
                        }
                }

                v_dep_partial += __shfl_down(v_dep_partial, 16);
                v_dep_partial += __shfl_down(v_dep_partial, 8);
                v_dep_partial += __shfl_down(v_dep_partial, 4);
                v_dep_partial += __shfl_down(v_dep_partial, 2);
                v_dep_partial += __shfl_down(v_dep_partial, 1);
                if((threadIdx.x&31) == 0) {
                        v_dep[index] += v_dep_partial;
                }

        }

        __syncthreads();

        for(i=0;i<buffer_p[1];i++) {
                index = buffer3[i];

                //float dep_s=v_dep[index]; int numpath = v_count[index];
                float v_center = (float)v_count[index];
                float v_dep_partial = 0.0;
                for(j=threadIdx.x;j<buffer4[i];j+=blockDim.x) {
                        int index_dst = csr_e[csr_v[index]+j];
                        if(vvc[index_dst] == iter+1) {
//                              atomicAdd(&v_dep[index_dst], ((float)v_count[index_dst]/numpath)*(1+dep_s));
//                              v_dep_partial += (((float)v_count[index_dst]/numpath)*(1+dep_s));
                                v_dep_partial += ((v_center/v_count[index_dst])*(1+v_dep[index_dst]));
                        }
                }
                v_dep_partial += __shfl_down(v_dep_partial, 16);
                v_dep_partial += __shfl_down(v_dep_partial, 8);
                v_dep_partial += __shfl_down(v_dep_partial, 4);
                v_dep_partial += __shfl_down(v_dep_partial, 2);
                v_dep_partial += __shfl_down(v_dep_partial, 1);
                if((threadIdx.x&31) == 0) {
                        s_buffer[threadIdx.x>>5] = v_dep_partial;
                }
                __syncthreads();
                if(threadIdx.x == 0) {
                        v_dep[index] += s_buffer[0]+s_buffer[1]+s_buffer[2]+s_buffer[3]+s_buffer[4]+s_buffer[5]+s_buffer[6]+s_buffer[7];
                }
                __syncthreads();
        }

}











__global__ void bc_phase1(int iter, int nv, int ne, int np, const short *v1, int *vv, int *v_count, float *v_dep, int *i1, int *i2, float *i3, const int *pb1, const int *__restrict__ mapper)
{
        int base_addr = blockIdx.x*PSIZE;
        int i, index; short temp2;
        __shared__ int sv[PSIZE];
        __shared__ int sv2[PSIZE];
        __shared__ float sv3[PSIZE];
        //initialize
        for(i=threadIdx.x; i<PSIZE; i+=blockDim.x) {
                sv[i] = vv[i+base_addr];
                sv2[i] = v_count[i+base_addr];
                sv3[i] = v_dep[i+base_addr];
        }
        __syncthreads();
//printf("th:%d bl:%d\n", threadIdx.x, blockIdx.x);
       //compute(scatter)
        for(i=pb1[blockIdx.x]+threadIdx.x; i<pb1[blockIdx.x+1]; i+=blockDim.x) {

                index = mapper[i>>3]+(threadIdx.x&7); // double-> /4
                temp2 = v1[i];
//printf("value : %d %d %d\n", temp2, PSIZE, index);
                if(temp2 < PSIZE) {
//printf("ok : %d\n", threadIdx.x);
                        //int pivot=sv[temp2];
//                      i1[index] = sv[temp2]; i2[index] = sv2[temp2]; i3[index] = sv3[temp2];
//                      i1[index] = sv[temp2]; // not needed
                        if(sv[temp2] == iter) i3[index] = (1.0f+sv3[temp2])/sv2[temp2];
                        else i3[index] = 0;
                }
        }
}

__global__ void bc_phase11(int iter, int dcnt, const int *dx, const int *dcum_size, const short *de, const int *dmap, const int *vv, const int *v_count, const float *v_dep, int *i1, int *i2, float *i3)
{
        __shared__ int sv[PSIZE];
        __shared__ int sv2[PSIZE];
        __shared__ float sv3[PSIZE];
//      __shared__ int dst_v[PSIZE];///
//int -> short (de)
        int i;
        int x_base = dx[blockIdx.x];
        short curr_index=SUNUSED;
	int curr_v=IUNUSED, temp_v;
	int base_index=dcum_size[blockIdx.x];
        int dmap_index=dmap[blockIdx.x];
        float curr_v2 = 0.0f;

        for(i=threadIdx.x;i<PSIZE;i+=blockDim.x) {
                sv[i] = vv[x_base*PSIZE+i];
                sv2[i] = v_count[x_base*PSIZE+i];
                sv3[i] = v_dep[x_base*PSIZE+i];
                //i3[dmap_index+i] = 0;
        }


        __syncthreads();

        for(i=base_index+threadIdx.x;i<dcum_size[blockIdx.x+1];i+=blockDim.x) {
                short edge_value = de[i];
                if(edge_value < 0) { // vertex
                        if(curr_index >= 0 && curr_index < PSIZE) {
                ////            if(curr_v < i1[dmap_index+curr_index]) {
                ////                    i1[dmap_index+curr_index] = curr_v;
                ////            }
                                if(curr_v2 > 0) {
                                        atomicAdd(&i3[dmap_index+curr_index], curr_v2);
                                }
                        }
                        curr_index = -(edge_value+1);
                        curr_v=BFS_INF;
                        curr_v2=0.0f;
                } else if(edge_value != SUNUSED) { // edge
                        temp_v = sv[edge_value];
                        if(temp_v < curr_v) curr_v = temp_v;
                        if(temp_v == iter) {
//printf(")%d\n", sv2[edge_value]);
                                curr_v2 += (1.0f + sv3[edge_value])/sv2[edge_value];
                        }
                }
        }
////    if(curr_index != UNUSED && curr_v < i1[dmap_index+curr_index])
////            i1[dmap_index+curr_index] = curr_v;
        if(curr_index >= 0 && curr_index < PSIZE && curr_v2 > 0)
                atomicAdd(&i3[dmap_index+curr_index], curr_v2);

/*      __syncthreads();///

        for(i=threadIdx.x;i<PSIZE;i+=blockDim.x) {///
                i1[dmap[blockIdx.x]+i] = dst_v[i];///
        }///
*/
}

__global__ void bc_phase2(int iter, int nv, int ne, int np, const short *v2, int *vv, int *v_count, float *v_dep, const int *i1, const int *i2, const float *i3, const int *pb2, int *finished)
{
        int base_addr = blockIdx.x*PSIZE;
        int i;
        __shared__ int sv[PSIZE];
        __shared__ int sv2[PSIZE];
        __shared__ float sv3[PSIZE];

        //initialize
        for(i=threadIdx.x; i<PSIZE; i+=blockDim.x) {
                sv[i] = vv[base_addr+i];
                sv2[i] = v_count[base_addr+i];
                sv3[i] = v_dep[base_addr+i];
        }
        __syncthreads();

        for(i=pb2[blockIdx.x]+threadIdx.x; i<pb2[blockIdx.x+1]; i+=blockDim.x) {
                int ii=v2[i];
                if(/*(i1[i] == iter) &&*/ (ii < PSIZE) && (sv[ii] == iter-1)) {
//printf("add\n");
//                      atomicAdd(&sv3[ii], ((float)sv2[ii]/i2[i])*(1+i3[i]));
                        atomicAdd(&sv3[ii], i3[i]*sv2[ii]);
                }
        }
        __syncthreads();

        //update
        for(i=threadIdx.x; i<PSIZE; i+=blockDim.x) {
                v_dep[base_addr+i] = sv3[i];
        }

}
#endif

#ifdef BC
int MultiGraph_V_E_V_pull(struct HYB *m, struct csc_package *inp, struct vector_data *vd, struct OUTPUT *out, int *g_iter) // result : dep
{
	int iter=*g_iter;

	int *accum_frpoint = vd->accum_frpoint, *accum_fr = vd->accum_fr;
	int nv = m->nv, ne = m->ne, np = m->np, dcnt = m->dcnt;
	int *vvc = vd->_vv, *v_countc = vd->_vv2, *ccsr_v = m->_ncsc_v, *ccsr_e = m->_ncsc_e;
	int *mapperc = m->_mapper;
	float *v_depc = vd->_vv3;

	short *v1c = m->_sp1, *v2c = m->_sp2;
	int *i1c = m->_i1, *i2c = m->_i2;
	float *i3c = m->_i3;

	int *d_dx = m->_dx, *d_dcum_size = m->_dccindex; short *d_de = m->_dp1;
	int *d_dmap = m->_dmap;

	int *pb1c = m->_ces;
	int *pb2c = m->_cey;
	int *finc;

	hipStream_t stream1,stream2;
        hipStreamCreate(&stream1);
        hipStreamCreate(&stream2);


/*
for(int i=0;i<iter;i++) {
	printf("%d\n", accum_frpoint[i]);
}*/

//return 0;
	accum_frpoint[0]=0;
	for(int i=1;i<=iter;i++) {
		if(accum_frpoint[i] == -1) accum_frpoint[i] = accum_frpoint[i-1];
	}

/*
printf("---\n");
for(int i=0;i<iter;i++) {
	printf("%d\n", accum_frpoint[i]);
}*/

	

        for(int i=iter-1; i>=0; i--) {
                if(accum_frpoint[i+1] > accum_frpoint[i]) {
                        process_fbc<<<((accum_frpoint[i+1]-accum_frpoint[i]+255)>>6),256>>>(nv, i, accum_frpoint[i+1]-accum_frpoint[i], &accum_fr[accum_frpoint[i]], vvc, v_countc, v_depc, ccsr_v, ccsr_e);
                } else {
                        bc_phase1<<<np,BSIZE,0,stream1>>>(i+1, nv, ne, np, v1c, vvc, v_countc, v_depc, i1c, i2c, i3c, pb1c, mapperc); // same as phase1, for efficiency
                        if(dcnt > 0) bc_phase11<<<dcnt,BSIZE,0,stream2>>>(i+1,dcnt,d_dx, d_dcum_size, d_de, d_dmap, vvc, v_countc, v_depc, i1c, i2c, i3c);
                        bc_phase2<<<np,BSIZE>>>(i+1, nv, ne, np, v2c, vvc, v_countc, v_depc, i1c, i2c, i3c, pb2c, finc);
                }
        }
	out->r = vd->_vv;
#ifdef V2
	out->r2 = vd->_vv2;
#endif
#ifdef V3
	out->r3 = vd->_vv3;
#endif
	return 0;
}

#endif
