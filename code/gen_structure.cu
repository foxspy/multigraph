#include "hip/hip_runtime.h"
#include "user_function.h"
//#include "common.h"

/*
__device__ inline int lane_id(void) { return (threadIdx.x&31); }

__device__ int warp_bcast(int v, int leader) { return __shfl(v, leader); }

__device__ int atomicAggInc(int *ctr) {
        int mask = __ballot(1);
        int leader = __ffs(mask) - 1;
        int res;
        if(lane_id() == leader)
                res = atomicAdd(ctr, __popc(mask));
        res = warp_bcast(res, leader);

        return (res + __popc(mask & ((1 << lane_id()) - 1)));
}

__device__ short atomicAddShort(short* address, short val) {
        unsigned int *base_address = (unsigned int *) ((char *)address - ((size_t)address & 2));
        unsigned int long_val = ((size_t)address & 2) ? ((unsigned int)val << 16) : (unsigned short)val;
        unsigned int long_old = atomicAdd(base_address, long_val);
        if((size_t)address & 2) {
                return (short)(long_old >> 16);
        } else {
                unsigned int overflow = ((long_old & 0xffff) + long_val) & 0xffff0000;
                if (overflow)
                        atomicSub(base_address, overflow);
                return (short)(long_old & 0xffff);
        }
}*/

__global__ void s_preprocessing(int nv, int *csc_v, int *csc_e, int *p1, int *p2)
{
	__shared__ int buffer1[64], buffer2[64], buffer3[64], buffer4[64];
	__shared__ int buffer_p[2];

	int i, j;
	int index = blockIdx.x*64 + (threadIdx.x>>2), bias;

	if(threadIdx.x < 2) {
		buffer_p[threadIdx.x] = 0;
	}
	__syncthreads();

	if(index < nv) {
		bias = 0;		
		int index_size = csc_v[index+1] - csc_v[index];

	        if(index_size >= 32) {
	                bias = index_size - (index_size&31);
	                if((threadIdx.x&3) == 0) {
        	                int p = atomicAggInc(&buffer_p[0]);
	                        buffer1[p] = index;
	                        buffer2[p] = bias;
	                }
	                if(index_size >= 256) {
	                        if((threadIdx.x&3) == 0) {
        	                        int p2 = atomicAggInc(&buffer_p[1]);
	                                buffer3[p2] = index;
	                                buffer4[p2] = index_size - (index_size&255);
	                        }
	                }
        	}

		for(i=bias+(threadIdx.x&3); i<index_size; i+=4) {
			int dst0 = csc_v[index]+i;
			int index_dst = csc_e[dst0];
			p1[dst0] = index;
			p2[dst0] = index_dst;
		}
	}
	__syncthreads();

	for(i=(threadIdx.x>>5);i<buffer_p[0];i+=8) {
		index = buffer1[i];
		int bf2 = buffer2[i];
		int bf22 = bf2 - (bf2&255);
		for(j=bf22+(threadIdx.x&31);j<bf2;j+=32) {
			int dst0 = csc_v[index]+j;
			int index_dst = csc_e[dst0];
			p1[dst0] = index;
			p2[dst0] = index_dst;
		}
	}
	for(i=0;i<buffer_p[1];i++) {
		index = buffer3[i];
		for(j=threadIdx.x;j<buffer4[i];j+=blockDim.x) {
			int dst0 = csc_v[index]+j;
			int index_dst = csc_e[dst0];
			p1[dst0] = index;
			p2[dst0] = index_dst;
		
		}
	}
}

void generate_MultiGraphS(struct MULTI_SPARSE *m, struct csc_package *inp)
{

int *tt;
	hipError_t cuda_stat;

	(m->nv) = inp->nv, (m->ne) = inp->ne;
	int *csc_v = inp->csc_v, *csc_e = inp->csc_e;
	int *_csc_v, *_csc_e;
	int *_p1, *_p2;

	cuda_stat = hipMalloc((void **) &_csc_v, sizeof(int)*((inp->nv)+1));
	cuda_stat = hipMalloc((void **) &_csc_e, sizeof(int)*(inp->ne));
	hipMemcpy(_csc_v, csc_v, sizeof(int)*((inp->nv)+1), hipMemcpyHostToDevice);
	hipMemcpy(_csc_e, csc_e, sizeof(int)*(inp->ne), hipMemcpyHostToDevice);
	cuda_stat = hipMalloc((void **) &_p1, sizeof(int)*(inp->ne));
	cuda_stat = hipMalloc((void **) &_p2, sizeof(int)*(inp->ne));

	hipDeviceSynchronize();
//for(int i=0;i<(inp->ne);i++) {
//	printf("%d ", inp->csc_e[i]);
//} printf("\n");

//INT_PRINT(_csc_v, (m->nv)+1);
//INT_PRINT(_csc_e, m->ne);

	double s_time=rtclock();
	hipDeviceSynchronize();
	s_preprocessing<<<((m->nv)+255)>>6, 256>>>(m->nv, _csc_v, _csc_e, _p1, _p2);
	hipDeviceSynchronize();
	double e_time=rtclock();
	fprintf(stdout, "preprocessing : %f ms,", (e_time - s_time)*1000);

	m->_p1 = _p1; m->_p2 = _p2;
	cuda_stat = hipMalloc((void **) &(m->_finished), sizeof(int));

//INT_PRINT(_csc_v, (m->nv)+1);
//INT_PRINT(_csc_e, m->ne);
//INT_PRINT(_p1, m->ne);
//INT_PRINT(_p2, m->ne);
}


__global__ void preprocessing_step000(int nv, int *csc_v, short *csc_occ)
{
        for(int index =(blockDim.x*blockIdx.x*SFACTOR)+(threadIdx.x&31)+((threadIdx.x>>5)<<5)*SFACTOR; index < nv; index += blockDim.x*gridDim.x*SFACTOR) {
//              if(csc_occ[csc_e[index]] < GROUP_T) atomicAddShort(&csc_occ[csc_e[index]], SFACTOR);
                int t = MIN(GROUP_T, csc_v[index+1] - csc_v[index]);
		csc_occ[index] = t;
        }
}


__global__ void preprocessing_step00(int ne, int *csc_e, short *csc_occ)
{
        for(int index =(blockDim.x*blockIdx.x*SFACTOR)+(threadIdx.x&31)+((threadIdx.x>>5)<<5)*SFACTOR; index < ne; index += blockDim.x*gridDim.x*SFACTOR) {
                if(csc_occ[csc_e[index]] < GROUP_T) atomicAddShort(&csc_occ[csc_e[index]], SFACTOR);
        }
}


__global__ void preprocessing_step01(int nv, int upper_nv, short *csc_occ, int *group_occ, int *gr) {
	__shared__ int sgroup_occ[6144];

	int i, j, bias;
	for(i=threadIdx.x;i<6144;i+=blockDim.x) {
		sgroup_occ[i] = 0;
	} 
	__syncthreads();

	for(i=blockDim.x*blockIdx.x+threadIdx.x;i<nv;i+=gridDim.x*blockDim.x) {
		short k = csc_occ[i];
		//if(k > 4096) printf("err : %d\n", k);
		if(k >= GROUP_T) {
			bias=0;
			atomicAdd(&sgroup_occ[GROUP_NUM*threadIdx.x], 1);
		}
		else if(k >= (GROUP_T>>1)) {
			bias=upper_nv;
			atomicAdd(&sgroup_occ[GROUP_NUM*threadIdx.x+1], 1);
		}
		else if(k >= (GROUP_T>>2)) {
			bias=upper_nv*2;
			atomicAdd(&sgroup_occ[GROUP_NUM*threadIdx.x+2], 1);
		}
		else if(k >= (GROUP_T>>3)) {
			bias=upper_nv*3;
			atomicAdd(&sgroup_occ[GROUP_NUM*threadIdx.x+3], 1);
		}
		else if(k >= (GROUP_T>>4)) {
			bias=upper_nv*4;
			atomicAdd(&sgroup_occ[GROUP_NUM*threadIdx.x+4], 1);
		} else {
			bias=upper_nv*5;
		}
		gr[i+bias] = 1;
	}
	__syncthreads();

	//reduction
	for(j=3072;j>=6;j>>=1) {
		for(i=threadIdx.x;i<j;i+=blockDim.x) {
			sgroup_occ[i] += sgroup_occ[i+j];
		}
		 __syncthreads();
	}
	if(threadIdx.x < GROUP_NUM) {
		atomicAdd(&group_occ[threadIdx.x], sgroup_occ[threadIdx.x]);
	}
}

__global__ void preprocessing_step02(int nv, int upper_nv, const int * __restrict__ csc_v, int *cgr, int *itable, int *csc_size, short *csc_occ,
int loc1, int loc2, int loc3, int loc4, int loc5)
{
//	int index=blockDim.x*blockIdx.x+threadIdx.x;
	for(int index=blockDim.x*blockIdx.x+threadIdx.x;index<nv;index+=blockDim.x*gridDim.x) {
		short k = csc_occ[index];
		int t;
		if(k >= GROUP_T) {
			itable[index] = cgr[index];
		} else if(k >= (GROUP_T>>1)) {
			itable[index] = loc1+cgr[index+upper_nv];
		} else if(k >= (GROUP_T>>2)) {
			itable[index] = loc2+cgr[index+upper_nv*2];
		} else if(k >= (GROUP_T>>3)) {
			itable[index] = loc3+cgr[index+upper_nv*3];
		} else if(k >= (GROUP_T>>4)) {
			itable[index] = loc4+cgr[index+upper_nv*4];
		} else {
			itable[index] = loc5+cgr[index+upper_nv*5];
		}
//if(itable[index] != index) printf("err : %d\n", itable[index], index);

		t = itable[index];
/*
if(t < 0 || t >= nv) {
int kkkk;
if(k >= GROUP_T) {
kkkk = cgr[index]-1;
} else if(k >= (GROUP_T>>1)) {
kkkk = loc1+cgr[index+upper_nv]-1;
} else if(k >= (GROUP_T>>2)) {
kkkk = loc2+cgr[index+upper_nv*2]-1;
} else if(k >= (GROUP_T>>3)) {
kkkk = loc3+cgr[index+upper_nv*3]-1;
} else if(k >= (GROUP_T>>4)) {
kkkk = loc4+cgr[index+upper_nv*4]-1;
} else {
kkkk = loc5+cgr[index+upper_nv*5]-1;
}

printf("err %d %d %d %d\n", index, csc_occ[index], kkkk, t);
}*/
		csc_size[t] = csc_v[index+1] - csc_v[index];
	}
}




__global__ void preprocessing_step03(int nv, int niter, const int * __restrict__ itable, int *csc_v, int *csc_e, int *ncsc_v, int *tmp_e, int *ncsc_e,
int g1, int g2, int g3, int g4, int g5
#ifdef E1
, E1T *csc_ev, E1T *ncsc_ev
#endif
)
{
        __shared__ int buffer1[64], buffer2[64], buffer22[64], buffer3[64];
        __shared__ int buffer01[64];
        __shared__ int buffer_p[2];
        __shared__ int sleft_v[64];

//      __shared__ int sr0[64], sr1[64], sr2[64], sr3[64], sr4[64], sr5[64];
        __shared__ int sr[64*6];

        int index, index_size, diff;
        int i, j, bias=0;
        int r0, r1, r2, r3, r4, r5;

        index=blockIdx.x*64+(threadIdx.x>>1);
//      for(index=blockIdx.x*64+(threadIdx.x>>2); index<niter; index+=gridDim.x*64) {
        for(i=threadIdx.x;i<64*6;i+=blockDim.x) {
                sr[i] = 0;
        }

                bias = 0;
                if(threadIdx.x < 2) {
                        buffer_p[threadIdx.x] = 0;
                }
                __syncthreads();
                if(index < nv) {
                        int right_v = csc_v[index];
                        index_size = csc_v[index+1] - right_v;
                        int left_v = ncsc_v[itable[index]];
                        if((threadIdx.x&1) == 0) {
                                sleft_v[(threadIdx.x>>1)] = left_v;
                        }
//                      diff = ncsc_v[itable[index]] - csc_v[index];

                        bias = (index_size&31);
                        if(index_size >= 32) {
//                              bias = index_size - (index_size&31);
                                if((threadIdx.x&1) == 0) {
                                        int p = atomicAggInc(&buffer_p[0]);
                                        buffer1[p] = (threadIdx.x>>1);
                                        buffer2[p] = bias;
                                        buffer22[p] = index_size;
                                        buffer3[p] = left_v;
                                        buffer01[p] = right_v;
                                }
                        }
                        r0 = r1 = r2 = r3 = r4 = r5 = 0;
                        for(i=(threadIdx.x&1); i<bias; i+=2) {
                                int tmp = itable[csc_e[right_v + i]];
                                if(tmp < g1) r0++;
                                else if(tmp < g2) r1++;
                                else if(tmp < g3) r2++;
                                else if(tmp < g4) r3++;
                                else if(tmp < g5) r4++;
//                              else r5++;

                                tmp_e[left_v + i] = tmp;
#ifdef E1
//                              ncsc_ev[left_v + i] = csc_ev[right_v + i];
#endif
                        }

                        r0 += __shfl_down(r0, 1);
                        r1 += __shfl_down(r1, 1);
                        r2 += __shfl_down(r2, 1);
                        r3 += __shfl_down(r3, 1);
                        r4 += __shfl_down(r4, 1);
//                      r5 += __shfl_down(r5, 1, 4);

                        if((threadIdx.x&1) == 0) {
                                sr[6*(threadIdx.x>>1)] = r0;
                                sr[6*(threadIdx.x>>1)+1] = r1;
                                sr[6*(threadIdx.x>>1)+2] = r2;
                                sr[6*(threadIdx.x>>1)+3] = r3;
                                sr[6*(threadIdx.x>>1)+4] = r4;
//                              sr[320+(threadIdx.x>>2)] = r5;

                        }

                }
                __syncthreads();

                for(i=(threadIdx.x>>5);i<buffer_p[0];i+=4) {
//                      int lindex = buffer1[i];
                        int bf2 = buffer2[i];
                        int bf22 = buffer22[i];
                        int bf3 = buffer3[i];
                        int bf4 = buffer01[i];
                        r0 = r1 = r2 = r3 = r4 = r5 = 0;
                        for(j=bf2+(threadIdx.x&31);j<bf22;j+=32) {
                                int tmp = itable[csc_e[bf4+j]];
                                if(tmp < g1) r0++;
                                else if(tmp < g2) r1++;
                                else if(tmp < g3) r2++;
                                else if(tmp < g4) r3++;
                                else if(tmp < g5) r4++;
//                              else r5++;

                                tmp_e[j+bf3] = tmp;
#ifdef E1
//                              ncsc_ev[j+bf3] = csc_ev[bf4+j];
#endif
                        }
                        r0 += __shfl_down(r0, 16);
                        r1 += __shfl_down(r1, 16);
                        r2 += __shfl_down(r2, 16);
                        r3 += __shfl_down(r3, 16);
                        r4 += __shfl_down(r4, 16);
//                      r5 += __shfl_down(r5, 16, 32);

                        r0 += __shfl_down(r0, 8);
                        r1 += __shfl_down(r1, 8);
                        r2 += __shfl_down(r2, 8);
                        r3 += __shfl_down(r3, 8);
                        r4 += __shfl_down(r4, 8);
//                      r5 += __shfl_down(r5, 8);

                        r0 += __shfl_down(r0, 4);
                        r1 += __shfl_down(r1, 4);
                        r2 += __shfl_down(r2, 4);
                        r3 += __shfl_down(r3, 4);
                        r4 += __shfl_down(r4, 4);
//                      r5 += __shfl_down(r5, 4);

                        r0 += __shfl_down(r0, 2);
                        r1 += __shfl_down(r1, 2);
                        r2 += __shfl_down(r2, 2);
                        r3 += __shfl_down(r3, 2);
                        r4 += __shfl_down(r4, 2);

                        r0 += __shfl_down(r0, 1);
                        r1 += __shfl_down(r1, 1);
                        r2 += __shfl_down(r2, 1);
                        r3 += __shfl_down(r3, 1);
                        r4 += __shfl_down(r4, 1);
//                      r5 += __shfl_down(r5, 1);

                        if((threadIdx.x&31) == 0) {
                                sr[6*buffer1[i]] += r0;
                                sr[6*buffer1[i]+1] += r1;
                                sr[6*buffer1[i]+2] += r2;
                                sr[6*buffer1[i]+3] += r3;
                                sr[6*buffer1[i]+4] += r4;
//                              sr[6*buffer1[i]+5] += r5;
                        }

                }

                __syncthreads();


        if(threadIdx.x < 64) {
                sr[6*threadIdx.x + 1] += sr[6*threadIdx.x];
                sr[6*threadIdx.x + 2] += sr[6*threadIdx.x+1];
                sr[6*threadIdx.x + 3] += sr[6*threadIdx.x+2];
                sr[6*threadIdx.x + 4] += sr[6*threadIdx.x+3];

                sr[6*threadIdx.x + 5] = sr[6*threadIdx.x+4];
                sr[6*threadIdx.x + 4] = sr[6*threadIdx.x+3];
                sr[6*threadIdx.x + 3] = sr[6*threadIdx.x+2];
                sr[6*threadIdx.x + 2] = sr[6*threadIdx.x+1];
                sr[6*threadIdx.x + 1] = sr[6*threadIdx.x];
                sr[6*threadIdx.x] = 0;
        }
                __syncthreads();



        //actual moving

        index=blockIdx.x*64+threadIdx.x;
//      for(index=blockIdx.x*64+(threadIdx.x>>2); index<niter; index+=gridDim.x*64) {

                if(index < nv && threadIdx.x < 64) {
                        int right_v = csc_v[index];
                        int left_v = sleft_v[threadIdx.x];
                        index_size = csc_v[index+1] - right_v;

                        bias = (index_size&31);


                        for(i=0; i<bias; i++) {
                                int tmp = tmp_e[left_v + i];
                                int loc;
                                if(tmp < g1) {
                                        loc = sr[6*threadIdx.x];
                                        sr[6*threadIdx.x]++;
                                } else if(tmp < g2) {
                                        loc = sr[6*threadIdx.x+1];
                                        sr[6*threadIdx.x+1]++;
                                } else if(tmp < g3) {
                                        loc = sr[6*threadIdx.x+2];
                                        sr[6*threadIdx.x+2]++;
                                } else if(tmp < g4) {
                                        loc = sr[6*threadIdx.x+3];
                                        sr[6*threadIdx.x+3]++;
                                } else if(tmp < g5) {
                                        loc = sr[6*threadIdx.x+4];
                                        sr[6*threadIdx.x+4]++;
                                } else {
                                        loc = sr[6*threadIdx.x+5];
                                        sr[6*threadIdx.x+5]++;
                                }

                                ncsc_e[left_v + loc] = tmp;
#ifdef E1
                                ncsc_ev[left_v + loc] = csc_ev[right_v + i];
#endif
                        }
                }
                __syncthreads();

                for(i=(threadIdx.x>>5);i<buffer_p[0];i+=4) {
                      int lindex = buffer1[i];
                        int bf2 = buffer2[i];
                        int bf22 = buffer22[i];
                        int bf3 = buffer3[i];
                        int bf4 = buffer01[i];
                        for(j=bf2+(threadIdx.x&31);j<bf22;j+=32) {
                                int tmp = tmp_e[bf3+j];
                                int loc;

//                              if(tmp < g1) loc = atomicAdd(&sr[6*lindex],1);
//                              else if(tmp < g2) loc = atomicAdd(&sr[6*lindex+1],1);
//                              else if(tmp < g3) loc = atomicAdd(&sr[6*lindex+2],1);
//                              else if(tmp < g4) loc = atomicAdd(&sr[6*lindex+3],1);
//                              else if(tmp < g5) loc = atomicAdd(&sr[6*lindex+4],1);
//                              else loc = atomicAdd(&sr[6*lindex+5],1);

                                if(tmp < g1) loc = atomicAggInc(&sr[6*lindex]);
                                if(tmp >= g1 && tmp < g2) loc = atomicAggInc(&sr[6*lindex+1]);
                                if(tmp >= g2 && tmp < g3) loc = atomicAggInc(&sr[6*lindex+2]);
                                if(tmp >= g3 && tmp < g4) loc = atomicAggInc(&sr[6*lindex+3]);
                                if(tmp >= g4 && tmp < g5) loc = atomicAggInc(&sr[6*lindex+4]);
                                if(tmp >= g5) loc = atomicAggInc(&sr[6*lindex+5]);



//                              if(tmp < g1) loc = atomicAggInc(&sr[6*lindex]);
//                              else if(tmp < g2) loc = atomicAggInc(&sr[6*lindex+1]);
//                              else if(tmp < g3) loc = atomicAggInc(&sr[6*lindex+2]);
//                              else if(tmp < g4) loc = atomicAggInc(&sr[6*lindex+3]);
//                              else if(tmp < g5) loc = atomicAggInc(&sr[6*lindex+4]);
//                              else loc = atomicAggInc(&sr[6*lindex+5]);

                                ncsc_e[loc+bf3] = tmp;
#ifdef E1
                                ncsc_ev[loc+bf3] = csc_ev[bf4+j];

#endif
			}
		}
}




















__global__ void preprocessing_step03r(int nv, int niter, const int * __restrict__ itable, const int *csc_v, int *csc_e, const int *ncsc_v, int *ncsc_e
#ifdef E1
, E1T *csc_ev, E1T *ncsc_ev
#endif
)
{
	__shared__ int buffer1[64], buffer2[64], buffer3[64], buffer4[64], buffer5[64], buffer6[64];
	__shared__ int buffer01[64], buffer02[64];
	__shared__ int buffer_p[2];

	int index, index_size, diff;
	int i, j, bias=0;

	index=blockIdx.x*64+(threadIdx.x>>2);
//	for(index=blockIdx.x*64+(threadIdx.x>>2); index<niter; index+=gridDim.x*64) {

		bias = 0;
		if(threadIdx.x < 2) {
			buffer_p[threadIdx.x] = 0;
		}
		__syncthreads();
		if(index < nv) {
			int right_v = csc_v[index];	
			index_size = csc_v[index+1] - right_v;		
			int left_v = right_v + ncsc_v[itable[index]] - right_v;
//			diff = ncsc_v[itable[index]] - csc_v[index];
	
		        if(index_size >= 32) {
				bias = index_size - (index_size&31);
		                if((threadIdx.x&3) == 0) {
		                        int p = atomicAggInc(&buffer_p[0]);
  //     					buffer1[p] = index;
       	                 	buffer2[p] = bias;
					buffer3[p] = left_v; 
					buffer01[p] = right_v;      
 	         	}
       	         	if(index_size >= 256) {
       	                 	if((threadIdx.x&3) == 0) {
       	                      		int p2 = atomicAggInc(&buffer_p[1]);
     //  	                        		buffer4[p2] = index;
       	                         	buffer5[p2] = index_size - (index_size&255);
						buffer6[p2] = left_v;
						buffer02[p2] = right_v;
       	                 	}
       	    	 	}
       	 	}
			for(i=bias+(threadIdx.x&3); i<index_size; i+=4) {
				ncsc_e[left_v + i] = itable[csc_e[right_v + i]];
#ifdef E1
				ncsc_ev[left_v + i] = csc_ev[right_v + i];
				
#endif
			}
		}
		__syncthreads();
	
		for(i=(threadIdx.x>>5);i<buffer_p[0];i+=8) {
//			int lindex = buffer1[i];
			int bf2 = buffer2[i];
			int bf22 = bf2 - (bf2&255);
			int bf3 = buffer3[i];
			int bf4 = buffer01[i];
			for(j=bf22+(threadIdx.x&31);j<bf2;j+=32) {
				ncsc_e[j+bf3] = itable[csc_e[bf4+j]];
#ifdef E1
				ncsc_ev[j+bf3] = csc_ev[bf4+j];

#endif
			} 
		}
		for(i=0;i<buffer_p[1];i++) {
//			int lindex = buffer4[i];
			int bf3 = buffer6[i];
			int bf4 = buffer02[i];
			for(j=threadIdx.x;j<buffer5[i];j+=blockDim.x) {
				ncsc_e[j+bf3] = itable[csc_e[bf4+j]];
#ifdef E1
				ncsc_ev[j+bf3] = csc_ev[bf4+j];

#endif
			}
		}
//		__syncthreads();

//	}
	
}


__global__ void
preprocessing_step1verybig(int *buff, int DFACTOR, int LOG_DFACTOR, int nv, int ne, int np, int upper_np, int *csc_v, const int * __restrict__ csc_e, int *es, int *pb1, int *pb2,
int *dcnt, int *dindex, int *dx, int *dy)
{

	__shared__ int occ[SM_SIZE00];
//	__shared__ int buffer1[BG], buffer3[BG];
//	__shared__ int buffer_p[2];

	int i, j, k, index, index_size, bias;

//	int dd=0, d0=0, d1=0, d2=0, d3=0;
//	int d4=0, d5=0, d6=0, d7=0;
	int dd=0;

	// get occurrance
	for(i=threadIdx.x;i<upper_np;i+=blockDim.x) {
		occ[i] = 0;
	}

	__syncthreads();
	for(index=blockIdx.x*PSIZE+(threadIdx.x>>0); index-blockIdx.x*PSIZE < PSIZE; ) {
//		__syncthreads();
		for(k=0;k<1;k++) {
			if(index < nv) {
				index_size = csc_v[index+1] - csc_v[index];
				// step 1 (small granularity)
				for(i=0; i<index_size; i++) {
//if(csc_v[index]+i < 0 || csc_v[index]+i >= ne) printf("etype1 %d %d\n", index, csc_v[index]+i);
//if(csc_e[csc_v[index]+i] < 0 || csc_e[csc_v[index]+i] >= nv) printf("etype2 %d %d\n", index, csc_e[csc_v[index]+i]);
					int index_pnt = csc_e[csc_v[index]+i] / PSIZE;
//if(DFACTOR*index_pnt+(threadIdx.x&(DFACTOR-1)) < 0 || DFACTOR*index_pnt+(threadIdx.x&(DFACTOR-1)) >= SM_SIZE00) printf("etype3 : %d %d\n", index, DFACTOR*index_pnt+(threadIdx.x&(DFACTOR-1)));
					int ti = index_pnt * DFACTOR + ((threadIdx.x>>5)&(DFACTOR-1));
					//atomicPSeg(&occ[index_pnt], index_pnt);
					if(index_pnt != blockIdx.x) {
						atomicPSeg(&occ[ti], ti);
					} else {
						dd++;
					}
	

				//occ[index_pnt]++;
				}
			}
			index += 1024;
		}
		//__syncthreads();
	}

        dd += __shfl_down(dd, 16);
        dd += __shfl_down(dd, 8);
        dd += __shfl_down(dd, 4);
        dd += __shfl_down(dd, 2);
        dd += __shfl_down(dd, 1);

        if((threadIdx.x&31)==0) {
                occ[SM_SIZE00-3-(threadIdx.x>>5)] = dd;
        }
        __syncthreads();
        if(threadIdx.x<32) {
                dd = occ[SM_SIZE00-3-threadIdx.x];
                dd += __shfl_down(dd, 16);
                dd += __shfl_down(dd, 8);
                dd += __shfl_down(dd, 4);
                dd += __shfl_down(dd, 2);
                dd += __shfl_down(dd, 1);
        }
        __syncthreads();
	
	if(threadIdx.x == 0) {
		occ[DFACTOR*blockIdx.x] = dd;
	}

	for(i=DFACTOR*np+threadIdx.x;i<SM_SIZE00;i+=blockDim.x) {
		occ[i] = 0;
	}
	__syncthreads();

	//make dense-information, normalization
	for(i=threadIdx.x;i<np;i+=blockDim.x) {
		int t=0;
		for(j=0;j<DFACTOR-1;j++) {
			t += occ[i*DFACTOR+j];
			occ[i*DFACTOR+j] = 0;
		}
		int t2=t+occ[i*DFACTOR+DFACTOR-1];
		occ[i*DFACTOR+DFACTOR-1] = 0;
		t2 = (((t2+7)>>3)<<3);
		//occ[i*DFACTOR+DFACTOR-1] = t2-t;
		occ[i*DFACTOR] = t2;
	}
	__syncthreads();

	int buffer_p = occ[SM_SIZE00-2];
	// init stream buffer
	for(i=threadIdx.x;i<np;i+=blockDim.x) { //actually, 1*np size is enough
		if(occ[DFACTOR*i] >=0) {
			pb2[blockIdx.x+np*i] = occ[DFACTOR*i]; //transposed
		} else {
			pb2[blockIdx.x+np*i] = PSIZE;
			occ[DFACTOR*i] = 0;
		}		
	}
	__syncthreads();

	if(threadIdx.x == 0) {
		occ[SM_SIZE00-2] = 0;
	}

	//prefix-sum for pb1 ( 8 chunks)

    int sync_upper_np = upper_np;
    if((upper_np&512) > 0) sync_upper_np += 512;

//if(threadIdx.x == 0 && blockIdx.x == 0) printf("(%d %d)\n", upper_np, sync_upper_np);

    for(i=threadIdx.x;i<sync_upper_np;i+=blockDim.x) {
	int d = (np_base>>1);
	int offset=1;
	for(; d > 0; d>>=1) {
		__syncthreads();
		j=(i&511);
		if(j < d && i<upper_np) {
			int a0 = (i>>9)*np_base;
			int ai = offset*(2*j+1)-1;
			int bi = offset*(2*j+2)-1;
//			if(ai < np*DFACTOR && bi < np*DFACTOR) {
				occ[a0+bi] += occ[a0+ai];
//			}
		}
		offset *= 2;
	}
    }
	__syncthreads();

	if(threadIdx.x < (upper_np>>9)) {
//		if(threadIdx.x == 0) es[blockIdx.x] = occ[-1];
		buff[blockIdx.x*32+threadIdx.x] = occ[(threadIdx.x+1)*np_base-1];
		occ[(threadIdx.x+1)*np_base-1] = 0;
	}

    for(i=threadIdx.x;i<sync_upper_np;i+=blockDim.x) {
		int offset = 512;
	for(int d=1; d<np_base; d*=2) {
		offset >>= 1;
		__syncthreads();
		j=(i&511);
		if(j < d & i<upper_np) {
			int a0 = (i>>9)*np_base;
			int ai = offset*(2*j+1)-1;
			int bi = offset*(2*j+2)-1;
//			if(ai < np*DFACTOR && bi < np*DFACTOR) {
				int dummy = occ[a0+ai];
				occ[a0+ai] = occ[a0+bi];
				occ[a0+bi] += dummy; 
//			}
			}
	}
    }
	__syncthreads();
	int base_value = 0;
	for(i=1;i<(upper_np>>9);i++) {
		base_value += buff[blockIdx.x*32+i-1];
		for(j=threadIdx.x;j<np_base;j+=blockDim.x) {
			occ[i*np_base+j] += base_value;
		}
		__syncthreads();
	}
	if(threadIdx.x == 0) {
		es[blockIdx.x] = occ[np_base*i-1];
	}
	__syncthreads();

	for(i=threadIdx.x; i<np; i+=blockDim.x) {
		pb1[blockIdx.x*np+i] = occ[i*DFACTOR];
	}	
}




__global__ void
preprocessing_step1big(int DFACTOR, int LOG_DFACTOR, int nv, int ne, int np, int upper_np, int *csc_v, const int * __restrict__ csc_e, int *es, int *pb1, int *pb2,
int *dcnt, int *dindex, int *dx, int *dy)
{

	__shared__ int occ[SM_SIZE0];
//	__shared__ int buffer1[BG], buffer3[BG];
//	__shared__ int buffer_p[2];

	int i, j, k, index, index_size, bias;

//	int dd=0, d0=0, d1=0, d2=0, d3=0;
//	int d4=0, d5=0, d6=0, d7=0;
	int dd=0;

	// get occurrance
	for(i=threadIdx.x;i<upper_np;i+=blockDim.x) {
		occ[i] = 0;
	}

	__syncthreads();
	for(index=blockIdx.x*PSIZE+(threadIdx.x>>0); index-blockIdx.x*PSIZE < PSIZE; ) {
//		__syncthreads();
		for(k=0;k<1;k++) {
			if(index < nv) {
				index_size = csc_v[index+1] - csc_v[index];
				// step 1 (small granularity)
				for(i=0; i<index_size; i++) {
//if(csc_v[index]+i < 0 || csc_v[index]+i >= ne) printf("etype1 %d %d\n", index, csc_v[index]+i);
//if(csc_e[csc_v[index]+i] < 0 || csc_e[csc_v[index]+i] >= nv) printf("etype2 %d %d\n", index, csc_e[csc_v[index]+i]);
					int index_pnt = csc_e[csc_v[index]+i] / PSIZE;
//if(DFACTOR*index_pnt+(threadIdx.x&(DFACTOR-1)) < 0 || DFACTOR*index_pnt+(threadIdx.x&(DFACTOR-1)) >= SM_SIZE0) printf("etype3 : %d %d\n", index, DFACTOR*index_pnt+(threadIdx.x&(DFACTOR-1)));
					int ti = index_pnt * DFACTOR + ((threadIdx.x>>5)&(DFACTOR-1));
					//atomicPSeg(&occ[index_pnt], index_pnt);
					if(index_pnt != blockIdx.x) {
						atomicPSeg(&occ[ti], ti);
					} else {
						dd++;
					}
	

				//occ[index_pnt]++;
				}
			}
			index += 1024;
		}
		//__syncthreads();
	}

        dd += __shfl_down(dd, 16);
        dd += __shfl_down(dd, 8);
        dd += __shfl_down(dd, 4);
        dd += __shfl_down(dd, 2);
        dd += __shfl_down(dd, 1);

        if((threadIdx.x&31)==0) {
                occ[SM_SIZE0-3-(threadIdx.x>>5)] = dd;
        }
        __syncthreads();
        if(threadIdx.x<32) {
                dd = occ[SM_SIZE0-3-threadIdx.x];
                dd += __shfl_down(dd, 16);
                dd += __shfl_down(dd, 8);
                dd += __shfl_down(dd, 4);
                dd += __shfl_down(dd, 2);
                dd += __shfl_down(dd, 1);
        }
        __syncthreads();
	
	if(threadIdx.x == 0) {
		occ[DFACTOR*blockIdx.x] = dd;
	}

	if(threadIdx.x < 2) {
		occ[SM_SIZE0-2+threadIdx.x] = 0;
	}
	__syncthreads();

	//make dense-information, normalization
	for(i=threadIdx.x;i<np;i+=blockDim.x) {
		int t=0;
		for(j=0;j<DFACTOR-1;j++) {
			t += occ[i*DFACTOR+j];
			occ[i*DFACTOR+j] = 0;
		}
		int t2=t+occ[i*DFACTOR+DFACTOR-1];
		occ[i*DFACTOR+DFACTOR-1] = 0;
		t2 = (((t2+7)>>3)<<3);
		//occ[i*DFACTOR+DFACTOR-1] = t2-t;
		occ[i*DFACTOR] = t2;
	}
	__syncthreads();

	int buffer_p = occ[SM_SIZE0-2];
	// init stream buffer
	for(i=threadIdx.x;i<np;i+=blockDim.x) { //actually, 1*np size is enough
		if(occ[DFACTOR*i] >=0) {
			pb2[blockIdx.x+np*i] = occ[DFACTOR*i]; //transposed
		} else {
			pb2[blockIdx.x+np*i] = PSIZE;
			occ[DFACTOR*i] = 0;
		}		
	}
	__syncthreads();

	if(threadIdx.x == 0) {
		occ[SM_SIZE0-2] = 0;
	}

	//prefix-sum for pb1 ( 8 chunks)

    int sync_upper_np = upper_np;
    if((upper_np&512) > 0) sync_upper_np += 512;

//if(threadIdx.x == 0 && blockIdx.x == 0) printf("(%d %d)\n", upper_np, sync_upper_np);

    for(i=threadIdx.x;i<sync_upper_np;i+=blockDim.x) {
	int d = (np_base>>1);
	int offset=1;
	for(; d > 0; d>>=1) {
		__syncthreads();
		j=(i&511);
		if(j < d && i<upper_np) {
			int a0 = (i>>9)*np_base;
			int ai = offset*(2*j+1)-1;
			int bi = offset*(2*j+2)-1;
//			if(ai < np*DFACTOR && bi < np*DFACTOR) {
				occ[a0+bi] += occ[a0+ai];
//			}
		}
		offset *= 2;
	}
    }
	__syncthreads();

	if(threadIdx.x < (upper_np>>9)) {
//		if(threadIdx.x == 0) es[blockIdx.x] = occ[-1];
		occ[upper_np+threadIdx.x] = occ[(threadIdx.x+1)*np_base-1];
		occ[(threadIdx.x+1)*np_base-1] = 0;
	}

    for(i=threadIdx.x;i<sync_upper_np;i+=blockDim.x) {
		int offset = 512;
	for(int d=1; d<np_base; d*=2) {
		offset >>= 1;
		__syncthreads();
		j=(i&511);
		if(j < d & i<upper_np) {
			int a0 = (i>>9)*np_base;
			int ai = offset*(2*j+1)-1;
			int bi = offset*(2*j+2)-1;
//			if(ai < np*DFACTOR && bi < np*DFACTOR) {
				int dummy = occ[a0+ai];
				occ[a0+ai] = occ[a0+bi];
				occ[a0+bi] += dummy; 
//			}
			}
	}
    }
	__syncthreads();
	int base_value = 0;
	for(i=1;i<(upper_np>>9);i++) {
		base_value += occ[upper_np+i-1];
		for(j=threadIdx.x;j<np_base;j+=blockDim.x) {
			occ[i*np_base+j] += base_value;
		}
		__syncthreads();
	}
	if(threadIdx.x == 0) {
		es[blockIdx.x] = occ[np_base*i-1];
	}
	__syncthreads();

	for(i=threadIdx.x; i<np; i+=blockDim.x) {
		pb1[blockIdx.x*np+i] = occ[i*DFACTOR];
	}	
}




__global__ void
preprocessing_step1medium(int DFACTOR, int LOG_DFACTOR, int nv, int ne, int np, int upper_np, int *csc_v, const int * __restrict__ csc_e, int *es, int *pb1, int *pb2,
int *dcnt, int *dindex, int *dx, int *dy)
{

	__shared__ int occ[SM_SIZE0];
//	__shared__ int buffer1[BG], buffer3[BG];
//	__shared__ int buffer_p[2];

	int i, j, k, index, index_size, bias;

//	int dd=0, d0=0, d1=0, d2=0, d3=0;
//	int d4=0, d5=0, d6=0, d7=0;
	int dd=0;

	// get occurrance
	for(i=threadIdx.x;i<upper_np;i+=blockDim.x) {
		occ[i] = 0;
	}

	__syncthreads();
	for(index=blockIdx.x*PSIZE+(threadIdx.x>>2); index-blockIdx.x*PSIZE < PSIZE; ) {
//		__syncthreads();
		for(k=0;k<4;k++) {
			if(index < nv) {
				index_size = csc_v[index+1] - csc_v[index];
				// step 1 (small granularity)
				for(i=(threadIdx.x&3); i<index_size; i+=4) {
//if(csc_v[index]+i < 0 || csc_v[index]+i >= ne) printf("etype1 %d %d\n", index, csc_v[index]+i);
//if(csc_e[csc_v[index]+i] < 0 || csc_e[csc_v[index]+i] >= nv) printf("etype2 %d %d\n", index, csc_e[csc_v[index]+i]);
					int index_pnt = csc_e[csc_v[index]+i] / PSIZE;
//if(DFACTOR*index_pnt+(threadIdx.x&(DFACTOR-1)) < 0 || DFACTOR*index_pnt+(threadIdx.x&(DFACTOR-1)) >= SM_SIZE0) printf("etype3 : %d %d\n", index, DFACTOR*index_pnt+(threadIdx.x&(DFACTOR-1)));
					int ti = index_pnt * DFACTOR + ((threadIdx.x>>5)&(DFACTOR-1));
					//atomicPSeg(&occ[index_pnt], index_pnt);
					if(index_pnt != blockIdx.x) {
						atomicPSeg(&occ[ti], ti);
					} else {
						dd++;
					}
	

				//occ[index_pnt]++;
				}
			}
			index += 256;
		}
		//__syncthreads();
	}

        dd += __shfl_down(dd, 16);
        dd += __shfl_down(dd, 8);
        dd += __shfl_down(dd, 4);
        dd += __shfl_down(dd, 2);
        dd += __shfl_down(dd, 1);

        if((threadIdx.x&31)==0) {
                occ[SM_SIZE0-3-(threadIdx.x>>5)] = dd;
        }
        __syncthreads();
        if(threadIdx.x<32) {
                dd = occ[SM_SIZE0-3-threadIdx.x];
                dd += __shfl_down(dd, 16);
                dd += __shfl_down(dd, 8);
                dd += __shfl_down(dd, 4);
                dd += __shfl_down(dd, 2);
                dd += __shfl_down(dd, 1);
        }
        __syncthreads();
	
	if(threadIdx.x == 0) {
		occ[DFACTOR*blockIdx.x] = dd;
	}

	if(threadIdx.x < 2) {
		occ[SM_SIZE0-2+threadIdx.x] = 0;
	}
	__syncthreads();

	//make dense-information, normalization
	for(i=threadIdx.x;i<np;i+=blockDim.x) {
		int t=0;
		for(j=0;j<DFACTOR-1;j++) {
			t += occ[i*DFACTOR+j];
			occ[i*DFACTOR+j] = 0;
		}
		int t2=t+occ[i*DFACTOR+DFACTOR-1];
		occ[i*DFACTOR+DFACTOR-1] = 0;
		t2 = (((t2+7)>>3)<<3);
		//occ[i*DFACTOR+DFACTOR-1] = t2-t;
		occ[i*DFACTOR] = t2;
	}
	__syncthreads();

	int buffer_p = occ[SM_SIZE0-2];
	// init stream buffer
	for(i=threadIdx.x;i<np;i+=blockDim.x) { //actually, 1*np size is enough
		if(occ[DFACTOR*i] >=0) {
			pb2[blockIdx.x+np*i] = occ[DFACTOR*i]; //transposed
		} else {
			pb2[blockIdx.x+np*i] = PSIZE;
			occ[DFACTOR*i] = 0;
		}		
	}
	__syncthreads();

	if(threadIdx.x == 0) {
		occ[SM_SIZE0-2] = 0;
	}

	//prefix-sum for pb1 ( 8 chunks)

    int sync_upper_np = upper_np;
    if((upper_np&512) > 0) sync_upper_np += 512;

//if(threadIdx.x == 0 && blockIdx.x == 0) printf("(%d %d)\n", upper_np, sync_upper_np);

    for(i=threadIdx.x;i<sync_upper_np;i+=blockDim.x) {
	int d = (np_base>>1);
	int offset=1;
	for(; d > 0; d>>=1) {
		__syncthreads();
		j=(i&511);
		if(j < d && i<upper_np) {
			int a0 = (i>>9)*np_base;
			int ai = offset*(2*j+1)-1;
			int bi = offset*(2*j+2)-1;
//			if(ai < np*DFACTOR && bi < np*DFACTOR) {
				occ[a0+bi] += occ[a0+ai];
//			}
		}
		offset *= 2;
	}
    }
	__syncthreads();

	if(threadIdx.x < (upper_np>>9)) {
//		if(threadIdx.x == 0) es[blockIdx.x] = occ[-1];
		occ[upper_np+threadIdx.x] = occ[(threadIdx.x+1)*np_base-1];
		occ[(threadIdx.x+1)*np_base-1] = 0;
	}

    for(i=threadIdx.x;i<sync_upper_np;i+=blockDim.x) {
		int offset = 512;
	for(int d=1; d<np_base; d*=2) {
		offset >>= 1;
		__syncthreads();
		j=(i&511);
		if(j < d & i<upper_np) {
			int a0 = (i>>9)*np_base;
			int ai = offset*(2*j+1)-1;
			int bi = offset*(2*j+2)-1;
//			if(ai < np*DFACTOR && bi < np*DFACTOR) {
				int dummy = occ[a0+ai];
				occ[a0+ai] = occ[a0+bi];
				occ[a0+bi] += dummy; 
//			}
			}
	}
    }
	__syncthreads();
	int base_value = 0;
	for(i=1;i<(upper_np>>9);i++) {
		base_value += occ[upper_np+i-1];
		for(j=threadIdx.x;j<np_base;j+=blockDim.x) {
			occ[i*np_base+j] += base_value;
		}
		__syncthreads();
	}
	if(threadIdx.x == 0) {
		es[blockIdx.x] = occ[np_base*i-1];
	}
	__syncthreads();

	for(i=threadIdx.x; i<np; i+=blockDim.x) {
		pb1[blockIdx.x*np+i] = occ[i*DFACTOR];
	}	
}








__global__ void
__launch_bounds__(BSIZE, 1)
preprocessing_step1(int DFACTOR, int LOG_DFACTOR, int nv, int ne, int np, int upper_np, int *csc_v, int *csc_e, int *es, int *pb1, int *pb2,
int *dcnt, int *dindex, int *dx, int *dy)
{

	__shared__ int occ[SM_SIZE];
	__shared__ short buffer1[BG], buffer3[BG];
//	__shared__ int buffer_p[2];

	int i, j, k, index, index_size, bias;

//	int dd=0, d0=0, d1=0, d2=0, d3=0;
//	int d4=0, d5=0, d6=0, d7=0;

	// get occurrance
	for(i=threadIdx.x;i</*upper_np*/SM_SIZE;i+=blockDim.x) {
		occ[i] = 0;
	}

	//__syncthreads();
	for(index=blockIdx.x*PSIZE+(threadIdx.x>>2); index-blockIdx.x*PSIZE < PSIZE; ) {
		if(threadIdx.x < 2) {
			occ[SM_SIZE-2+threadIdx.x] = 0;
		}
		__syncthreads();
		for(k=0;k<4;k++) {
			if(index < nv) {
				index_size = csc_v[index+1] - csc_v[index];
				if(index_size >= 32) {
					bias = index_size - (index_size&31);
					if((threadIdx.x&3) == 0) {
						int p = atomicAggInc(&occ[SM_SIZE-2]);
						buffer1[p] = index - blockIdx.x*PSIZE;
					}
					if(index_size >= 256) {
						if((threadIdx.x&3) == 0) {
							int p2 = atomicAggInc(&occ[SM_SIZE-1]);
							buffer3[p2] = index - blockIdx.x*PSIZE;
						}
					}
				} else {
					bias = 0;
				}
				// step 1 (small granularity)
				for(i=bias+(threadIdx.x&3); i<index_size; i+=4) {
//if(csc_v[index]+i < 0 || csc_v[index]+i >= ne) printf("etype1 %d %d\n", index, csc_v[index]+i);
//if(csc_e[csc_v[index]+i] < 0 || csc_e[csc_v[index]+i] >= nv) printf("etype2 %d %d\n", index, csc_e[csc_v[index]+i]);
					int index_pnt = csc_e[csc_v[index]+i] / PSIZE;
//if(DFACTOR*index_pnt+(threadIdx.x&(DFACTOR-1)) < 0 || DFACTOR*index_pnt+(threadIdx.x&(DFACTOR-1)) >= SM_SIZE) printf("etype3 : %d %d\n", index, DFACTOR*index_pnt+(threadIdx.x&(DFACTOR-1)));
					int ti = index_pnt * DFACTOR + ((threadIdx.x>>5)&(DFACTOR-1));
					//atomicPSeg(&occ[index_pnt], index_pnt);
					atomicPSeg(&occ[ti], ti);

/*
					if(index_pnt == blockIdx.x) {
						dd++;
					} else if(index_pnt < 8) {
						if(index_pnt == 0) d0++;
						else if(index_pnt == 1) d1++;
						else if(index_pnt == 2) d2++;
						else if(index_pnt == 3) d3++;
						else if(index_pnt == 4) d4++;
						else if(index_pnt == 5) d5++;
						else if(index_pnt == 6) d6++;
						else if(index_pnt == 7) d7++;
					} else {
						int ix = DFACTOR*index_pnt+((threadIdx.x>>5)&(DFACTOR-1));
						atomicSeg(&occ[ix], ix);
						int tr = __shfl(index_pnt, 0);
						int tr2 = __shfl(index_pnt, 16);
						if(tr == index_pnt || tr2 == index_pnt) {
							if(tr == index_pnt) atomicAggInc(&occ[DFACTOR*index_pnt+((threadIdx.x>>5)&(DFACTOR-1))]);
							if(tr != tr2 && tr2 == index_pnt) atomicAggInc(&occ[DFACTOR*index_pnt+((threadIdx.x>>5)&(DFACTOR-1))]);
						} else {
							atomicAdd(&occ[DFACTOR*index_pnt+((threadIdx.x>>5)&(DFACTOR-1))], 1);
						}
					}*/



				//occ[index_pnt]++;
				}
			}
			index += 256;
		}
		__syncthreads();
		// step 2 (medium granularity)
		for(i=(threadIdx.x>>5);i<occ[SM_SIZE-2];i+=32) {
			int lindex = buffer1[i]+blockIdx.x*PSIZE;
			int dummy = csc_v[lindex+1] - csc_v[lindex];
			int bf2 = dummy - (dummy&31);
			int bf22 = bf2 - (bf2&255);
			for(j=bf22+(threadIdx.x&31);j<bf2;j+=32) {
//if(csc_v[lindex]+j < 0 || csc_v[lindex]+j >= ne) printf("etype2-1 %d %d\n", lindex, csc_v[lindex]+j);
//if(csc_e[csc_v[lindex]+j] < 0 || csc_e[csc_v[lindex]+j] >= nv) printf("etype2-2 %d %d\n", lindex, csc_e[csc_v[lindex]+j]);
				int index_pnt = csc_e[csc_v[lindex]+j] / PSIZE;
				int ti = index_pnt * DFACTOR + (lindex & (DFACTOR-1));
//if(DFACTOR*index_pnt+(threadIdx.x&(DFACTOR-1)) < 0 || DFACTOR*index_pnt+(threadIdx.x&(DFACTOR-1)) >= SM_SIZE) printf("etype3 : %d %d\n", lindex, DFACTOR*index_pnt+(threadIdx.x&(DFACTOR-1)));
				//atomicSeg(&occ[index_pnt], index_pnt);
				atomicSeg(&occ[ti], ti);
				/*
				if(index_pnt == blockIdx.x) {
					dd++;
				} else if(index_pnt < 8) {
					if(index_pnt == 0) d0++;
					else if(index_pnt == 1) d1++;
					else if(index_pnt == 2) d2++;
					else if(index_pnt == 3) d3++;
					else if(index_pnt == 4) d4++;
					else if(index_pnt == 5) d5++;
					else if(index_pnt == 6) d6++;
					else if(index_pnt == 7) d7++;
				} else {
					int ix = DFACTOR*index_pnt+(lindex&(DFACTOR-1));
					atomicSeg(&occ[ix], ix);
					
					int tr = __shfl(index_pnt, 0);
					int tr2 = __shfl(index_pnt, 16);
					if(tr == index_pnt || tr2 == index_pnt) {
						if(tr == index_pnt) atomicAggInc(&occ[DFACTOR*index_pnt+(lindex&(DFACTOR-1))]);
						if(tr != tr2 && tr2 == index_pnt) atomicAggInc(&occ[DFACTOR*index_pnt+(lindex&(DFACTOR-1))]);
					} else {
						atomicAdd(&occ[DFACTOR*index_pnt+(lindex&(DFACTOR-1))], 1);
					}

					//atomicAdd(&occ[DFACTOR*index_pnt+(lindex&(DFACTOR-1))], 1);
				}*/
				//occ[index_pnt]++;
			}
		}
		// step 3 (large granularity)
		for(i=(threadIdx.x>>8);i<occ[SM_SIZE-1];i+=4) {
			int lindex = buffer3[i]+blockIdx.x*PSIZE;
			int h = ((threadIdx.x>>5)&7);
			int bf2 = csc_v[lindex+1] - csc_v[lindex];
			bf2 = bf2 - (bf2&255);
			for(j=(threadIdx.x&255);j<bf2;j+=256) {
//if(csc_v[lindex]+j < 0 || csc_v[lindex]+j >= ne) printf("etype3-1 %d %d\n", lindex, csc_v[lindex]+j);
//if(csc_e[csc_v[lindex]+j] < 0 || csc_e[csc_v[lindex]+j] >= nv) printf("etype3-2 %d %d\n", lindex, csc_e[csc_v[lindex]+j]);
				int index_pnt = csc_e[csc_v[lindex]+j] / PSIZE;
//if(DFACTOR*index_pnt+(threadIdx.x&(DFACTOR-1)) < 0 || DFACTOR*index_pnt+(threadIdx.x&(DFACTOR-1)) >= SM_SIZE) printf("etype3 : %d %d\n", lindex, DFACTOR*index_pnt+(threadIdx.x&(DFACTOR-1)));
				int ti = index_pnt * DFACTOR + (h & (DFACTOR-1));
				//atomicSeg(&occ[index_pnt], index_pnt);
				atomicSeg(&occ[ti], ti);
				
				/*
				if(index_pnt == blockIdx.x) {
					dd++;
				} else if(index_pnt < 8) {
					if(index_pnt == 0) d0++;
					else if(index_pnt == 1) d1++;
					else if(index_pnt == 2) d2++;
					else if(index_pnt == 3) d3++;
					else if(index_pnt == 4) d4++;
					else if(index_pnt == 5) d5++;
					else if(index_pnt == 6) d6++;
					else if(index_pnt == 7) d7++;
				} else {
//					atomicAdd(&occ[DFACTOR*index_pnt+(lindex&(DFACTOR-1))], 1);
					int ix = DFACTOR*index_pnt+((h)&(DFACTOR-1));
					atomicSeg(&occ[ix], ix);	
					
					int tr = __shfl(index_pnt, 0);
					int tr2 = __shfl(index_pnt, 16);
					if(tr == index_pnt || tr2 == index_pnt) {
						if(tr == index_pnt) atomicAggInc(&occ[DFACTOR*index_pnt+((h)&(DFACTOR-1))]);
						if(tr != tr2 && tr2 == index_pnt) atomicAggInc(&occ[DFACTOR*index_pnt+((h)&(DFACTOR-1))]);
					} else {
						atomicAdd(&occ[DFACTOR*index_pnt+((h)&(DFACTOR-1))], 1);
					}
				}*/

				//occ[index_pnt]++;
			}
		}
		__syncthreads();
	}

//if(threadIdx.x == 0)
//	for(i=0;i<4;i++) printf("%d\n", occ[i]);

/*
	//reduction
	for(j=(DFACTOR>>1);j>=1;j>>=1) {
		for(i=threadIdx.x;i<np*j;i+=blockDim.x) {
			occ[i] += occ[i+j*np];
		}
		if(j > 1) { // safe
			 __syncthreads();
		}
	}
*/

	if(threadIdx.x < 2) {
		occ[SM_SIZE-2+threadIdx.x] = 0;
	}
	__syncthreads();
	//make dense-information, normalization
	for(i=threadIdx.x;i<np;i+=blockDim.x) {
		int t=0;
		for(j=0;j<DFACTOR-1;j++) {
			t += occ[i*DFACTOR+j];
			occ[i*DFACTOR+j] = 0;
		}
		int t2=t+occ[i*DFACTOR+DFACTOR-1];
		occ[i*DFACTOR+DFACTOR-1] = 0;
		t2 = (((t2+7)>>3)<<3);
		//occ[i*DFACTOR+DFACTOR-1] = t2-t;
		if(t2 >= THRESHOLD) {
			int k = atomicAdd(dcnt, 1);
			t2 = ((((t2+BSIZE-1)>>LOG_BSIZE)<<LOG_BSIZE)+PSIZE+BSIZE);// + PSIZE*3;
			//d_check[np*blockIdx.x+i] = k;
			dx[k] = blockIdx.x;
			dy[k] = i;
			//for(j=0;j<1;j++) {
			//	if(j < DFACTOR-1) dindex[k+j] = occ[i*DFACTOR+j];
			//	else dindex[k+j] = t2-t;
				dindex[k] = t2;
				occ[i*DFACTOR] = -1;
			//}
			int k2 = atomicAdd(&occ[SM_SIZE-2], 1);
			buffer1[k2] = i; // loc
			buffer3[k2] = k; // value
		}
		else occ[i*DFACTOR] = t2;
	}
	__syncthreads();

	int buffer_p = occ[SM_SIZE-2];
	// init stream buffer
	for(i=threadIdx.x;i<np;i+=blockDim.x) { //actually, 1*np size is enough
		if(occ[DFACTOR*i] >=0) {
			pb2[blockIdx.x+np*i] = occ[DFACTOR*i]; //transposed
		} else {
			pb2[blockIdx.x+np*i] = PSIZE;
			occ[DFACTOR*i] = 0;
		}		
	}
	__syncthreads();

	if(threadIdx.x == 0) {
		occ[SM_SIZE-2] = 0;
	}

/*
__syncthreads();
if(blockIdx.x == 0 && threadIdx.x == 0) {
	for(i=0;i<1025;i++) {
		printf("%d ", occ[i]);
	}
	printf("\nstep1\n\n");
}
__syncthreads();*/

	//prefix-sum for pb1 ( 8 chunks)

    int sync_upper_np = upper_np;
    if((upper_np&512) > 0) sync_upper_np += 512;

//if(threadIdx.x == 0 && blockIdx.x == 0) printf("(%d %d)\n", upper_np, sync_upper_np);

    for(i=threadIdx.x;i<sync_upper_np;i+=blockDim.x) {
	int d = (np_base>>1);
	int offset=1;
	for(; d > 0; d>>=1) {
		__syncthreads();
		j=(i&511);
		if(j < d && i<upper_np) {
			int a0 = (i>>9)*np_base;
			int ai = offset*(2*j+1)-1;
			int bi = offset*(2*j+2)-1;
//			if(ai < np*DFACTOR && bi < np*DFACTOR) {
				occ[a0+bi] += occ[a0+ai];
//			}
		}
		offset *= 2;
	}
    }
	__syncthreads();

	if(threadIdx.x < (upper_np>>9)) {
//		if(threadIdx.x == 0) es[blockIdx.x] = occ[-1];
		occ[upper_np+threadIdx.x] = occ[(threadIdx.x+1)*np_base-1];
		occ[(threadIdx.x+1)*np_base-1] = 0;
	}

    for(i=threadIdx.x;i<sync_upper_np;i+=blockDim.x) {
		int offset = 512;
	for(int d=1; d<np_base; d*=2) {
		offset >>= 1;
		__syncthreads();
		j=(i&511);
		if(j < d & i<upper_np) {
			int a0 = (i>>9)*np_base;
			int ai = offset*(2*j+1)-1;
			int bi = offset*(2*j+2)-1;
//			if(ai < np*DFACTOR && bi < np*DFACTOR) {
				int dummy = occ[a0+ai];
				occ[a0+ai] = occ[a0+bi];
				occ[a0+bi] += dummy; 
//			}
			}
	}
    }
	__syncthreads();
	int base_value = 0;
	for(i=1;i<(upper_np>>9);i++) {
		base_value += occ[upper_np+i-1];
		for(j=threadIdx.x;j<np_base;j+=blockDim.x) {
			occ[i*np_base+j] += base_value;
		}
		__syncthreads();
	}
	__syncthreads();
	if(threadIdx.x == 0) {
		es[blockIdx.x] = occ[np_base*i-1];
	}
	__syncthreads();


/*
__syncthreads();
if(blockIdx.x == 0 && threadIdx.x == 0) {
	for(i=0;i<1025;i++) {
		printf("%d ", occ[i]);
	}
	printf("\nstep2\n\n");
}
__syncthreads();
return;*/

/*
if(blockIdx.x == 0 && threadIdx.x == 0) {
	for(i=0;i<np*DFACTOR; i++)
		printf("(%d) ", occ[i]);
	printf("\n"); 
} __syncthreads();
*/
	for(i=threadIdx.x; i<buffer_p; i+=blockDim.x) {
		occ[DFACTOR*buffer1[i]] = -(buffer3[i]+1);
	}
	__syncthreads();

	for(i=threadIdx.x; i<np; i+=blockDim.x) {
		pb1[blockIdx.x*np+i] = occ[i*DFACTOR];
	}	


}

__global__ void	preprocessing_step11(int np, int DFACTOR, int *cpb2, int *cey)
{
	int index=blockIdx.x*blockDim.x+threadIdx.x;
	if(index <= np)
		cey[index] = cpb2[np*DFACTOR*index];
}




__global__ void
preprocessing_step2verybig(int DFACTOR, int nv, int ne, int upper_ne, int np, int upper_np, int *csc_v, int *csc_e, const int * __restrict__ pb1, const int * __restrict__ pb2, int *es,
int dcnt, int *dindex, int *dx, int *dy, short *sp1, short *sp2, short *fx, short *fy, int *mapper
#ifdef E1
, E1T *csc_ev, E1T *sp1v, E1T *fz
#endif
)
{
	__shared__ int occ[SM_SIZE00];
//        __shared__ int buffer1[BG], buffer3[BG];
//        __shared__ int buffer_p[2];

//	__shared__ int spb1[SM_SIZE00];
//	__shared__ int spb2[SM_SIZE00];

	int i, j, index, index_size, bias;

	for(i=threadIdx.x;i<np;i+=blockDim.x) {
		occ[i] = 0; 
//		spb1[i] = pb1[np*DFACTOR*blockIdx.x+i];
//		spb2[i] = pb2[np*DFACTOR*blockIdx.x+i];
	}
	__syncthreads();

        for(index=blockIdx.x*PSIZE+(threadIdx.x>>0); index-blockIdx.x*PSIZE < PSIZE; ) {
//                if(threadIdx.x < 2) {
//                        occ[SM_SIZE00-2+threadIdx.x] = 0;
//                }
//                __syncthreads();
                for(int lk=0;lk<1;lk++) {
                        if(index < nv) {
                                index_size = csc_v[index+1] - csc_v[index];
                                // step 1 (small granularity)
                                for(i=0; i<index_size; i++) {
                                        int index_pnt = csc_e[csc_v[index]+i] / PSIZE;
					int k;
					//int ti = index_pnt * DFACTOR + ((threadIdx.x>>5)&(DFACTOR-1));
					//k = atomicPSeg(&occ[ti], ti);
					if(index_pnt != blockIdx.x) {
						k = atomicPSeg(&occ[index_pnt], index_pnt);
					} 
					if(index_pnt == blockIdx.x) {
						k = atomicAggInc(&occ[index_pnt]);
					}
					int flag = pb1[blockIdx.x*np+index_pnt];
					int dst_v0 = csc_v[index]+i;
					int dst_v = csc_e[dst_v0] - index_pnt * PSIZE; //dst
//if(dst_v < 0 || dst_v >= PSIZE) printf("e : %d\n", dst_v);
					if(flag >= 0) { //sparse

						flag += es[blockIdx.x];
//if(k+flag < 0 || k+flag >= upper_ne) printf("bound1 %d (%d %d %d %d) %d\n",k, pb1[0],pb1[1],pb1[2],pb1[3], upper_ne);
						sp1[k+flag] = index - blockIdx.x*PSIZE; //src
#ifdef E1
						sp1v[k+flag] = csc_ev[dst_v0]; //src
//sp1v[k+flag] = 100;
//printf("%d %d %d\n", k+flag, dst_v0, csc_ev[dst_v0]);
#endif
						sp2[k+pb2[index_pnt*np+blockIdx.x]] = dst_v; //dst

						if(((k+flag)&7) == 0) {
							mapper[(k+flag)>>3] = k+pb2[index_pnt*np+blockIdx.x];
						}
					}
                                }
                        }
                        index += 1024;
		}
        }
}




__global__ void
preprocessing_step2big(int DFACTOR, int nv, int ne, int upper_ne, int np, int upper_np, int *csc_v, int *csc_e, const int * __restrict__ pb1, const int * __restrict__ pb2, int *es,
int dcnt, int *dindex, int *dx, int *dy, short *sp1, short *sp2, short *fx, short *fy, int *mapper
#ifdef E1
, E1T *csc_ev, E1T *sp1v, E1T *fz
#endif
)
{
	__shared__ int occ[SM_SIZE0];
//        __shared__ int buffer1[BG], buffer3[BG];
//        __shared__ int buffer_p[2];

//	__shared__ int spb1[SM_SIZE0];
//	__shared__ int spb2[SM_SIZE0];

	int i, j, index, index_size, bias;

	for(i=threadIdx.x;i<np;i+=blockDim.x) {
		occ[i] = 0; 
//		spb1[i] = pb1[np*DFACTOR*blockIdx.x+i];
//		spb2[i] = pb2[np*DFACTOR*blockIdx.x+i];
	}
	__syncthreads();

        for(index=blockIdx.x*PSIZE+(threadIdx.x>>0); index-blockIdx.x*PSIZE < PSIZE; ) {
//                if(threadIdx.x < 2) {
//                        occ[SM_SIZE0-2+threadIdx.x] = 0;
//                }
//                __syncthreads();
                for(int lk=0;lk<1;lk++) {
                        if(index < nv) {
                                index_size = csc_v[index+1] - csc_v[index];
                                // step 1 (small granularity)
                                for(i=0; i<index_size; i++) {
                                        int index_pnt = csc_e[csc_v[index]+i] / PSIZE;
					int k;
					//int ti = index_pnt * DFACTOR + ((threadIdx.x>>5)&(DFACTOR-1));
					//k = atomicPSeg(&occ[ti], ti);
					if(index_pnt != blockIdx.x) {
						k = atomicPSeg(&occ[index_pnt], index_pnt);
					} 
					if(index_pnt == blockIdx.x) {
						k = atomicAggInc(&occ[index_pnt]);
					}
					int flag = pb1[blockIdx.x*np+index_pnt];
					int dst_v0 = csc_v[index]+i;
					int dst_v = csc_e[dst_v0] - index_pnt * PSIZE; //dst
//if(dst_v < 0 || dst_v >= PSIZE) printf("e : %d\n", dst_v);
					if(flag >= 0) { //sparse

						flag += es[blockIdx.x];
//if(k+flag < 0 || k+flag >= upper_ne) printf("bound1 %d (%d %d %d %d) %d\n",k, pb1[0],pb1[1],pb1[2],pb1[3], upper_ne);
						sp1[k+flag] = index - blockIdx.x*PSIZE; //src
#ifdef E1
						sp1v[k+flag] = csc_ev[dst_v0]; //src
//sp1v[k+flag] = 100;
//printf("%d %d %d\n", k+flag, dst_v0, csc_ev[dst_v0]);
#endif
						sp2[k+pb2[index_pnt*np+blockIdx.x]] = dst_v; //dst

						if(((k+flag)&7) == 0) {
							mapper[(k+flag)>>3] = k+pb2[index_pnt*np+blockIdx.x];
						}
					}
                                }
                        }
                        index += 1024;
		}
        }
}


__global__ void
__launch_bounds__(BSIZE, 1)
preprocessing_step2medium(int DFACTOR, int nv, int ne, int upper_ne, int np, int upper_np, int *csc_v, int *csc_e, const int * __restrict__ pb1, const int * __restrict__ pb2, int *es,
int dcnt, int *dindex, int *dx, int *dy, short *sp1, short *sp2, short *fx, short *fy, int *mapper
#ifdef E1
, E1T *csc_ev, E1T *sp1v, E1T *fz
#endif
)
{
	__shared__ int occ[SM_SIZE0];
//        __shared__ int buffer1[BG], buffer3[BG];
//        __shared__ int buffer_p[2];

//	__shared__ int spb1[SM_SIZE0];
//	__shared__ int spb2[SM_SIZE0];

	int i, j, index, index_size, bias;

	for(i=threadIdx.x;i<np;i+=blockDim.x) {
		occ[i] = 0; 
//		spb1[i] = pb1[np*DFACTOR*blockIdx.x+i];
//		spb2[i] = pb2[np*DFACTOR*blockIdx.x+i];
	}
	__syncthreads();

        for(index=blockIdx.x*PSIZE+(threadIdx.x>>2); index-blockIdx.x*PSIZE < PSIZE; ) {
//                if(threadIdx.x < 2) {
//                        occ[SM_SIZE0-2+threadIdx.x] = 0;
//                }
//                __syncthreads();
                for(int lk=0;lk<4;lk++) {
                        if(index < nv) {
                                index_size = csc_v[index+1] - csc_v[index];
                                // step 1 (small granularity)
                                for(i=(threadIdx.x&3); i<index_size; i+=4) {
                                        int index_pnt = csc_e[csc_v[index]+i] / PSIZE;
					int k;
					//int ti = index_pnt * DFACTOR + ((threadIdx.x>>5)&(DFACTOR-1));
					//k = atomicPSeg(&occ[ti], ti);
					if(index_pnt != blockIdx.x) {
						k = atomicPSeg(&occ[index_pnt], index_pnt);
					} 
					if(index_pnt == blockIdx.x) {
						k = atomicAggInc(&occ[index_pnt]);
					}
					int flag = pb1[blockIdx.x*np+index_pnt];
					int dst_v0 = csc_v[index]+i;
					int dst_v = csc_e[dst_v0] - index_pnt * PSIZE; //dst
//if(dst_v < 0 || dst_v >= PSIZE) printf("e : %d\n", dst_v);
					if(flag >= 0) { //sparse

						flag += es[blockIdx.x];
//if(k+flag < 0 || k+flag >= upper_ne) printf("bound1 %d (%d %d %d %d) %d\n",k, pb1[0],pb1[1],pb1[2],pb1[3], upper_ne);
						sp1[k+flag] = index - blockIdx.x*PSIZE; //src
#ifdef E1
						sp1v[k+flag] = csc_ev[dst_v0]; //src
//sp1v[k+flag] = 100;
//printf("%d %d %d\n", k+flag, dst_v0, csc_ev[dst_v0]);
#endif
						sp2[k+pb2[index_pnt*np+blockIdx.x]] = dst_v; //dst

						if(((k+flag)&7) == 0) {
							mapper[(k+flag)>>3] = k+pb2[index_pnt*np+blockIdx.x];
						}
					}
                                }
                        }
                        index += 256;
		}
        }
}









__global__ void
__launch_bounds__(BSIZE, 1)
preprocessing_step2(int DFACTOR, int nv, int ne, int upper_ne, int np, int upper_np, int *csc_v, int *csc_e, const int * __restrict__ pb1, const int * __restrict__ pb2, int *es,
int dcnt, int *dindex, int *dx, int *dy, short *sp1, short *sp2, short *fx, short *fy, int *mapper
#ifdef E1
, E1T *csc_ev, E1T *sp1v, E1T *fz
#endif
)
{
	__shared__ int occ[SM_SIZE];
        __shared__ short buffer1[BG], buffer3[BG];
//        __shared__ int buffer_p[2];

//	__shared__ int spb1[SM_SIZE];
//	__shared__ int spb2[SM_SIZE];

	int i, j, index, index_size, bias;

	for(i=threadIdx.x;i<np;i+=blockDim.x) {
		occ[i] = 0; 
//		spb1[i] = pb1[np*DFACTOR*blockIdx.x+i];
//		spb2[i] = pb2[np*DFACTOR*blockIdx.x+i];
	}

        for(index=blockIdx.x*PSIZE+(threadIdx.x>>2); index-blockIdx.x*PSIZE < PSIZE; ) {
                if(threadIdx.x < 2) {
                        occ[SM_SIZE-2+threadIdx.x] = 0;
                }
                __syncthreads();
                for(int lk=0;lk<4;lk++) {
                        if(index < nv) {
                                index_size = csc_v[index+1] - csc_v[index];
                                if(index_size >= 32) {
                                        bias = index_size - (index_size&31);
                                        if((threadIdx.x&3) == 0) {
                                                int p = atomicAggInc(&occ[SM_SIZE-2]);
                                                buffer1[p] = index - blockIdx.x*PSIZE;
                                        }
                                        if(index_size >= 256) {
                                                if((threadIdx.x&3) == 0) {
                                                        int p2 = atomicAggInc(&occ[SM_SIZE-1]);
                                                        buffer3[p2] = index - blockIdx.x*PSIZE;
                                                }
                                        }
                                } else {
                                        bias = 0;
                                }
                                // step 1 (small granularity)
                                for(i=bias+(threadIdx.x&3); i<index_size; i+=4) {
                                        int index_pnt = csc_e[csc_v[index]+i] / PSIZE;
					int k;
//if(csc_v[index]+i < 0 || csc_v[index]+i >= ne) printf("err0 %d \n", csc_v[index]+i);
//if(csc_e[csc_v[index]+i] < 0 || csc_e[csc_v[index]+i] >= nv) printf("err %d \n", csc_e[csc_v[index]+i]);
//if(DFACTOR*index_pnt+(threadIdx.x&(DFACTOR-1)) < 0 || DFACTOR*index_pnt+(threadIdx.x&(DFACTOR-1)) >= SM_SIZE) printf("err2\n");
					//int ti = index_pnt * DFACTOR + ((threadIdx.x>>5)&(DFACTOR-1));
					//k = atomicPSeg(&occ[ti], ti);
					k = atomicPSeg(&occ[index_pnt], index_pnt);

					int flag = pb1[blockIdx.x*np+index_pnt];
					int dst_v0 = csc_v[index]+i;
					int dst_v = csc_e[dst_v0] - index_pnt * PSIZE; //dst
//if(dst_v < 0 || dst_v >= PSIZE) printf("e : %d\n", dst_v);
					if(flag >= 0) { //sparse

						flag += es[blockIdx.x];
//if(k+flag < 0 || k+flag >= upper_ne) printf("bound1 %d (%d %d %d %d) %d\n",k, pb1[0],pb1[1],pb1[2],pb1[3], upper_ne);
						sp1[k+flag] = index - blockIdx.x*PSIZE; //src
#ifdef E1
						sp1v[k+flag] = csc_ev[dst_v0]; //src
//sp1v[k+flag] = 100;
//printf("%d %d %d\n", k+flag, dst_v0, csc_ev[dst_v0]);
#endif
//if(index - blockIdx.x*PSIZE < 0 || index - blockIdx.x*PSIZE >= PSIZE) printf("err_occ %d\n", index);
//if(k+pb2[index_pnt*np*DFACTOR+blockIdx.x*DFACTOR+(threadIdx.x&(DFACTOR-1))] < 0 
//|| k+pb2[index_pnt*np*DFACTOR+blockIdx.x*DFACTOR+(threadIdx.x&(DFACTOR-1))] >= upper_ne)
//printf("bound2 %d %d\n", k+pb2[index_pnt*np*DFACTOR+blockIdx.x*DFACTOR+(threadIdx.x&(DFACTOR-1))], upper_ne); 
						sp2[k+pb2[index_pnt*np+blockIdx.x]] = dst_v; //dst

						if(((k+flag)&7) == 0) {
							mapper[(k+flag)>>3] = k+pb2[index_pnt*np+blockIdx.x];
						}
					} else { //dense
//printf("er\n");
						flag = -(flag+1);
// printf("%d %d %d\n", dcnt, flag, dindex[flag]+k);
						fx[dindex[flag]+k] = index - blockIdx.x*PSIZE; //src
						fy[dindex[flag]+k] = dst_v;//dst
#ifdef E1
						fz[dindex[flag]+k] = csc_ev[dst_v0];//dst
#endif
					}
                                }
                        }
                        index += 256;
                }
                __syncthreads();
                // step 2 (medium granularity)
                for(i=(threadIdx.x>>5);i<occ[SM_SIZE-2];i+=32) {
                        int lindex = buffer1[i] + blockIdx.x*PSIZE;
                        int dummy = csc_v[lindex+1] - csc_v[lindex];
                        int bf2 = dummy - (dummy&31);
                        int bf22 = bf2 - (bf2&255);
                        for(j=bf22+(threadIdx.x&31);j<bf2;j+=32) {
                                int index_pnt = csc_e[csc_v[lindex]+j] / PSIZE;
				//int ti = index_pnt * DFACTOR + (lindex & (DFACTOR-1));
				int k;
				k = atomicSeg(&occ[index_pnt], index_pnt);
				//k = atomicSeg(&occ[ti], ti);

				//int k = atomicAdd(&occ[DFACTOR*index_pnt+(threadIdx.x&(DFACTOR-1))],1);
				int flag = pb1[blockIdx.x*np+index_pnt];
				int dst_v0 = csc_v[lindex]+j;
				int dst_v = csc_e[dst_v0] - index_pnt * PSIZE; //dst
//if(dst_v > PSIZE) printf("e : %d\n", dst_v);
				if(flag >= 0) { //sparse

					flag += es[blockIdx.x];
					sp1[k+flag] = lindex - blockIdx.x*PSIZE; //src
#ifdef E1
					sp1v[k+flag] = csc_ev[dst_v0]; //src
//sp1v[k+flag] = 1000;
#endif

//if(lindex - blockIdx.x*PSIZE < 0 || lindex - blockIdx.x*PSIZE >= PSIZE) printf("err_occ %d\n", lindex);
					sp2[k+pb2[index_pnt*np+blockIdx.x]] = dst_v; //dst
					if(((k+flag)&7) == 0) {
						mapper[(k+flag)>>3] = k+pb2[index_pnt*np+blockIdx.x];
					}
				} else { //dense
//printf("er\n");
					flag = -(flag+1);
//if(threadIdx.x==0) printf("%d %d %d\n", dcnt, flag, dindex[flag]+k);
// printf("%d %d %d\n", dcnt, flag, dindex[flag]+k);
					fx[dindex[flag]+k] = lindex - blockIdx.x*PSIZE; //src
					fy[dindex[flag]+k] = dst_v;//dst
#ifdef E1
					fz[dindex[flag]+k] = csc_ev[dst_v0];//dst
#endif
				}
                        }
                }
                // step 3 (large granularity)
                for(i=(threadIdx.x>>8);i<occ[SM_SIZE-1];i+=4) {
                        int lindex = buffer3[i] + blockIdx.x*PSIZE;
			int h = ((threadIdx.x>>5)&7);
                        int bf2 = csc_v[lindex+1] - csc_v[lindex];
                        bf2 = bf2 - (bf2&255);
                        for(j=(threadIdx.x&255);j<bf2;j+=256) {
                                int index_pnt = csc_e[csc_v[lindex]+j] / PSIZE;
				//int ti = index_pnt * DFACTOR + (h & (DFACTOR-1));
				int k;
				k = atomicSeg(&occ[index_pnt], index_pnt);
				//k = atomicSeg(&occ[ti], ti);

				//int k = atomicAdd(&occ[DFACTOR*index_pnt+(threadIdx.x&(DFACTOR-1))],1);
				int flag = pb1[blockIdx.x*np+index_pnt];
				int dst_v0 = csc_v[lindex]+j;
				int dst_v = csc_e[dst_v0] - index_pnt * PSIZE; //dst
//if(dst_v > PSIZE) printf("e : %d\n", dst_v);
				if(flag >= 0) { //sparse
				
					flag += es[blockIdx.x];
					sp1[k+flag] = lindex - blockIdx.x*PSIZE; //src
#ifdef E1
					sp1v[k+flag] = csc_ev[dst_v0]; //src
//sp1v[k+flag] = 10000;
#endif

//if(lindex - blockIdx.x*PSIZE < 0 || lindex - blockIdx.x*PSIZE >= PSIZE) printf("err_occ %d\n", lindex);
					sp2[k+pb2[index_pnt*np+blockIdx.x]] = dst_v; //dst
					if(((k+flag)&7) == 0) {
						mapper[(k+flag)>>3] = k+pb2[index_pnt*np+blockIdx.x];
					}
				} else { //dense
//printf("er\n");
					flag = -(flag+1);
//printf("%d %d %d\n", dcnt, flag, dindex[flag]+k);
					fx[dindex[flag]+k] = lindex - blockIdx.x*PSIZE; //src
					fy[dindex[flag]+k] = dst_v;//dst
#ifdef E1
					fz[dindex[flag]+k] = csc_ev[dst_v0];//dst
#endif
				}
                        }
                }
                __syncthreads();
        }
}


/*
__global__ void
//__launch_bounds__(BSIZE, 2)
preprocessing_step22(int DFACTOR, int nv, int ne, int upper_ne, int np, int upper_np, int *csc_v, int *csc_e, const int * __restrict__ pb1, const int * __restrict__ pb2, int *es,
int dcnt, int *dindex, int *dx, int *dy, short *sp1, short *sp2, short *fx, short *fy, int *mapper
#ifdef E1
, E1T *csc_ev, E1T *sp1v, E1T *fz
#endif
)
{
	__shared__ int occ[SM_SIZE];
        __shared__ int buffer1[BG], buffer3[BG];
//        __shared__ int buffer_p[2];

//	__shared__ int spb1[SM_SIZE];
//	__shared__ int spb2[SM_SIZE];

	int i, j, index, index_size, bias;

	for(i=threadIdx.x;i<DFACTOR*np;i+=blockDim.x) {
		occ[i] = 0; 
//		spb1[i] = pb1[np*DFACTOR*blockIdx.x+i];
//		spb2[i] = pb2[np*DFACTOR*blockIdx.x+i];
	}

        for(index=blockIdx.x*PSIZE+(threadIdx.x>>2); index-blockIdx.x*PSIZE < PSIZE; ) {
                if(threadIdx.x < 2) {
                        occ[SM_SIZE-2+threadIdx.x] = 0;
                }
                __syncthreads();
                for(int lk=0;lk<4;lk++) {
                        if(index < nv) {
                                index_size = csc_v[index+1] - csc_v[index];
                                if(index_size >= 32) {
                                        bias = index_size - (index_size&31);
                                        if((threadIdx.x&3) == 0) {
                                                int p = atomicAggInc(&occ[SM_SIZE-2]);
                                                buffer1[p] = index;
                                        }
                                        if(index_size >= 256) {
                                                if((threadIdx.x&3) == 0) {
                                                        int p2 = atomicAggInc(&occ[SM_SIZE-1]);
                                                        buffer3[p2] = index;
                                                }
                                        }
                                } else {
                                        bias = 0;
                                }
                                // step 1 (small granularity)
                                for(i=bias+(threadIdx.x&3); i<index_size; i+=4) {
                                        int index_pnt = csc_e[csc_v[index]+i] / PSIZE;

//if(csc_v[index]+i < 0 || csc_v[index]+i >= ne) printf("err0 %d \n", csc_v[index]+i);
//if(csc_e[csc_v[index]+i] < 0 || csc_e[csc_v[index]+i] >= nv) printf("err %d \n", csc_e[csc_v[index]+i]);
//if(DFACTOR*index_pnt+(threadIdx.x&(DFACTOR-1)) < 0 || DFACTOR*index_pnt+(threadIdx.x&(DFACTOR-1)) >= SM_SIZE) printf("err2\n");
					int k = atomicAdd(&occ[DFACTOR*index_pnt+(threadIdx.x&(DFACTOR-1))],1);
					int flag = pb1[blockIdx.x*np*DFACTOR+index_pnt*DFACTOR+(threadIdx.x&(DFACTOR-1))];
					int dst_v0 = csc_v[index]+i;
					int dst_v = csc_e[dst_v0] - index_pnt * PSIZE; //dst
//if(dst_v < 0 || dst_v >= PSIZE) printf("e : %d\n", dst_v);
					if(flag >= 0) { //sparse

						flag += es[blockIdx.x];
//if(k+flag < 0 || k+flag >= upper_ne) printf("bound1 %d (%d %d %d %d) %d\n",k, pb1[0],pb1[1],pb1[2],pb1[3], upper_ne);
						sp1[k+flag] = index - blockIdx.x*PSIZE; //src
#ifdef E1
						sp1v[k+flag] = csc_ev[dst_v0]; //src
//sp1v[k+flag] = 100;
//printf("%d %d %d\n", k+flag, dst_v0, csc_ev[dst_v0]);
#endif
//if(index - blockIdx.x*PSIZE < 0 || index - blockIdx.x*PSIZE >= PSIZE) printf("err_occ %d\n", index);
//if(k+pb2[index_pnt*np*DFACTOR+blockIdx.x*DFACTOR+(threadIdx.x&(DFACTOR-1))] < 0 
//|| k+pb2[index_pnt*np*DFACTOR+blockIdx.x*DFACTOR+(threadIdx.x&(DFACTOR-1))] >= upper_ne)
//printf("bound2 %d %d\n", k+pb2[index_pnt*np*DFACTOR+blockIdx.x*DFACTOR+(threadIdx.x&(DFACTOR-1))], upper_ne); 
						sp2[k+pb2[index_pnt*np*DFACTOR+blockIdx.x*DFACTOR+(threadIdx.x&(DFACTOR-1))]] = dst_v; //dst

						if(((k+flag)&7) == 0) {
							mapper[(k+flag)>>3] = k+pb2[index_pnt*np*DFACTOR+blockIdx.x*DFACTOR+(threadIdx.x&(DFACTOR-1))];
						}
					}
                                }
                        }
                        index += 256;
                }
                __syncthreads();
                // step 2 (medium granularity)
                for(i=(threadIdx.x>>5);i<occ[SM_SIZE-2];i+=32) {
                        int lindex = buffer1[i];
                        int dummy = csc_v[lindex+1] - csc_v[lindex];
                        int bf2 = dummy - (dummy&31);
                        int bf22 = bf2 - (bf2&255);
                        for(j=bf22+(threadIdx.x&31);j<bf2;j+=32) {
                                int index_pnt = csc_e[csc_v[lindex]+j] / PSIZE;
				int k = atomicAdd(&occ[DFACTOR*index_pnt+(threadIdx.x&(DFACTOR-1))],1);
				int flag = pb1[blockIdx.x*np*DFACTOR+index_pnt*DFACTOR+(threadIdx.x&(DFACTOR-1))];
				int dst_v0 = csc_v[lindex]+j;
				int dst_v = csc_e[dst_v0] - index_pnt * PSIZE; //dst
//if(dst_v > PSIZE) printf("e : %d\n", dst_v);
				if(flag >= 0) { //sparse

					flag += es[blockIdx.x];
					sp1[k+flag] = lindex - blockIdx.x*PSIZE; //src
#ifdef E1
					sp1v[k+flag] = csc_ev[dst_v0]; //src
//sp1v[k+flag] = 1000;
#endif

//if(lindex - blockIdx.x*PSIZE < 0 || lindex - blockIdx.x*PSIZE >= PSIZE) printf("err_occ %d\n", lindex);
					sp2[k+pb2[index_pnt*np*DFACTOR+blockIdx.x*DFACTOR+(threadIdx.x&(DFACTOR-1))]] = dst_v; //dst
					if(((k+flag)&7) == 0) {
						mapper[(k+flag)>>3] = k+pb2[index_pnt*np*DFACTOR+blockIdx.x*DFACTOR+(threadIdx.x&(DFACTOR-1))];
					}
				}                         }
                }
                // step 3 (large granularity)
                for(i=(threadIdx.x>>8);i<occ[SM_SIZE-1];i+=4) {
                        int lindex = buffer3[i];
                        int bf2 = csc_v[lindex+1] - csc_v[lindex];
                        bf2 = bf2 - (bf2&255);
                        for(j=(threadIdx.x&255);j<bf2;j+=256) {
                                int index_pnt = csc_e[csc_v[lindex]+j] / PSIZE;
				int k = atomicAdd(&occ[DFACTOR*index_pnt+(threadIdx.x&(DFACTOR-1))],1);
				int flag = pb1[blockIdx.x*np*DFACTOR+index_pnt*DFACTOR+(threadIdx.x&(DFACTOR-1))];
				int dst_v0 = csc_v[lindex]+j;
				int dst_v = csc_e[dst_v0] - index_pnt * PSIZE; //dst
//if(dst_v > PSIZE) printf("e : %d\n", dst_v);
				if(flag >= 0) { //sparse
				
					flag += es[blockIdx.x];
					sp1[k+flag] = lindex - blockIdx.x*PSIZE; //src
#ifdef E1
					sp1v[k+flag] = csc_ev[dst_v0]; //src
//sp1v[k+flag] = 10000;
#endif

//if(lindex - blockIdx.x*PSIZE < 0 || lindex - blockIdx.x*PSIZE >= PSIZE) printf("err_occ %d\n", lindex);
					sp2[k+pb2[index_pnt*np*DFACTOR+blockIdx.x*DFACTOR+(threadIdx.x&(DFACTOR-1))]] = dst_v; //dst
					if(((k+flag)&7) == 0) {
						mapper[(k+flag)>>3] = k+pb2[index_pnt*np*DFACTOR+blockIdx.x*DFACTOR+(threadIdx.x&(DFACTOR-1))];
					}
				} 
                        }
                }
                __syncthreads();
        }
}
*/



__global__ void preprocessing_step21(int DFACTOR, int np, int *pb2, int *dx, int *dy, short *sp2, int *dmap)
{
	int base_addr = pb2[dy[blockIdx.x]*DFACTOR*np+dx[blockIdx.x]*DFACTOR];	
	short i;
	if(threadIdx.x == 0)
		dmap[blockIdx.x] = base_addr;
	for(i=threadIdx.x;i<PSIZE;i+=blockDim.x) {
		sp2[base_addr+i] = i;
	}
}

__global__ void preprocessing_step3(int DFACTOR, int LOG_DFACTOR, int *dindex, short *fx, short *fy, int *docc)
{
	__shared__ int socc[PSIZE];

	int i;
//	int height;
	for(i=threadIdx.x;i<PSIZE;i+=blockDim.x) {
		socc[i] = 0;
	}
	__syncthreads();

//	if(threadIdx.x == 0) printf("error : %d %d\n", dindex[blockIdx.x<<LOG_DFACTOR], dindex[(blockIdx.x<<LOG_DFACTOR)+DFACTOR]);
//	height = ((dindex[(blockIdx.x<<LOG_DFACTOR)+DFACTOR] - dindex[blockIdx.x<<LOG_DFACTOR])>>LOG_BSIZE);
	for(i=dindex[blockIdx.x<<LOG_DFACTOR]+threadIdx.x;i<dindex[(blockIdx.x<<LOG_DFACTOR)+DFACTOR];i+=blockDim.x) {
//if(i < 0) printf("errrr\n");
//if(fy[i] > PSIZE) printf("err : %d\n", fy[i]);
		if(fy[i] < SUNUSED) {
//if(fy[i] >= PSIZE || fy[i] < 0) printf("type1 err\n");
			atomicAdd(&socc[fy[i]], 1);
		}
	}
	__syncthreads();

	for(i=threadIdx.x;i<PSIZE;i+=blockDim.x) {
		if(socc[i] > 0) {
			socc[i]++;
		} 
//		if((threadIdx.x&1) == 0) socc[i] ++;
//		socc[i]++;
	}
	__syncthreads();
	for(i=threadIdx.x;i<PSIZE;i+=blockDim.x) {
		docc[blockIdx.x*PSIZE + i] = socc[i];
//		if(docc[blockIdx.x*PSIZE*2 + i] > PSIZE) printf("ee\n");
	}
}

/*
__global__ void preprocessing_step31(int *dindex, int *docc)
{
	int i, height;
	height = ((dindex[(blockIdx.x<<LOG_DFACTOR)+DFACTOR] - dindex[blockIdx.x<<LOG_DFACTOR])>>LOG_BSIZE);

	for(i=threadIdx.x;i<PSIZE*2;i+=blockDim.x) {
//		docc[blockIdx.x*PSIZE*2 + i] += (docc[blockIdx.x*PSIZE*2 + i]/height);
		int t = docc[blockIdx.x*PSIZE*2+i];
		for(int t1 = t/height; t1 > 0; t1 /= height)
			t += t1;
		docc[blockIdx.x*PSIZE*2 + i ] = t;

	}
}*/

__global__ void
__launch_bounds__(BSIZE, 1) 
preprocessing_step4(int DFACTOR, int LOG_DFACTOR, int *dindex, short *fx, short *fy, const int * __restrict__ docc, short *dp1
#ifdef E1
, E1T *dp1v, E1T *fz
#endif
)
{
	__shared__ int socc[PSIZE];

	int i;
	int height, base_addr=dindex[blockIdx.x<<LOG_DFACTOR];
	for(i=threadIdx.x;i<PSIZE;i+=blockDim.x) {
		socc[i] = 0;
	}
	__syncthreads();

	height = ((dindex[(blockIdx.x<<LOG_DFACTOR)+DFACTOR] - base_addr)>>LOG_BSIZE)-1;
//if((dindex[(blockIdx.x<<LOG_DFACTOR)+DFACTOR] - base_addr) % BSIZE != 0) printf("not-divide\n");

	for(i=base_addr+threadIdx.x;i<dindex[(blockIdx.x<<LOG_DFACTOR)+DFACTOR];i+=blockDim.x) {
		if(fy[i] < SUNUSED) {
//if(fy[i] < 0 || fy[i] > PSIZE) printf("err : %d\n", fy[i]);
			int k, k2, kq, kr;
			while(1) {
				k = atomicAdd(&socc[fy[i]], 1);
				k2 = docc[blockIdx.x*PSIZE+fy[i]]+k;
//if(2*fy[i]+bias < PSIZE-1 && k2 > docc[blockIdx.x*PSIZE*2+2*fy[i]+bias+1]) printf("e00 : %d %d %d %d %d\n", height, fy[i], k2,docc[blockIdx.x*PSIZE*2+fy[i]*2+1+bias], docc[blockIdx.x*PSIZE*2+2*fy[i]+bias]); 
//if(2*fy[i]+bias == PSIZE-1 && k2 >= dindex[(blockIdx.x+1)<<LOG_DFACTOR] - dindex[blockIdx.x<<LOG_DFACTOR])
//printf("e01 : %d %d\n", k2, dindex[(blockIdx.x+1)<<LOG_DFACTOR] - dindex[blockIdx.x<<LOG_DFACTOR]);
				kq = k2/height;
				kr = k2 - kq*height;
//if(kq >= BSIZE || kr >= height) printf("e0 : (%d %d) %d %d\n", k2, height, kq, kr);
				if(k == 0) {
//if(BSIZE+base_addr+kr*BSIZE+kq >= dindex[(blockIdx.x+1)*DFACTOR]) printf("err (%d %d) %d %d\n", blockIdx.x, threadIdx.x, BSIZE+base_addr+kr*BSIZE+kq, dindex[(blockIdx.x+1)*DFACTOR]); 
//				if(k == 0 && ((bias == 0) ||
//				(bias == 1 && (docc[blockIdx.x*PSIZE*4+fy[i]*4+1] == docc[blockIdx.x*PSIZE*4+fy[i]*4])) ||
//				(bias == 2 && (docc[blockIdx.x*PSIZE*4+fy[i]*4+2] == docc[blockIdx.x*PSIZE*4+fy[i]*4])) ||
//				(bias == 3 && (docc[blockIdx.x*PSIZE*4+fy[i]*4+3] == docc[blockIdx.x*PSIZE*4+fy[i]*4]))
//				)) {
					dp1[base_addr+BSIZE+kr*BSIZE+kq] = -(fy[i]+1);
				} else {
					if(kr == 0) {
						dp1[base_addr+kq] = -(fy[i]+1);
					}
					dp1[base_addr+BSIZE+kr*BSIZE+kq] = fx[i];
#ifdef E1
					dp1v[base_addr+BSIZE+kr*BSIZE+kq] = fz[i];
#endif
					break;
				}
			}
		}
	}

}

__global__ void preprocessing_step41(int s3g, int LOG_DFACTOR, int *dcindex, int *dccindex)
{
	int index = blockIdx.x*blockDim.x+threadIdx.x;
	if(index <= s3g) {
		dccindex[index] = dcindex[index<<LOG_DFACTOR];
	}
}


void generate_MultiGraph(struct HYB *m, struct csc_package *inp)
{
	hipError_t cuda_stat;

	float add_factor = 1.4;

        int nv = inp->nv, ne = inp->ne, np = inp->np;
        int upper_nv = inp->upper_nv, upper_np = inp->upper_np;
        int DFACTOR = inp->DFACTOR, LOG_DFACTOR = inp->LOG_DFACTOR;
        int *csc_v = inp->csc_v, *csc_e = inp->csc_e;
#ifdef E1
	int *csc_ev = inp->csc_ev;
#endif

	int upper_ne = ne + 8*np*np;
	int start_point;
	int *_csc_v, *_csc_e;
	int dcnt;

	short *_csc_occ;
	int *_group_occ;
	int *_es, *_ces, *_pb1, *_pb2, *_cpb2;
	int *_cey;
	int *_dcnt, *_dx, *_dy, *_dccindex, *_dcindex, *_dindex;
	short *_fx, *_fy, *_sp1, *_sp2;
	int *_mapper;
	int *_docc, *_dcocc;	
	short *_dp1;
	int *_gr, *_cgr;
	int *_itable, *_csc_size; //index, loc(size)
	int *_ncsc_v, *_ncsc_e;
#ifdef E1
	E1T *_csc_ev, *_ncsc_ev, *_sp1v, *_dp1v, *_fz;
#endif
	int *_dmap;

	int *tt; short *tt2;//will be remomved
	int *ncsc_v = (int *)malloc(sizeof(int)*(nv+1));
	int *ncsc_e = (int *)malloc(sizeof(int)*ne);
#ifdef E1
	E1T *ncsc_ev = (E1T *)malloc(sizeof(E1T)*ne);
#endif

	int *_buff;
	hipMalloc((void **) &_buff, sizeof(int)*32*np);

	cuda_stat = hipMalloc((void **) &_csc_v, sizeof(int)*(nv+1));
	if(cuda_stat != hipSuccess) { fprintf(stderr, "_csc_v %s\n", hipGetErrorString(cuda_stat)); }
	cuda_stat = hipMalloc((void **) &_csc_e, sizeof(int)*ne);
	if(cuda_stat != hipSuccess) { fprintf(stderr, "_csc_e %s\n", hipGetErrorString(cuda_stat)); }
	cuda_stat = hipMalloc((void **) &_csc_occ, sizeof(short)*ne);
	if(cuda_stat != hipSuccess) { fprintf(stderr, "_csc_occ %s\n", hipGetErrorString(cuda_stat)); }
	cuda_stat = hipMalloc((void **) &_group_occ, sizeof(int)*GROUP_NUM);
	if(cuda_stat != hipSuccess) { fprintf(stderr, "_group_occ %s\n", hipGetErrorString(cuda_stat)); }
	hipMemcpy(_csc_v, csc_v, sizeof(int)*(nv+1), hipMemcpyHostToDevice);
	hipMemcpy(_csc_e, csc_e, sizeof(int)*ne, hipMemcpyHostToDevice);
	hipMemset(_csc_occ, 0, sizeof(short)*ne);
	hipMemset(_group_occ, 0, sizeof(int)*GROUP_NUM);
	



//printf("gr : %d\n", upper_nv);
	cuda_stat = hipMalloc((void **) &_gr, sizeof(int)*upper_nv*GROUP_NUM);
	if(cuda_stat != hipSuccess) { fprintf(stderr, "_gr %s\n", hipGetErrorString(cuda_stat)); }
	cuda_stat = hipMalloc((void **) &_cgr, sizeof(int)*upper_nv*GROUP_NUM);
	if(cuda_stat != hipSuccess) { fprintf(stderr, "_cgr %s\n", hipGetErrorString(cuda_stat)); }
	hipMemset(_gr, 0, sizeof(int)*upper_nv*GROUP_NUM);
	hipMemset(_cgr, 0, sizeof(int)*upper_nv*GROUP_NUM);


	cuda_stat = hipMalloc((void **) &_itable, sizeof(int)*(upper_nv+1));
	if(cuda_stat != hipSuccess) { fprintf(stderr, "_itable %s\n", hipGetErrorString(cuda_stat)); }
	int *it = (int *)malloc(sizeof(int)*(upper_nv+1));
	for(int i=0;i<=upper_nv;i++) 
		it[i] = i;
	hipMemcpy(_itable, it, sizeof(int)*(upper_nv+1), hipMemcpyHostToDevice);

	cuda_stat = hipMalloc((void **) &_csc_size, sizeof(int)*(nv+1));
	if(cuda_stat != hipSuccess) { fprintf(stderr, "_csc_size %s\n", hipGetErrorString(cuda_stat)); }
//	hipMemset(_itable, 0, sizeof(int)*(upper_nv+1));
	hipMemset(_csc_size, 0, sizeof(int)*(nv+1));
	free(it);

#ifdef E1
	cuda_stat = hipMalloc((void **) &_csc_ev, sizeof(E1T)*ne);
	if(cuda_stat != hipSuccess) { fprintf(stderr, "_csc_ev %s\n", hipGetErrorString(cuda_stat)); }
	hipMemcpy(_csc_ev, csc_ev, sizeof(E1T)*ne, hipMemcpyHostToDevice); // detailed value may be needed
#endif



        CUDPPHandle theCudpp;
        cudppCreate(&theCudpp);
        CUDPPConfiguration config;
        config.op = CUDPP_ADD;
        config.datatype = CUDPP_INT;
        config.algorithm = CUDPP_SCAN;
        config.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_EXCLUSIVE;
        CUDPPHandle scanplan1 = 0;
        CUDPPResult res = cudppPlan(theCudpp, &scanplan1, config, DFACTOR*np*np+1, 1, 0);

        CUDPPHandle theCudpp2;
        cudppCreate(&theCudpp2);
        CUDPPConfiguration config2;
        config2.op = CUDPP_ADD;
        config2.datatype = CUDPP_INT;
        config2.algorithm = CUDPP_SCAN;
        config2.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_EXCLUSIVE;
        CUDPPHandle scanplan2 = 0;
        CUDPPResult res2 = cudppPlan(theCudpp2, &scanplan2, config2, DFACTOR*np*np+1, 1, 0);

/*
        CUDPPHandle theCudpp3;
        cudppCreate(&theCudpp3);
        CUDPPConfiguration config3;
        config3.op = CUDPP_ADD;
        config3.datatype = CUDPP_INT;
        config3.algorithm = CUDPP_SCAN;
        config3.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_INCLUSIVE;
        CUDPPHandle scanplan3 = 0;
        CUDPPResult res3 = cudppPlan(theCudpp3, &scanplan3, config3, upper_nv*GROUP_NUM, GROUP_NUM, upper_nv);
*/

	CUDPPHandle theCudpp33;
        cudppCreate(&theCudpp33);
        CUDPPConfiguration config33;
        config33.op = CUDPP_ADD;
        config33.datatype = CUDPP_INT;
        config33.algorithm = CUDPP_SCAN;
        config33.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_EXCLUSIVE;
        CUDPPHandle scanplan33 = 0;
        CUDPPResult res33 = cudppPlan(theCudpp33, &scanplan33, config33, upper_nv, 1, 0);



        CUDPPHandle theCudpp4;
        cudppCreate(&theCudpp4);
        CUDPPConfiguration config4;
        config4.op = CUDPP_ADD;
        config4.datatype = CUDPP_INT;
        config4.algorithm = CUDPP_SCAN;
        config4.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_EXCLUSIVE;
        CUDPPHandle scanplan4 = 0;
        CUDPPResult res4 = cudppPlan(theCudpp4, &scanplan4, config4, nv+1, 1, 0);

	CUDPPHandle theCudpp5;
        cudppCreate(&theCudpp5);
        CUDPPConfiguration config5;
        config5.op = CUDPP_ADD;
        config5.datatype = CUDPP_INT;
        config5.algorithm = CUDPP_SCAN;
        config5.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_EXCLUSIVE;
        CUDPPHandle scanplan5 = 0;

       
/* // 8*K
        CUDPPHandle theCudpp4;
        cudppCreate(&theCudpp4);
        CUDPPConfiguration config4;
        config4.op = CUDPP_ADD;
        config4.datatype = CUDPP_INT;
        config4.algorithm = CUDPP_SCAN;
        config4.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_EXCLUSIVE;
        CUDPPHandle scanplan4 = 0;
	int SS=24;
        CUDPPResult res4 = cudppPlan(theCudpp4, &scanplan4, config4, SS*SS, SS, SS);
	int *ain, *aout;
	tt = (int *)malloc(sizeof(int)*SS*SS);
	for(int i=0;i<SS*SS;i++) tt[i]=1;
	cuda_stat = hipMalloc((void **) &ain, sizeof(int)*SS*SS);
	cuda_stat = hipMalloc((void **) &aout,sizeof(int)*SS*SS);
	hipMemcpy(ain, tt, sizeof(int)*SS*SS, hipMemcpyHostToDevice);
	res4 = cudppMultiScan(scanplan4, aout, ain, SS, SS);
	hipMemcpy(tt, aout, sizeof(int)*SS*SS, hipMemcpyDeviceToHost);
	for(int i=0;i<SS*SS;i++) {
		fprintf(stdout, "%d ", tt[i]);
	} fprintf(stdout, "\n");
	exit(0);
*/

	hipDeviceProp_t devProp;
	hipGetDeviceProperties(&devProp, 0);
	int mp = devProp.multiProcessorCount;
	m->mp = mp;
//	int s00g=ne/(128*SFACTOR), s00b=128;
	int s00g=mp*16, s00b=128;
	int s01g=mp*2, s01b=BSIZE; // strict split
//	int s02g=CEIL(nv, 128), s02b=128;
	int s02g=mp*16, s02b=128;
//	int s03g=CEIL(nv, 64), s03b=256, s03n=CEIL(nv,64)*64;
	int s03g=(nv+63)/64, s03b=256, s03n=CEIL(nv,64)*64;
	////int s03g=(nv+63)/64, s03b=128, s03n=CEIL(nv,64)*64;
	int s1g=np, s1b=BSIZE;
	int s11g=CEIL(np+1+127,128), s11b=128;
	int s2size = DFACTOR*np*np+1;
	int s3g, s3b=BSIZE;
	int totalsize_dense;
	int group_occ[GROUP_NUM];
	double trans_begin, trans_end;

//fprintf(stdout, "%d %d\n", nv, ne);

	double s_time = rtclock(); hipDeviceSynchronize();

    if(inp->tra) {

	preprocessing_step00<<<s00g, s00b>>>(nv, _csc_v, _csc_occ);

//SHORT_PRINT(_csc_occ,500);

	preprocessing_step01<<<s01g ,s01b>>>(nv, upper_nv, _csc_occ, _group_occ, _gr);


	res33 = cudppScan(scanplan33, &_cgr[0], &_gr[0], upper_nv);
	res33 = cudppScan(scanplan33, &_cgr[upper_nv], &_gr[upper_nv], upper_nv);
	res33 = cudppScan(scanplan33, &_cgr[upper_nv*2], &_gr[upper_nv*2], upper_nv);
	res33 = cudppScan(scanplan33, &_cgr[upper_nv*3], &_gr[upper_nv*3], upper_nv);
	res33 = cudppScan(scanplan33, &_cgr[upper_nv*4], &_gr[upper_nv*4], upper_nv);
	res33 = cudppScan(scanplan33, &_cgr[upper_nv*5], &_gr[upper_nv*5], upper_nv);


//	res3 = cudppMultiScan(scanplan3, _cgr, _gr, upper_nv, GROUP_NUM);
	hipMemcpyAsync(group_occ, _group_occ, sizeof(int)*GROUP_NUM, hipMemcpyDeviceToHost);

//INT_PRINT(&_gr[upper_nv*5], 10000);
//INT_PRINT(&_cgr[upper_nv*5], 10000);
//return;

//INT_PRINT(_group_occ,6);


/*
INT_PRINT(&_gr[nv-100], 100);
INT_PRINT(&_gr[upper_nv+nv-100], 100);
INT_PRINT(&_gr[2*upper_nv+nv-100], 100);
INT_PRINT(&_gr[3*upper_nv+nv-100], 100);
INT_PRINT(&_gr[4*upper_nv+nv-100], 100);
INT_PRINT(&_gr[5*upper_nv+nv-100], 100);

INT_PRINT(&_cgr[nv-100], 100);
INT_PRINT(&_cgr[upper_nv]+nv-100, 100);
INT_PRINT(&_cgr[2*upper_nv+nv-100], 100);
INT_PRINT(&_cgr[3*upper_nv+nv-100], 100);
INT_PRINT(&_cgr[4*upper_nv+nv-100], 100);
INT_PRINT(&_cgr[5*upper_nv+nv-100], 100);
*/
//return;

//INT_PRINT(&_cgr[5*upper_nv], upper_nv);

	hipFree(_gr);
//INT_PRINT(_group_occ,6);
#ifdef E1_NO
	cuda_stat = hipMalloc((void **) &_ncsc_ev, sizeof(E1T)*ne);
	if(cuda_stat != hipSuccess) { fprintf(stderr, "_ncsc_ev %s\n", hipGetErrorString(cuda_stat)); }
	hipMemset(_ncsc_ev, UIN, sizeof(E1T)*ne); // detailed value may be needed
#endif

        int cpu1 = group_occ[0];
        int cpu2 = cpu1+group_occ[1];
        int cpu3 = cpu2+group_occ[2];
        int cpu4 = cpu3+group_occ[3];
        int cpu5 = cpu4+group_occ[4];
        int cpu_threshold = cpu5;


	// _itable : for vertices reordering
	preprocessing_step02<<<s02g, s02b>>>(nv, upper_nv, _csc_v, _cgr, _itable, _csc_size, _csc_occ,
	cpu1, cpu2, cpu3, cpu4, cpu5);

	hipFree(_cgr);
	hipFree(_csc_occ);

//INT_PRINT(_group_occ,6);


	cuda_stat = hipMalloc((void **) &_ncsc_v, sizeof(int)*(nv+1));
	if(cuda_stat != hipSuccess) { fprintf(stderr, "_ncsc_v %s\n", hipGetErrorString(cuda_stat)); }
	int *_tmp_e;
        cuda_stat = hipMalloc((void **) &_tmp_e, sizeof(int)*ne);
        if(cuda_stat != hipSuccess) { fprintf(stderr, "_tmp_e %s\n", hipGetErrorString(cuda_stat)); }
	cuda_stat = hipMalloc((void **) &_ncsc_e, sizeof(int)*ne);
	if(cuda_stat != hipSuccess) { fprintf(stderr, "_ncsc_e %s\n", hipGetErrorString(cuda_stat)); }
#ifdef E1
	cuda_stat = hipMalloc((void **) &_ncsc_ev, sizeof(E1T)*ne);
	if(cuda_stat != hipSuccess) { fprintf(stderr, "_ncsc_ev %s\n", hipGetErrorString(cuda_stat)); }
#endif
//	hipMemset(_ncsc_v, 0, sizeof(int)*(nv+1));
//	hipMemset(_ncsc_e, 0, sizeof(int)*ne);


//INT_PRINT(_itable, 10000);

//INT_PRINT(_itable, 50);
//INT_PRINT(_csc_size, 50);
//return;

	res4 = cudppScan(scanplan4, _ncsc_v, _csc_size, 1+nv);

//INT_PRINT(&_csc_size[10-10],10000);

//	preprocessing_step03<<<s03g, s03b>>>(nv, s03n, _itable, _csc_v, _csc_e, _ncsc_v, _tmp_e, _ncsc_e
	preprocessing_step03r<<<s03g, s03b>>>(nv, s03n, _itable, _csc_v, _csc_e, _ncsc_v, _ncsc_e
//	,cpu1, cpu2, cpu3, cpu4, cpu5
#ifdef E1
	,_csc_ev, _ncsc_ev
#endif
	);

	hipFree(_csc_v); hipFree(_csc_e); hipFree(_tmp_e);
#ifdef E1
	hipFree(_csc_ev);
#endif

//INT_PRINT(_ncsc_e,1000);
//exit(0);	

    } else {
	hipFree(_gr);
	hipFree(_cgr);
	hipFree(_csc_occ);
	_ncsc_v = _csc_v; _ncsc_e = _csc_e;
#ifdef E1
	_ncsc_ev = _csc_ev;
#endif
    }


	cuda_stat = hipMalloc((void **) &_es, sizeof(int)*(np+1));
	if(cuda_stat != hipSuccess) { fprintf(stderr, "_es %s\n", hipGetErrorString(cuda_stat)); }
	cuda_stat = hipMalloc((void **) &_ces, sizeof(int)*(np+1));
	if(cuda_stat != hipSuccess) { fprintf(stderr, "_ces %s\n", hipGetErrorString(cuda_stat)); }
	cuda_stat = hipMalloc((void **) &_cey, sizeof(int)*(np+1));
	if(cuda_stat != hipSuccess) { fprintf(stderr, "_cey %s\n", hipGetErrorString(cuda_stat)); }
	cuda_stat = hipMalloc((void **) &_pb1, sizeof(int)*np*np*DFACTOR);
	if(cuda_stat != hipSuccess) { fprintf(stderr, "_pb1 %s\n", hipGetErrorString(cuda_stat)); }
	cuda_stat = hipMalloc((void **) &_pb2, sizeof(int)*(np*np*DFACTOR+1));
	if(cuda_stat != hipSuccess) { fprintf(stderr, "_pb2 %s\n", hipGetErrorString(cuda_stat)); }
	cuda_stat = hipMalloc((void **) &_cpb2, sizeof(int)*(np*np*DFACTOR+1));
	if(cuda_stat != hipSuccess) { fprintf(stderr, "_cpb2 %s\n", hipGetErrorString(cuda_stat)); }
	hipMemset(_es, 0, sizeof(int)*(np+1));
	hipMemset(_ces, 0, sizeof(int)*(np+1));
	hipMemset(_cey, 0, sizeof(int)*(np+1));
	hipMemset(_pb1, 0, sizeof(int)*np*np*DFACTOR);
	hipMemset(_pb2, 0, sizeof(int)*(np*np*DFACTOR+1));
	hipMemset(_cpb2, 0, sizeof(int)*(np*np*DFACTOR+1));


	cuda_stat = hipMalloc((void **) &_dcnt, sizeof(int));
	if(cuda_stat != hipSuccess) { fprintf(stderr, "_dcnt %s\n", hipGetErrorString(cuda_stat)); }
	cuda_stat = hipMalloc((void **) &_dx, sizeof(int)*np*np);
	if(cuda_stat != hipSuccess) { fprintf(stderr, "_dx %s\n", hipGetErrorString(cuda_stat)); }
	cuda_stat = hipMalloc((void **) &_dy, sizeof(int)*np*np);
	if(cuda_stat != hipSuccess) { fprintf(stderr, "_dy %s\n", hipGetErrorString(cuda_stat)); }
	cuda_stat = hipMalloc((void **) &_dccindex, sizeof(int)*(np*np+1));
	if(cuda_stat != hipSuccess) { fprintf(stderr, "_dccindex %s\n", hipGetErrorString(cuda_stat)); }
	cuda_stat = hipMalloc((void **) &_dcindex, sizeof(int)*(np*np*DFACTOR+1));
	if(cuda_stat != hipSuccess) { fprintf(stderr, "_dcindex %s\n", hipGetErrorString(cuda_stat)); }
	cuda_stat = hipMalloc((void **) &_dindex, sizeof(int)*(np*np*DFACTOR+1));
	hipMemset(_dcnt, 0, sizeof(int));
	hipMemset(_dx, 0, sizeof(int)*np*np);
	hipMemset(_dy, 0, sizeof(int)*np*np);
	hipMemset(_dccindex, 0, sizeof(int)*(np*np+1)); //cummulative index
	hipMemset(_dcindex, 0, sizeof(int)*(np*np*DFACTOR+1)); //cummulative index
	hipMemset(_dindex, 0, sizeof(int)*(np*np*DFACTOR+1));



//fprintf(stderr, "ok2\n");

	if(nv <= 12000000) {
		preprocessing_step1<<<s1g, s1b>>>(DFACTOR, LOG_DFACTOR, nv, ne, np, upper_np, _ncsc_v, _ncsc_e, _es, _pb1, _pb2,
		_dcnt, _dindex, _dx, _dy);
	} else if(nv <= 17000000) {
		preprocessing_step1medium<<<s1g, s1b>>>(DFACTOR, LOG_DFACTOR, nv, ne, np, upper_np, _ncsc_v, _ncsc_e, _es, _pb1, _pb2,
		_dcnt, _dindex, _dx, _dy);
	} else {
		if(PSIZE <= 4096) {
			preprocessing_step1verybig<<<s1g, s1b>>>(_buff, DFACTOR, LOG_DFACTOR, nv, ne, np, upper_np, _ncsc_v, _ncsc_e, _es, _pb1, _pb2,
			_dcnt, _dindex, _dx, _dy);
		} else {
			preprocessing_step1big<<<s1g, s1b>>>(DFACTOR, LOG_DFACTOR, nv, ne, np, upper_np, _ncsc_v, _ncsc_e, _es, _pb1, _pb2,
			_dcnt, _dindex, _dx, _dy);
		}
	}

	DFACTOR = 1; LOG_DFACTOR=0;

//INT_PRINT(_pb1,np*np*DFACTOR);
//INT_PRINT(_pb1,50);
//INT_PRINT(_pb2,50);
//return;



//fprintf(stderr, "ok21\n");

	hipMemcpyAsync(&dcnt, _dcnt, sizeof(int), hipMemcpyDeviceToHost);	

//fprintf(stderr, "small1\n");
	res = cudppScan(scanplan1, _ces, _es, np+1);

//fprintf(stderr, "small2\n");
	res = cudppScan(scanplan1, _cpb2, _pb2 , s2size); // need to be added


//fprintf(stderr, "small3\n");
	res2 = cudppScan(scanplan2, _dcindex, _dindex, 1+dcnt);

//fprintf(stderr, "small4\n");

	hipMemcpyAsync(&totalsize_dense, &_dcindex[dcnt], sizeof(int), hipMemcpyDeviceToHost); 
//INT_PRINT(_dcindex, 100);
//totalsize_dense*=2;

	hipFree(_dindex);hipFree(_es); hipFree(_pb2); 


//fprintf(stderr, "ok22\n");

	cuda_stat = hipMalloc((void **) &_docc, (dcnt>>LOG_DFACTOR)*sizeof(int)*PSIZE*2);
	if(cuda_stat != hipSuccess) { fprintf(stderr, "_docc %s\n", hipGetErrorString(cuda_stat)); }
	cuda_stat = hipMalloc((void **) &_dcocc, (dcnt>>LOG_DFACTOR)*sizeof(int)*PSIZE*2);
	if(cuda_stat != hipSuccess) { fprintf(stderr, "_dcocc %s\n", hipGetErrorString(cuda_stat)); }

	preprocessing_step11<<<s11g, s11b>>>(np, DFACTOR, _cpb2, _cey);
//hipMemset(_docc, 0, dcnt*sizeof(int)*PSIZE);
//INT_PRINT(_ces, 100);
//INT_PRINT(_pb1, 100);
//INT_PRINT(_pb2, 100);

//INT_PRINT(_ncsc_v,100);
//INT_PRINT(_ncsc_e,100);
	int ssize;
	hipMemcpyAsync(&ssize, &_cey[np], sizeof(int), hipMemcpyDeviceToHost);

//	ssize *=2;
//fprintf(stderr, "ssize : %d\n", ssize);

//fprintf(stderr, "ok3\n");

	cuda_stat = hipMalloc((void **) &_fx, sizeof(short)*totalsize_dense);
	if(cuda_stat != hipSuccess) { fprintf(stderr, "_fx %s\n", hipGetErrorString(cuda_stat)); }
	cuda_stat = hipMalloc((void **) &_fy, sizeof(short)*totalsize_dense);
	if(cuda_stat != hipSuccess) { fprintf(stderr, "_fy %s\n", hipGetErrorString(cuda_stat)); }
	hipMemset(_fx, UIN, sizeof(short)*totalsize_dense);
	hipMemset(_fy, UIN, sizeof(short)*totalsize_dense);

#ifdef E1
	cuda_stat = hipMalloc((void **) &_fz, sizeof(E1T)*totalsize_dense);
	if(cuda_stat != hipSuccess) { fprintf(stderr, "_fz %s\n", hipGetErrorString(cuda_stat)); }
#endif


	cuda_stat = hipMalloc((void **) &_sp1, sizeof(short)*ssize);
	if(cuda_stat != hipSuccess) { fprintf(stderr, "_sp1 %s\n", hipGetErrorString(cuda_stat)); }
	hipMemset(_sp1, UIN, sizeof(short)*ssize); // detailed value may be needed

	cuda_stat = hipMalloc((void **) &_sp2, sizeof(short)*ssize);
	if(cuda_stat != hipSuccess) { fprintf(stderr, "_sp2 %s\n", hipGetErrorString(cuda_stat)); }
	cuda_stat = hipMalloc((void **) &_mapper, sizeof(int)*CEIL(ssize,8));
	if(cuda_stat != hipSuccess) { fprintf(stderr, "_mapper %s\n", hipGetErrorString(cuda_stat)); }
	hipMemset(_sp2, UIN, sizeof(short)*ssize);
	hipMemset(_mapper, 0, sizeof(int)*CEIL(ssize,8));
#ifdef E1
	cuda_stat = hipMalloc((void **) &_sp1v, sizeof(E1T)*ssize);
	if(cuda_stat != hipSuccess) { fprintf(stderr, "_sp1v %s\n", hipGetErrorString(cuda_stat)); }
	hipMemset(_sp1v, UIN, sizeof(E1T)*ssize); // detailed value may be needed
#endif


//fprintf(stderr, "ok4\n");

    if(nv <= 12000000) {
	preprocessing_step2<<<s1g, s1b>>>(DFACTOR, nv, ne, upper_ne, np, upper_np, _ncsc_v, _ncsc_e, _pb1, _cpb2, _ces,
	dcnt, _dcindex, _dx, _dy, _sp1, _sp2, _fx, _fy, _mapper
#ifdef E1
	, _ncsc_ev, _sp1v, _fz
#endif
	);
    } else if(nv <= 17000000) {
	preprocessing_step2medium<<<s1g, s1b>>>(DFACTOR, nv, ne, upper_ne, np, upper_np, _ncsc_v, _ncsc_e, _pb1, _cpb2, _ces,
	dcnt, _dcindex, _dx, _dy, _sp1, _sp2, _fx, _fy, _mapper
#ifdef E1
	, _ncsc_ev, _sp1v, _fz
#endif
	);
    } else {
	if(PSIZE <= 4096) {
		preprocessing_step2verybig<<<s1g, s1b>>>(DFACTOR, nv, ne, upper_ne, np, upper_np, _ncsc_v, _ncsc_e, _pb1, _cpb2, _ces,
		dcnt, _dcindex, _dx, _dy, _sp1, _sp2, _fx, _fy, _mapper
#ifdef E1
		, _ncsc_ev, _sp1v, _fz
#endif
		);
	} else {
		preprocessing_step2big<<<s1g, s1b>>>(DFACTOR, nv, ne, upper_ne, np, upper_np, _ncsc_v, _ncsc_e, _pb1, _cpb2, _ces,
		dcnt, _dcindex, _dx, _dy, _sp1, _sp2, _fx, _fy, _mapper
#ifdef E1
		, _ncsc_ev, _sp1v, _fz
#endif
		);
	}

    }
	hipDeviceSynchronize();
	trans_begin = rtclock(); // for very large graph
	hipMemcpy(ncsc_v, _ncsc_v, sizeof(int)*(nv+1), hipMemcpyDeviceToHost);
	hipMemcpy(ncsc_e, _ncsc_e, sizeof(int)*ne, hipMemcpyDeviceToHost);
#ifdef E1
	hipMemcpy(ncsc_ev, _ncsc_ev, sizeof(E1T)*ne, hipMemcpyDeviceToHost);
#endif
	hipFree(_ncsc_v); hipFree(_ncsc_e);
#ifdef E1
hipFree(_ncsc_ev);
#endif
	trans_end = rtclock();	
	s3g = (dcnt >> LOG_DFACTOR);

	
//fprintf(stderr, "succbefore\n");

	if(dcnt > 0) {
		cuda_stat = hipMalloc((void **) &_dmap, sizeof(int)*(s3g));
	if(cuda_stat != hipSuccess) { fprintf(stderr, "_dmap %s\n", hipGetErrorString(cuda_stat)); }

	cuda_stat = hipMalloc((void **) &_dp1, totalsize_dense*sizeof(short));
	if(cuda_stat != hipSuccess) { fprintf(stderr, "_dp1 %s\n", hipGetErrorString(cuda_stat)); }
#ifdef E1
//	cuda_stat = hipMalloc((void **) &_dp1, totalsize_dense*sizeof(short));
//	if(cuda_stat != hipSuccess) { fprintf(stderr, "_dp1 %s\n", hipGetErrorString(cuda_stat)); }
	cuda_stat = hipMalloc((void **) &_dp1v, totalsize_dense*sizeof(E1T));
	if(cuda_stat != hipSuccess) { fprintf(stderr, "_dp1v %s\n", hipGetErrorString(cuda_stat)); }
#endif


//fprintf(stderr, "ok5\n");


		hipMemset(_dp1, UIN, sizeof(short)*totalsize_dense);

		CUDPPResult res5 = cudppPlan(theCudpp5, &scanplan5, config5, s3g*PSIZE, s3g, PSIZE);

		preprocessing_step3<<<s3g, s3b>>>(DFACTOR, LOG_DFACTOR, _dcindex, _fx, _fy, _docc);		

//INT_PRINT(_dcindex, np*DFACTOR+1);
//TEMP_PRINT(_fx,57344);
//SHORT_PRINT(_fy,100000);
//INT_PRINT(_docc, 100);
//return;
//exit(0);
//fprintf(stderr, "ox1\n");	
		res5 = cudppMultiScan(scanplan5, _dcocc, _docc, PSIZE, s3g);
//fprintf(stderr, "ox2\n");	

//INT_PRINT(_dcocc, PSIZE*2);
//		preprocessing_step31<<<s3g, s3b>>>(_dcindex, _dcocc);

//INT_PRINT(_dcocc, PSIZE*2);
//return;

//SHORT_PRINT(_fx, 100);
//SHORT_PRINT(_fy, 100);
//INT_PRINT(_docc, 100);
//INT_PRINT(_dcocc, 100);
//return;

//fprintf(stderr, "ok6\n");


		preprocessing_step4<<<s3g, s3b>>>(DFACTOR, LOG_DFACTOR, _dcindex, _fx, _fy, _dcocc, _dp1
#ifdef E1
		, _dp1v, _fz
#endif
		);

		preprocessing_step41<<<(s3g+127)>>7, 128>>>(s3g, LOG_DFACTOR, _dcindex, _dccindex);

//INT_PRINT(_dcindex, 100);
//INT_PRINT(_dccindex, 100);
//return;

//INT_PRINT(&_dcocc[PSIZE],PSIZE);
//printf("\n");
//INT_PRINT(&_docc[PSIZE],PSIZE);
//return;




	

	}

hipFree(_fx); hipFree(_fy);
#ifdef E1
hipFree(_fz);
#endif 

/*	cuda_stat = hipMalloc((void **) &_sp2, sizeof(short)*ssize);
	if(cuda_stat != hipSuccess) { fprintf(stderr, "_sp2 %s\n", hipGetErrorString(cuda_stat)); }
	cuda_stat = hipMalloc((void **) &_mapper, sizeof(int)*CEIL(ssize,8));
	if(cuda_stat != hipSuccess) { fprintf(stderr, "_mapper %s\n", hipGetErrorString(cuda_stat)); }
	hipMemset(_sp2, UIN, sizeof(short)*ssize);
	hipMemset(_mapper, 0, sizeof(int)*CEIL(ssize,8));
#ifdef E1
	cuda_stat = hipMalloc((void **) &_sp1v, sizeof(E1T)*ssize);
	if(cuda_stat != hipSuccess) { fprintf(stderr, "_sp1v %s\n", hipGetErrorString(cuda_stat)); }
	hipMemset(_sp1v, UIN, sizeof(E1T)*ssize); // detailed value may be needed
#endif
*/

if(dcnt > 0) {
	preprocessing_step21<<<s3g, BSIZE>>>(DFACTOR, np, _cpb2, _dx, _dy, _sp2, _dmap);
}
/*
	preprocessing_step22<<<s1g, s1b>>>(DFACTOR, nv, ne, upper_ne, np, upper_np, _ncsc_v, _ncsc_e, _pb1, _cpb2, _ces,
	dcnt, _dcindex, _dx, _dy, _sp1, _sp2, _fx, _fy, _mapper
#ifdef E1
	, _ncsc_ev, _sp1v, _fz
#endif
	);*/


	hipDeviceSynchronize(); double e_time = rtclock();
//	fprintf(stdout, "preprocessing : %f ms,", (e_time - trans_end + trans_begin - s_time)*1000);
//exit(0);
//TEMP_PRINT(_ces, np+1);
////TEMP_PRINT(_pb1, DFACTOR*np*np);

	// disallocate non-used variables
	cudppDestroy(scanplan1); cudppDestroy(theCudpp);
	cudppDestroy(scanplan2); cudppDestroy(theCudpp2);
//	cudppDestroy(scanplan3); cudppDestroy(theCudpp3);
	cudppDestroy(scanplan33); cudppDestroy(theCudpp33);
	cudppDestroy(scanplan4); cudppDestroy(theCudpp4);
	hipFree(_pb1); 
	hipFree(_dcindex);
	hipFree(_docc);
	hipFree(_csc_size); 
	hipFree(_cpb2);

	hipMalloc((void **) &_ncsc_v, sizeof(int)*(nv+1));
	hipMemcpy(_ncsc_v, ncsc_v, sizeof(int)*(nv+1), hipMemcpyHostToDevice);
	hipMalloc((void **) &_ncsc_e, sizeof(int)*(ne));
	hipMemcpy(_ncsc_e, ncsc_e, sizeof(int)*ne, hipMemcpyHostToDevice);
#ifdef E1
	hipMalloc((void **) &_ncsc_ev, sizeof(E1T)*(ne));
	hipMemcpy(_ncsc_ev, ncsc_ev, sizeof(E1T)*ne, hipMemcpyHostToDevice);
#endif

	V1T *_i1;

	cuda_stat = hipMalloc((void **) &_i1, sizeof(V1T)*ssize);
	if(cuda_stat != hipSuccess) { fprintf(stderr, "_i1 %s\n", hipGetErrorString(cuda_stat)); }

	m->nv = nv; m->ne = ne; m->np = np;
	m->upper_nv = upper_nv; m->upper_np = upper_np;
	m->upper_ne = (m->ne) + (m->np)*(m->np)*8;
//	m->start_point = inp->start_p; // will be changed

	m->_ncsc_v = _ncsc_v; m->_ncsc_e = _ncsc_e;
	m->_itable = _itable; m->_ces = _ces; m->_cey = _cey;
	m->_sp1 = _sp1; m->_sp2 = _sp2;
	m->_mapper = _mapper;
	m->dcnt = s3g;
	m->_dx = _dx; m->_dy = _dy; m->_dccindex = _dccindex; m->_dmap = _dmap;
	m->_dp1 = _dp1;
#ifdef E1
	m->_ncsc_ev = _ncsc_ev; m->_sp1v = _sp1v; m->_dp1v = _dp1v;
#endif

	m->_i1 = _i1; // dep

#ifdef V2
	V2T *_i2;

	cuda_stat = hipMalloc((void **) &_i2, sizeof(V2T)*ssize);
	if(cuda_stat != hipSuccess) { fprintf(stderr, "_i2 %s\n", hipGetErrorString(cuda_stat)); }
	m->_i2 = _i2;
#endif
#ifdef V3
	V3T *_i3;

	cuda_stat = hipMalloc((void **) &_i3, sizeof(V3T)*ssize);
	if(cuda_stat != hipSuccess) { fprintf(stderr, "_i3 %s\n", hipGetErrorString(cuda_stat)); }
	m->_i3 = _i3;

#endif

        m->dsample_size = (m->np)/dsample_factor;
        m->dsample_dcnt = (m->dcnt)/dsample_factor;
        hipStreamCreate(&(m->stream1));
        hipStreamCreate(&(m->stream2));

	cuda_stat = hipMalloc((void **) &(m->_sample_partial), sizeof(float)*(m->mp)*16*32);
	if(cuda_stat != hipSuccess) { fprintf(stderr, "_sample_partial %s\n", hipGetErrorString(cuda_stat)); }
        cuda_stat = hipMalloc((void **) &(m->_temp_front), sizeof(int)*SCATTER_FACTOR);
	if(cuda_stat != hipSuccess) { fprintf(stderr, "_temp_front %s\n", hipGetErrorString(cuda_stat)); }
        hipMemset(m->_temp_front, 0, sizeof(int)*SCATTER_FACTOR);

	m->d_time = -1;
	cuda_stat = hipMalloc((void **) &(m->_finished), sizeof(int));
	if(cuda_stat != hipSuccess) { fprintf(stderr, "_finished %s\n", hipGetErrorString(cuda_stat)); }

	m->fflag = 0;
	m->degree = inp->degree;

	m->ssize = ssize;

	free(ncsc_v); free(ncsc_e);
#ifdef E1
	free(ncsc_ev);
#endif

//exit(0);
}

/*
void refreshV(struct HYB *m)
{

#ifdef INT_T
	hipMemset(_i1, UIN, sizeof(V1T)*ssize);
#endif
#ifdef FLOAT_T
	float *t0;
	t0 = (float *)malloc(sizeof(float)*ssize);
	for(int ik=0; ik<ssize; ik++) {
		t0[ik] = -10000;
//		t0[ik] = IUNUSED;
	}
	hipMemcpy(_i1, t0, sizeof(float)*ssize, hipMemcpyHostToDevice);
	free(t0);
#endif
#ifdef V2
	hipMemset(_i2, 0, sizeof(V2T)*ssize);
#endif
#ifdef V3
	hipMemset(_i3, 0, sizeof(V3T)*ssize);
#endif

}*/

