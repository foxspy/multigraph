#include "hip/hip_runtime.h"
//#include "common.h"
#include "user_function.h"

double rtclock(void)
{
  struct timezone Tzp;
  struct timeval Tp;
  int stat;
  stat = gettimeofday (&Tp, &Tzp);
  if (stat != 0) printf("Error return from gettimeofday: %d",stat);
  return(Tp.tv_sec + Tp.tv_usec*1.0e-6);
}

/*
__device__ inline int lane_id(void) { return (threadIdx.x&31); }

__device__ int warp_bcast(int v, int leader) { return __shfl(v, leader); }

__device__ int atomicAggInc(int *ctr) {
        int mask = __ballot(1);
        int leader = __ffs(mask) - 1;
        int res;
        if(lane_id() == leader)
                res = atomicAdd(ctr, __popc(mask));
        res = warp_bcast(res, leader);

        return (res + __popc(mask & ((1 << lane_id()) - 1)));
}

__device__ short atomicAddShort(short* address, short val) {
        unsigned int *base_address = (unsigned int *) ((char *)address - ((size_t)address & 2));
        unsigned int long_val = ((size_t)address & 2) ? ((unsigned int)val << 16) : (unsigned short)val;
        unsigned int long_old = atomicAdd(base_address, long_val);
        if((size_t)address & 2) {
                return (short)(long_old >> 16);
        } else {
                unsigned int overflow = ((long_old & 0xffff) + long_val) & 0xffff0000;
                if (overflow)
                        atomicSub(base_address, overflow);
                return (short)(long_old & 0xffff);
        }
}*/
