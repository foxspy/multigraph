#include "hip/hip_runtime.h"
#include "user_function.h"
//#include "common.h"

#if defined PR_D || defined PR_T

#if defined BFS || defined CC || defined BC || defined PR_T || defined PR_D
	#define SAME_FRONTIER
#endif
#if defined SSSP
	#define POSITIVE_FRONTIER
#endif

__global__ void _phase_sample(int *vv, float *sample_partial, int iter
#ifdef POSITIVE_FRONTIER
, int *vfc
#endif
)
{
	int index=blockIdx.x*blockDim.x+threadIdx.x;
#ifdef SAME_FRONTIER
	int val = vv[index];
#endif
#ifdef POSITIVE_FRONTIER
	int val = vfc[index];
#endif
	float fval;
	if(val == iter) fval = 1; else fval = 0;
        for(int offset = 16; offset > 0; offset = offset >> 1) {
                fval += __shfl_down(val, offset);
        }
        if(threadIdx.x == 0) sample_partial[blockIdx.x] = fval;
}

__global__ void _phase_sample_reduction(int size, float *sample_partial)
{
	__shared__ float sval[32];

	float val;
	if(threadIdx.x < size) val = sample_partial[threadIdx.x];	
	else val = 0;
        for(int offset = 16; offset > 0; offset = offset >> 1) {
                val += __shfl_down(val, offset);
        }
	if((threadIdx.x&31) == 0) {
		sval[threadIdx.x>>5] = val;
	}
	__syncthreads();
	if(threadIdx.x < 32) {
		val = sval[threadIdx.x];
	        for(int offset = 16; offset > 0; offset = offset >> 1) {
                	val += __shfl_down(val, offset);
      		}
	}
	if(threadIdx.x == 0) {
		sample_partial[threadIdx.x] = val;
	}

}

__global__ void _make_frontier(int nv, int iter, int *vv, int *f1, int *f2, int *f3, int *f4, int *pointer, int *vfc)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int warp_id = ((threadIdx.x>>5)&3);
	if(index < nv) {
#if defined SAME_FRONTIER
		if(vv[index] == iter) {
#endif
#if defined POSITIVE_FRONTIER
		if(vfc[index] == iter) {
#endif
			if(warp_id == 0) f1[atomicAggInc(&pointer[0])] = index;
			else if(warp_id == 1) f2[atomicAggInc(&pointer[1])] = index;
			else if(warp_id == 2) f3[atomicAggInc(&pointer[2])] = index;
			else f4[atomicAggInc(&pointer[3])] = index;
		}
	}
}


__global__ void _pseudo_phase2(int nv, int ne, int np, short *sp2, V1T *vv, V1T *i1, int *ces, int *finished, int iter, int *vfc
#ifdef V2
, V2T *vv2, V2T *i2
#endif
)
{
	__shared__ V1T sv[PSIZE];

	int base_addr = blockIdx.x*PSIZE;
	int i;
	int *dummy;

        for(i=threadIdx.x; i<PSIZE; i+=blockDim.x) {
                ////sv[i] = IUNUSED;
#ifdef BFS
//		sv[i] = BFS_INF;
#endif
/*
		initialize(&sv[i], IUNUSED
#ifdef V2
		, dummy, 0
#endif
		);*/
		sv[i] = 0.0f;

        }
        __syncthreads();

//if(cpb2[fac*blockIdx.x] < 0 || cpb2[fac*(blockIdx.x+1)] > 1648304) printf("errr %d %d %d\n", blockIdx.x, cpb2[fac*blockIdx.x], cpb2[fac*(blockIdx.x+1)]);

        for(i=ces[blockIdx.x]+threadIdx.x;i<ces[blockIdx.x+1]; i+=blockDim.x) {
                int ii=sp2[i];
/*
#ifdef BFS
		if(i1[i] < IUNUSED && ii < PSIZE) {
			sv[ii] = MIN(sv[ii], i1[i]);
		}
#endif
*/

		if(i1[i] > -0.01f && ii < PSIZE) {
			atomicAdd(&sv[ii], i1[i]);
/*
			accumulate(&sv[ii], i1[i], iter
#ifdef V2
			, dummy, *dummy
#endif
			);*/

//printf("(%d %d)", i1[i], sv2[ii]);
		}

        }
        __syncthreads();


        int flag = false;
        for(i=threadIdx.x; i<PSIZE; i+=blockDim.x) {

#ifdef PR_T
                float f1 = (1-PR_DAMPING_FACTOR) + sv[i] * PR_DAMPING_FACTOR;
                float f2 = vv[i+base_addr];
                if(fabs(f1 - f2) > PR_TOLERANCE) {
                        vv[i+base_addr] = f1;
                        flag = true;
                }
#endif
#ifdef PR_D
                float f1 = (1-PR_DAMPING_FACTOR) + sv[i] * PR_DAMPING_FACTOR;
                float f2 = vv[i+base_addr];
                if(vfc[i+base_addr] == 1 && fabs(f1 - f2) > PR_TOLERANCE*f2) {
                                flag = true;
                }
#endif

/*
		flag = update_condition(&vv[i+base_addr], &vv[i+base_addr], sv[i], iter, &vfc[i+base_addr]
#ifdef V2
		, &vv2[i+base_addr], &vv2[i+base_addr], sv2[i]
#endif
		);
*/
                ////if(vv[i+base_addr] > sv[i]) {
                ////        vv[i+base_addr] = sv[i];
                ////        flag = true;
                ////}
        }
//if(flag == true) printf("true %d\n", threadIdx.x);
        if(__syncthreads_or(flag)) {
                if(threadIdx.x == 0) {
                        (*finished) = 1;
                }
        }

}







__global__ void _process_f1(int nv, int tf_size, int *csc_v, int *csc_e, int *gl, int nn1, int *f1, int *n_f1, int nn2, int *f2, int *n_f2, int nn3, int *f3, int *n_f3, int nn4, int *f4, int *n_f4, V1T *vv, V1T *nvv, int iter, int flag, int *vfc
#ifdef V2
, V2T *vv2
#endif
#ifdef E1
, E1T *csc_ev
#endif
)
{
	__shared__ int buffer1[64], buffer2[64], buffer3[64], buffer4[64];
	float buffer5[64];
	__shared__ int buffer_p[2];

	int i, j;
	int index = blockIdx.x*64 + (threadIdx.x>>2);
	int base = blockIdx.x*64;
//	int checker = (blockIdx.x & (sample_factor-1));
//	if(flag == 0 && checker > 0) return;
//	else if(flag == 1 && checker == 0) return;

	int warp_id, index_size, bias;
	if(threadIdx.x < 2) {
		buffer_p[threadIdx.x] = 0;
	}
/*
	if(threadIdx.x < 64 && base+threadIdx.x < nv) {
		if(vv2[base+threadIdx.x] > 0) {
			buffer5[threadIdx.x] = vv[base+threadIdx.x] / (float)vv2[base+threadIdx.x];
		}
		else {
			buffer5[threadIdx.x] = 0;
		}
	}*/
	__syncthreads();
		warp_id = ((threadIdx.x>>5)&3);
	if(index < nv) {
		bias = 0;
			
			index_size = csc_v[index+1] - csc_v[index];
			if(index_size >= 32) {
			bias = index_size - (index_size&31);
			if((threadIdx.x&3) == 0) {
				int p = atomicAggInc(&buffer_p[0]);
				buffer1[p] = index;
				buffer2[p] = bias;
			}
			if(index_size >= 256) {
				if((threadIdx.x&3) == 0) {
					int p2 = atomicAggInc(&buffer_p[1]);
					buffer3[p2] = index;
					buffer4[p2] = index_size - (index_size&255);
				}
			}
		}
		float delta;
		if(vv2[index] != 0) delta = vv[index]/(float)vv2[index];
		else delta = 0;
		for(i=bias+(threadIdx.x&3); i<index_size; i+=4) {
			int index_dst = csc_e[csc_v[index]+i];

/*			V1T tmp;
#ifdef V2
			V2T tmp2;
#endif
			get_partial_result(&tmp, buffer5[index-base]
#ifdef V2
			, &tmp2, vv2[index]
#endif
#ifdef E1
			, csc_ev[csc_v[index]+i]
#endif
			); // bfs can be optimized
			int *dummy;*/
//			atomicAdd(&nvv[index_dst], buffer5[index-base]);
			if(vv2[index] != 0) atomicAdd(&nvv[index_dst], delta);


/*
			if(fused_update_condition(&vv[index_dst], &nvv[index_dst], tmp, iter, &vfc[index_dst]
#ifdef V2
			, dummy, dummy, tmp2
#endif
			)) {
				if(warp_id == 0) n_f1[atomicAggInc(&gl[0])] = index_dst;
				else if(warp_id == 1) n_f2[atomicAggInc(&gl[1])] = index_dst;
				else if(warp_id == 2) n_f3[atomicAggInc(&gl[2])] = index_dst;
				else n_f4[atomicAggInc(&gl[3])] = index_dst;
			}*/

			//if(vv[index_dst] == IUNUSED) {
			//	nvv[index_dst] = iter;
			//	if(warp_id == 0) n_f1[atomicAggInc(&gl[0])] = index_dst;
			//	else if(warp_id == 1) n_f2[atomicAggInc(&gl[1])] = index_dst;
			//	else if(warp_id == 2) n_f3[atomicAggInc(&gl[2])] = index_dst;
			//	else n_f4[atomicAggInc(&gl[3])] = index_dst;
			//}
		}
	}
	__syncthreads();
	for(i=(threadIdx.x>>5);i<buffer_p[0];i+=8) {
		index = buffer1[i];
		int bf2 = buffer2[i];
		int bf22 = bf2 - (bf2&255);

		float delta;
		if(vv2[index] != 0) delta = vv[index]/(float)vv2[index];
		else delta = 0;
		for(j=bf22+(threadIdx.x&31);j<bf2;j+=32) {
			int index_dst = csc_e[csc_v[index]+j];
//if(index_dst < 0 || index_dst >= nv) printf("err\n");

/*			V1T tmp;
#ifdef V2
			V2T tmp2;
#endif
			get_partial_result(&tmp, buffer5[index-base]
#ifdef V2
			, &tmp2, vv2[index]
#endif
#ifdef E1
			, csc_ev[csc_v[index]+j]
#endif
			); // bfs can be optimized
*/
			//atomicAdd(&nvv[index_dst], buffer5[index-base]);
			if(vv2[index] != 0) atomicAdd(&nvv[index_dst], delta);

/*			if(fused_update_condition(&vv[index_dst], &nvv[index_dst], tmp, iter, &vfc[index_dst]
#ifdef V2
			, &vv2[index_dst], &vv2[index_dst], tmp2
#endif
			)) {
				if(warp_id == 0) n_f1[atomicAggInc(&gl[0])] = index_dst;
				else if(warp_id == 1) n_f2[atomicAggInc(&gl[1])] = index_dst;
				else if(warp_id == 2) n_f3[atomicAggInc(&gl[2])] = index_dst;
				else n_f4[atomicAggInc(&gl[3])] = index_dst;
			}*/

			//if(vv[index_dst] == IUNUSED) {
			//	nvv[index_dst] = iter;
			//	if(warp_id == 0) n_f1[atomicAggInc(&gl[0])] = index_dst;
			//	else if(warp_id == 1) n_f2[atomicAggInc(&gl[1])] = index_dst;
			//	else if(warp_id == 2) n_f3[atomicAggInc(&gl[2])] = index_dst;
			//	else n_f4[atomicAggInc(&gl[3])] = index_dst;
			//}	
		}
	}
	for(i=0;i<buffer_p[1];i++) {
		index = buffer3[i];

		float delta;
		if(vv2[index] != 0) delta = vv[index]/(float)vv2[index];
		else delta = 0;


		for(j=threadIdx.x;j<buffer4[i];j+=blockDim.x) {
			int index_dst = csc_e[csc_v[index]+j];
//if(index_dst < 0 || index_dst >= nv) printf("err\n");
/*
			V1T tmp;
#ifdef V2
			V2T tmp2;
#endif
			get_partial_result(&tmp, buffer5[index-base]
#ifdef V2
			, &tmp2, vv2[index]
#endif
#ifdef E1
			, csc_ev[csc_v[index]+j]
#endif
			); // bfs can be optimized*/

			//atomicAdd(&nvv[index_dst], buffer5[index-base]);
			if(vv2[index] != 0) atomicAdd(&nvv[index_dst], delta);

/*			if(fused_update_condition(&vv[index_dst], &nvv[index_dst], tmp, iter, &vfc[index_dst]
#ifdef V2
			, &vv2[index_dst], &vv2[index_dst], tmp2
#endif
			)) {
				if(warp_id == 0) n_f1[atomicAggInc(&gl[0])] = index_dst;
				else if(warp_id == 1) n_f2[atomicAggInc(&gl[1])] = index_dst;
				else if(warp_id == 2) n_f3[atomicAggInc(&gl[2])] = index_dst;
				else n_f4[atomicAggInc(&gl[3])] = index_dst;
			}*/

			//if(vv[index_dst] == IUNUSED) {
			//	nvv[index_dst] = iter;
			//	       if(warp_id == 0) n_f1[atomicAggInc(&gl[0])] = index_dst;
			//	       else if(warp_id == 1) n_f2[atomicAggInc(&gl[1])] = index_dst;
			//	       else if(warp_id == 2) n_f3[atomicAggInc(&gl[2])] = index_dst;
			//	       else n_f4[atomicAggInc(&gl[3])] = index_dst;
			//}
		}
	}
}


__global__ void process_f3(int nv, int tf_size, int *gl, int nn1, int *f1, int *n_f1, int nn2, int *f2, int *n_f2, int nn3, int *f3, int *n_f3, int nn4, int *f4, int *n_f4, V1T *vvc, V1T *n_vvc, int *v_front)
{
        int index = blockIdx.x*256 + (threadIdx.x); // blocksize = 128, 4thread=1vertex
        int warp_id;
if(index < nv) {

        warp_id = ((threadIdx.x>>5)&3);
        V1T ff1 = (1-PR_DAMPING_FACTOR) + n_vvc[index] * PR_DAMPING_FACTOR;
        V1T ff2 = vvc[index];
        if(v_front[index] == 1 && fabs(ff1 - ff2) > PR_TOLERANCE*ff2) {
                vvc[index] = ff1;
                if(warp_id == 0) n_f1[atomicAggInc(&gl[0])] = index;
                else if(warp_id == 1) n_f2[atomicAggInc(&gl[1])] = index;
                else if(warp_id == 2) n_f3[atomicAggInc(&gl[2])] = index;
                else n_f4[atomicAggInc(&gl[3])] = index;
        } else {
//      printf("no conv\n");
                vvc[index] = ff1;
                v_front[index] = 0;//reverse
        }
//	n_vvc[index] = 0;

}




}

	// relaseValue can be added
__global__ void _phase1(int nv, int ne, int upper_ne, int np, short *sp1, V1T *vv, V1T *i1, int *ces, int *mapper
#ifdef V2
, V2T *vv2, V2T *i2
#endif
#ifdef E1
, E1T *sp1v
#endif
)
{
	__shared__ V1T sv[PSIZE];

//	if(sampled == 1 && (blockIdx.x & (ssample-1)) == 0) return;

	int base_addr = blockIdx.x*PSIZE;
	int i, index;
	short temp;
	for(i=threadIdx.x; i<PSIZE; i+=blockDim.x) {
		sv[i] = vv[i+base_addr];
#ifdef V2
		if(vv2[base_addr+i] != 0) sv[i] = sv[i]/(float)vv2[base_addr+i];
		else sv[i] = 0;
#endif



	}
	__syncthreads();
//printf("(%d %d)\n", ces[blockIdx.x], ces[blockIdx.x+1]);
	for(i=ces[blockIdx.x]+threadIdx.x; i<ces[blockIdx.x+1]; i+=blockDim.x) {
//printf("(%d)\n", i);
		index = mapper[i>>3]+(threadIdx.x&7);
		temp = sp1[i];
		if(temp < PSIZE) {
//		int temp3 = sv2[temp];
			i1[index]=sv[temp];
//			if(temp3 != 0) i1[index]=sv[temp]/(float)temp3; else i1[index] = 0.0f;
//#ifdef BFS
//			i1[index] = sv[temp]+1;
//#endif
			////i1[index] = sv[temp]+1;
/*			get_partial_result(&i1[index], sv[temp]
#ifdef V2
			, &i2[index], sv2[temp]
#endif
#ifdef E1
			, sp1v[i]
#endif
			);*/

		}
	}
}
	// DFACTOR, LOG_DFACTOR will be removed
__global__ void _phase11(int nv, int np, V1T *vv, V1T *i1, int *dx, int *dy, int *dindex, short *dp1, int *dmap, int iter
#ifdef V2
, V2T *vv2, V2T *i2
#endif
#ifdef E1
, E1T *dp1v
#endif
)
{
	__shared__ V1T sv[PSIZE];

	int i;
	int x_base = dx[blockIdx.x];//, y_base = dy[blockIdx.x];

//	if(sampled == 1 && (x_base & (ssample-1)) == 0) return;

//if(x_base < 0 || x_base >= np || y_base < 0 || y_base >= np) printf("err0\n");
	short curr_index = SUNUSED;
	V1T curr_v, temp_v;
#ifdef V2
	V2T curr_v2, temp_v2;
#endif
	initialize(&curr_v, 0
#ifdef V2
	, &curr_v2, 0
#endif
	);

	int dmap_index = dmap[blockIdx.x];
	for(i=threadIdx.x;i<PSIZE;i+=blockDim.x) {
//if(x_base*PSIZE+i < 0 || x_base*PSIZE+i >= nv) printf("err1\n");
		sv[i] = vv[x_base*PSIZE+i];

#ifdef V2
		if(vv2[x_base*PSIZE+i] != 0) sv[i] = sv[i]/(float)vv2[x_base*PSIZE+i];
		else sv[i] = 0;
#endif
//		sv2[i] = vv2[x_base*PSIZE+i];
		i1[dmap_index+i] = 0.0f;
	}
	__syncthreads();

	for(i=dindex[blockIdx.x]+threadIdx.x;i<dindex[blockIdx.x+1];i+=blockDim.x) {
		short edge_value = dp1[i];
		if(edge_value < 0) {
//if(edge_value < -1*PSIZE) printf("errxx\n");
			if(curr_index != SUNUSED) {
/*
#ifdef BFS
				if(curr_v <= iter && curr_v < i1[dmap_index+curr_index]) {
					i1[dmap_index+curr_index] = curr_v;
				}
#endif
#ifdef SSSP
				atomicMin(&i1[dmap_index+curr_index], curr_v);
#endif
#ifdef BC
				if(curr_v < iter && curr_v < i1[dmap_index+curr_index]) {
					i1[dmap_index+curr_index] = curr_v+1;
				}
				if((curr_v == iter-1 && (i1[dmap_index+curr_index] == BFS_INF || i1[dmap_index+curr_index] == iter))) {
					atomicAdd(&i2[dmap_index+curr_index], curr_v2);
				}
#endif
*/
				accumulate(&i1[dmap_index+curr_index], curr_v, iter
#ifdef V2
				, &i2[dmap_index+curr_index], curr_v2
#endif
				);
			}
			curr_index = -(edge_value+1);
//if(curr_index < 0 || curr_index >= PSIZE) printf("erryy\n");
			initialize(&curr_v, 0
#ifdef V2
			, &curr_v2, 0
#endif
			);
//			curr_v = IUNUSED;
		} else if(edge_value != SUNUSED) {
			temp_v = sv[edge_value];
			V1T tmp;
#ifdef V2
			V2T tmp2;
#endif
			get_partial_result(&tmp, temp_v
#ifdef V2
			, &tmp2, temp_v2
#endif
#ifdef E1
			, dp1v[i]
#endif
			);
			accumulate_nonAtomics(&curr_v, tmp, iter
#ifdef V2
			, &curr_v2, tmp2
#endif
			);
			//if(temp_v + 1 < curr_v) curr_v = temp_v+1;
		}
	}
	if(curr_index != SUNUSED) {
		accumulate(&i1[dmap_index+curr_index], curr_v, iter
#ifdef V2
		, &i2[dmap_index+curr_index], curr_v2
#endif
		);
	}
	
	//if(curr_index != SUNUSED && curr_v < i1[dmap_index + curr_index] && curr_v <= iter)
	//	i1[dmap_index+curr_index] = curr_v;
}

	//DFAFCTOR will be removed
__global__ void _phase2(int nv, int ne, int np, short *sp2, V1T *vv, V1T *i1, int *ces, int *finished, int iter, int *vfc
#ifdef V2
, V2T *vv2, V2T *i2
#endif
)
{
	__shared__ V1T sv[PSIZE];

	int base_addr = blockIdx.x*PSIZE;
	int i;
	int *dummy;

        for(i=threadIdx.x; i<PSIZE; i+=blockDim.x) {
                ////sv[i] = IUNUSED;
#ifdef BFS
//		sv[i] = BFS_INF;
#endif
/*
		initialize(&sv[i], IUNUSED
#ifdef V2
		, dummy, 0
#endif
		);*/
		sv[i] = 0.0f;

        }
        __syncthreads();

//if(cpb2[fac*blockIdx.x] < 0 || cpb2[fac*(blockIdx.x+1)] > 1648304) printf("errr %d %d %d\n", blockIdx.x, cpb2[fac*blockIdx.x], cpb2[fac*(blockIdx.x+1)]);

        for(i=ces[blockIdx.x]+threadIdx.x;i<ces[blockIdx.x+1]; i+=blockDim.x) {
                int ii=sp2[i];
/*
#ifdef BFS
		if(i1[i] < IUNUSED && ii < PSIZE) {
			sv[ii] = MIN(sv[ii], i1[i]);
		}
#endif
*/

		if(i1[i] > -0.01f && ii < PSIZE) {
			atomicAdd(&sv[ii], i1[i]);
/*
			accumulate(&sv[ii], i1[i], iter
#ifdef V2
			, dummy, *dummy
#endif
			);*/

//printf("(%d %d)", i1[i], sv2[ii]);
		}

        }
        __syncthreads();


        int flag = false;
        for(i=threadIdx.x; i<PSIZE; i+=blockDim.x) {

#ifdef PR_T
                float f1 = (1-PR_DAMPING_FACTOR) + sv[i] * PR_DAMPING_FACTOR;
                float f2 = vv[i+base_addr];
                if(fabs(f1 - f2) > PR_TOLERANCE) {
                        vv[i+base_addr] = f1;
                        flag = true;
                }
#endif
#ifdef PR_D
                float f1 = (1-PR_DAMPING_FACTOR) + sv[i] * PR_DAMPING_FACTOR;
                float f2 = vv[i+base_addr];
                if(vfc[i+base_addr] == 1 && fabs(f1 - f2) > PR_TOLERANCE*f2) {
                                vv[i+base_addr] = f1;
                                flag = true;
                } else {
                                vv[i+base_addr] = f1;
                                vfc[i+base_addr] = 0;
                }

#endif

/*
		flag = update_condition(&vv[i+base_addr], &vv[i+base_addr], sv[i], iter, &vfc[i+base_addr]
#ifdef V2
		, &vv2[i+base_addr], &vv2[i+base_addr], sv2[i]
#endif
		);
*/
                ////if(vv[i+base_addr] > sv[i]) {
                ////        vv[i+base_addr] = sv[i];
                ////        flag = true;
                ////}
        }
//if(flag == true) printf("true %d\n", threadIdx.x);
        if(__syncthreads_or(flag)) {
                if(threadIdx.x == 0) {
                        (*finished) = 1;
                }
        }

}


//__global__ vv_kernel<<<(nv+255)>>8, 256>>>(m->nv, m->_vv, m->_finished); 


int MultiGraph_V_E_V_NOTIDEM(struct HYB *m, struct csc_package *inp, struct vector_data *vd, struct OUTPUT *out, int *g_iter) // result : dep
{

	int iter=*g_iter;
	int finished=0;
	int palgo = vd->algo;

//	int *_vv=vd->_vv;
//	int *_curr_f=vd->_curr_f, *_next_f=vd->_next_f, *_gl=vd->_gl;
	int *_temp_f;
//	int *h_fs=vd->h_fs;

//	initValue(m, &_vv);

//	initFrontier(m, &(vd->_curr_f), &_next_f, &_gl);

//	hipMalloc((void **) &_finished, sizeof(int));

	// sampling
//        hipStream_t stream1=m->stream1, stream2=m->stream2;


//	int *_dummy_vv=vd->_nvv;

	int d_threshold=IUNUSED, sampled = 0; // if added

	// add variables
	float sample_partial;
	//int *_temp_front;
	//hipMalloc((void **) &_temp_front, sizeof(int)*SCATTER_FACTOR);
	//hipMemset(_temp_front, 0, sizeof(int)*SCATTER_FACTOR);
#if defined SPARSE_MODE || DENSE_MODE
m->d_time = 100;
#endif

	if(m->d_time < 0) {
		hipDeviceSynchronize();
		double sample_start = rtclock(); // start time

			_phase1<<<(m->dsample_size),BSIZE,0,m->stream1>>>((m->nv), (m->ne), (m->upper_ne), (m->np), (m->_sp1), (vd->_vv), (m->_i1), (m->_ces), (m->_mapper)
#ifdef V2
			, (vd->_vv2), (m->_i2)
#endif
#ifdef E1
			, (m->_sp1v)
#endif
			);
			_phase11<<<(m->dsample_dcnt),BSIZE,0,m->stream2>>>((m->nv), (m->np), (vd->_vv), (m->_i1), (m->_dx), (m->_dy), (m->_dccindex), (m->_dp1), (m->_dmap), iter
#ifdef V2
			, (vd->_vv2), (m->_i2)
#endif
#ifdef E1
			, (m->_dp1v)
#endif
			);

			_pseudo_phase2<<<(m->dsample_size),BSIZE>>>((m->nv), (m->ne), (m->np), (m->_sp2), (vd->_vv), (m->_i1), (m->_cey), m->_finished, iter, vd->_vfc
#ifdef V2
		, (vd->_vv2), (m->_i2)
#endif
			);


		hipDeviceSynchronize();
		double sample_end = rtclock();
		(m->d_time) = (sample_end - sample_start) * dsample_factor * 2;
#ifdef TRACE
		fprintf(stdout, "sample : %f\n", 1000* ( m->d_time));
#endif
	}

#if defined SPARSE_MODE
m->fflag = 2;
vd->algo = SPARSE;
#endif
#if defined DENSE_MODE
m->fflag = 2;
vd->algo = DENSE;
#endif		
	
		if(m->fflag < 1) {

			hipDeviceSynchronize();
			double sample_start2 = rtclock();



			hipMemset(vd->_gl, 0, sizeof(int)*SCATTER_FACTOR);

			_process_f1<<<((vd->htot_size)+255)>>(6+2), 256>>>((m->nv), (vd->htot_size), (m->_ncsc_v), (m->_ncsc_e), vd->_gl, vd->h_fs[0], &(vd->_curr_f[0]), &(vd->_next_f[0]),
			vd->h_fs[1], &(vd->_curr_f[m->nv]), &(vd->_next_f[(m->nv)]), vd->h_fs[2], &(vd->_curr_f[(m->nv)*2]), &(vd->_next_f[(m->nv)*2]), vd->h_fs[3], &(vd->_curr_f[(m->nv)*3]), &(vd->_next_f[(m->nv)*3]), (vd->_vv), (vd->_nvv), iter, sampled, (vd->_vfc)
#ifdef V2
			, vd->_vv2
#endif
#ifdef E1
, (m->_ncsc_ev)
#endif
);
			hipDeviceSynchronize();
			double sample_end2 = rtclock();
			double s_time = (sample_end2 - sample_start2) * ssample_factor * 4;
			if((m->d_time) > s_time) vd->algo = SPARSE;
			else vd->algo = DENSE;

		}

		if(palgo == vd->algo) m->fflag++;

		if(vd->algo == DENSE) {
#ifdef TRACE
			hipDeviceSynchronize();
			double local_start = rtclock();
#endif
			if(d_threshold == IUNUSED) d_threshold = MIN(d_threshold, (vd->htot_size));
//printf("((%d))\n", (vd->htot_size));
			vd->fhave = 0;
			finished = 0;
			hipMemcpyAsync(m->_finished, &finished, sizeof(int), hipMemcpyHostToDevice);
	
			_phase1<<<(m->np),BSIZE,0,m->stream1>>>((m->nv), (m->ne), (m->upper_ne), (m->np), (m->_sp1), (vd->_vv), (m->_i1), (m->_ces), (m->_mapper)
#ifdef V2
			, (vd->_vv2), (m->_i2)
#endif
#ifdef E1
			, (m->_sp1v)
#endif
			);
			_phase11<<<(m->dcnt),BSIZE,0,m->stream2>>>((m->nv), (m->np), (vd->_vv), (m->_i1), (m->_dx), (m->_dy), (m->_dccindex), (m->_dp1), (m->_dmap), iter
#ifdef V2
			, (vd->_vv2), (m->_i2)
#endif
#ifdef E1
			, (m->_dp1v)
#endif
			);

			_phase2<<<(m->np),BSIZE>>>((m->nv), (m->ne), (m->np), (m->_sp2), (vd->_vv), (m->_i1), (m->_cey), m->_finished, iter, vd->_vfc
#ifdef V2
		, (vd->_vv2), (m->_i2)
#endif
			);
			hipMemcpyAsync(&finished, m->_finished, sizeof(int), hipMemcpyDeviceToHost);
//			if(finished == 0) break;
			if(finished == 0) {
				 out->r = (vd->_vv); return false; 
			}
			*g_iter = iter+1;
#ifdef TRACE
			hipDeviceSynchronize();
			double local_end = rtclock();
			printf("0 %d %f\n", iter-1, (local_end-local_start)*1000);
//			printf("0 %d %f %d\n", iter-1, (local_end-local_start)*1000, d_threshold);
#endif
		} else {
#ifdef TRACE
			hipDeviceSynchronize();
			double local_start = rtclock();
#endif

//printf("%d %d %d %d %d\n", vd->h_fs[0], vd->h_fs[1], vd->h_fs[2], vd->h_fs[3], vd->htot_size);
hipMemset(vd->_nvv, 0, sizeof(float)*(m->nv));
hipDeviceSynchronize();

			_process_f1<<<((m->nv)+255)>>6, 256>>>((m->nv), (vd->htot_size), (m->_ncsc_v), (m->_ncsc_e), vd->_gl, vd->h_fs[0], &(vd->_curr_f[0]), &(vd->_next_f[0]),
			vd->h_fs[1], &(vd->_curr_f[m->nv]), &(vd->_next_f[(m->nv)]), vd->h_fs[2], &(vd->_curr_f[(m->nv)*2]), &(vd->_next_f[(m->nv)*2]), vd->h_fs[3], &(vd->_curr_f[(m->nv)*3]), &(vd->_next_f[(m->nv)*3]), (vd->_vv), (vd->_nvv), iter, sampled, (vd->_vfc)
#ifdef V2
			, vd->_vv2
#endif
#ifdef E1
, (m->_ncsc_ev)
#endif
);

			hipMemset(vd->_gl, 0, sizeof(int)*SCATTER_FACTOR);			

//printf("%d %d %d %d %d\n", vd->h_fs[0], vd->h_fs[1], vd->h_fs[2], vd->h_fs[3], vd->htot_size);

			process_f3<<<((m->nv)+255)>>8, 256>>>(m->nv, vd->htot_size, vd->_gl
			,vd->h_fs[0], &(vd->_curr_f[0]), &(vd->_next_f[0]),
			vd->h_fs[1], &(vd->_curr_f[m->nv]), &(vd->_next_f[(m->nv)]), vd->h_fs[2], &(vd->_curr_f[(m->nv)*2]), &(vd->_next_f[(m->nv)*2]), vd->h_fs[3], &(vd->_curr_f[(m->nv)*3]), &(vd->_next_f[(m->nv)*3]), (vd->_vv), (vd->_nvv), (vd->_vfc));

//printf("%d %d %d %d %d\n", vd->h_fs[0], vd->h_fs[1], vd->h_fs[2], vd->h_fs[3], vd->htot_size);
			
			hipMemcpy(vd->h_fs, vd->_gl, sizeof(int)*SCATTER_FACTOR, hipMemcpyDeviceToHost);
			(vd->htot_size) = (vd->h_fs[0])+(vd->h_fs[1])+(vd->h_fs[2])+(vd->h_fs[3]);

			if(vd->htot_size == 0) { (out->r) = (vd->_vv); return false; }


//if(iter == 50) return false;
//fprintf(stdout, "%d %d\n", iter, vd->htot_size);

			*g_iter = iter+1;
//			_temp_f = (vd->_curr_f); (vd->_curr_f) = (vd->_next_f); (vd->_next_f) = _temp_f;
#ifdef TRACE
			hipDeviceSynchronize();
			double local_end = rtclock();
			printf("1 %d %f\n", iter-1, (local_end-local_start)*1000);
#endif
		}		
//	}
//	hipDeviceSynchronize();
//	double total_end = rtclock();
//	fprintf(stdout, "%d %f\n", iter, (total_end - sample_start)*1000);
	out->r = (vd->_vv);
#ifdef V2
	out->r2 = (vd->_vv2);
#endif
#ifdef V3
	out->r3 = (vd->_vv3);
#endif
	return true;
}


#endif
