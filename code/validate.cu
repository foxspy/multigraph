#include "hip/hip_runtime.h"
//#include "common.h"
#include "user_function.h"

// need to be update (CSC)
// vv -> in HYB?
void verifyResults(struct csc_package *inp, GRAPHTYPE *m, struct OUTPUT *out, int source_vertex) // vv : dep
{
#ifdef VALIDATE

V1T *_vv = out->r;
int *tt;
//INT_PRINT(_vv,20);

	int *csc_v = inp->csc_v, *csc_e = inp->csc_e;
	int i;
#ifdef E1
	E1T *csc_ev = inp->csc_ev;
#endif
	int nv = m->nv;
        V1T *vv;
        vv = (V1T *)malloc(sizeof(V1T)*nv);
        hipMemcpyAsync(vv, _vv, sizeof(V1T)*nv, hipMemcpyDeviceToHost);

#ifndef CC
	int *_itable = m->_itable;
	int *itable;
        itable = (int *)malloc(sizeof(int)*nv);
        hipMemcpyAsync(itable, _itable, sizeof(int)*nv, hipMemcpyDeviceToHost);
#endif

        V1T *c_vv;
        c_vv = (V1T *)malloc(sizeof(V1T)*nv);

#if defined PR_D || defined PR_T
//	float *c_vv = (float *)malloc(sizeof(float)*nv);
	float *c_nvv = (float *)malloc(sizeof(float)*nv);
	memset(c_vv, 0, sizeof(float)*nv);
	
	int *vmark = (int *)malloc(sizeof(int)*nv);
	for(i=0;i<nv;i++)
		vmark[i]=1;

	int flag;



	while(1) {
		flag = false;
		memset(c_nvv, 0, sizeof(float)*nv);
		for(i=0;i<nv;i++) {
			int deg = (csc_v[i+1]-csc_v[i]);
			for(int j=csc_v[i]; j<csc_v[i+1]; j++) {
				int k = csc_e[j];
				if(deg > 0) 
					c_nvv[k] += c_vv[i]/(float)deg;
			}
		}

#ifdef PR_T
		for(i=0;i<nv;i++) {
	                float f1 = (1-PR_DAMPING_FACTOR) + c_nvv[i] * PR_DAMPING_FACTOR;
	    		float f2 = c_vv[i];
	                if(fabs(f1 - f2) > PR_TOLERANCE) {
	                        c_vv[i] = f1;
	                        flag = true;
	                } else {
				c_vv[i] = f2;
			}				
		}
#endif
#ifdef PR_D
		for(i=0;i<nv;i++) {
	                float f1 = (1-PR_DAMPING_FACTOR) + c_nvv[i] * PR_DAMPING_FACTOR;
	                float f2 = c_vv[i];
	                if(vmark[i] == 1 && fabs(f1 - f2) > PR_TOLERANCE*f2) {
	                                c_vv[i] = f1;
	                                flag = true;
	                } else {
	                                c_vv[i] = f1;
	                                vmark[i] = 0;
	                }
		}

#endif
		if(flag == false) break;
/*	
		for(i=0;i<100;i++) {
			fprintf(stdout, "%d ", c_vv[i]);
		} fprintf(stdout, "\n");*/

	}

	for(i=0;i<nv;i++) {
		int t = MAX(vv[itable[i]],c_vv[i]);
		if(t == 0) continue;
		if(abs(vv[itable[i]] - c_vv[i])/t > THR) {
			break;
//			printf("%d diff : %f %f\n", i, vv[itable[i]], c_vv[i]);
		}
	}
	if(i == nv) printf("validation : PASS\n");
	else printf("validation : FAIL\n");

#endif

#if defined BFS 
        int *c_q;
        int qhead=0, qtail=1;
        c_q = (int *)malloc(sizeof(int)*nv);
        memset(c_vv, UIN, sizeof(int)*nv);
        c_vv[source_vertex] = 0; c_q[0] = source_vertex; // 0 = start_point

        int c_iter=0;
        while(1) {
                if(qhead == qtail) break;
                c_iter++;
                i = c_q[qhead%nv];
                qhead++;
                for(int j=csc_v[i]; j<csc_v[i+1]; j++) {
                        int k = csc_e[j];
                        if(c_vv[k] == IUNUSED) {
                                c_vv[k] = c_vv[i]+1;
                                c_q[qtail%nv] = k;
                                qtail++;
                        }
                }
        }


#endif
#ifdef SSSP
        memset(c_vv, UIN, sizeof(int)*nv);
        c_vv[source_vertex] = 0; // 0 = start_point
	int conv;
	while(1) {
		conv = true;
		for(i=0; i<nv; i++) {
			for(int j=csc_v[i]; j<csc_v[i+1]; j++) {
				if(c_vv[i] + csc_ev[j] < c_vv[csc_e[j]]) {
					conv = false;
					c_vv[csc_e[j]] = c_vv[i] + csc_ev[j];
				}
			}
		}
		if(conv == true) break;
	}

#endif
#ifdef BC
        int *c_q;
        int qhead=0, qtail=1;
        c_q = (int *)malloc(sizeof(int)*nv*1.5);
        memset(c_vv, UIN, sizeof(int)*nv);
        c_vv[source_vertex] = 0; c_q[0] = source_vertex; // 0 = start_point

	int *c_vv2 = (int *)malloc(sizeof(int)*nv);
	memset(c_vv2, 0, sizeof(int)*nv);
	c_vv2[source_vertex] = 1;

	float *c_vv3 = (float *)malloc(sizeof(float)*nv);
	memset(c_vv3, 0, sizeof(int)*nv);


        int c_iter=0;
        while(1) {
                if(qhead == qtail) break;
                c_iter++;
                i = c_q[qhead];
                qhead++;
                for(int j=csc_v[i]; j<csc_v[i+1]; j++) {
                        int k = csc_e[j];
                        if(c_vv[k] == IUNUSED) {
                                c_vv[k] = c_vv[i]+1;
                                c_q[qtail] = k;
                                qtail++;
                        }
			if(c_vv[i] + 1 == c_vv[k]) {
				c_vv2[k] += c_vv2[i];
			}
                }
       }
	for(i=qtail-1; i>=0; i--) {
		for(int j=csc_v[c_q[i]]; j<csc_v[c_q[i]+1];j++) {
			int k = csc_e[j];
			if(c_vv[k] == c_vv[c_q[i]]-1) {
				c_vv3[k] += ((float)c_vv2[k]/c_vv2[c_q[i]])*(1.0f + c_vv3[c_q[i]]);
			}
		}	
	}

	


////fprintf(stdout, "***\n");
	int *_vv2 = out->r2;
	int *vv2 = (int *)malloc(sizeof(int)*nv);
	hipMemcpyAsync(vv2, _vv2, sizeof(float)*nv, hipMemcpyDeviceToHost);
	float *_vv3 = out->r3;
	float *vv3 = (float *)malloc(sizeof(float)*nv);
	hipMemcpyAsync(vv3, _vv3, sizeof(float)*nv, hipMemcpyDeviceToHost);

/*
//cpu code
	//vv1,vv2,vv3
	for(i=0;i<nv;i++) {
		int t = MAX(vv3[itable[i]],c_vv3[i]);
		if(t == 0) continue;
		if(abs(vv3[itable[i]] - c_vv3[i])/t > THR) {
			break;
//			printf("%d diff : %f %f\n", i, vv3[itable[i]], c_vv3[i]);
		}
	}

	if(i == nv) printf("PASS\n");
	else printf("FAIL\n");
*/
/*	FILE *fp2 = fopen("out2.txt", "w");
	for(i=0;i<nv;i++) {
		fprintf(fp2, "%d:\t%d\t%d\t%f\n", i, vv[itable[i]], vv2[itable[i]], vv3[itable[i]]);
	}
	fclose(fp2);*/
#endif
#ifdef CC
        int *c_q;
        c_q = (int *)malloc(sizeof(int)*nv);
	int kcnt=0;
	int *adjust_vv = (int *)malloc(sizeof(int)*nv);
	memset(adjust_vv, -1, sizeof(int)*nv);

int *temp_vv = (int *)malloc(sizeof(int)*nv);
for(i=0;i<nv;i++)
temp_vv[i] = vv[i];

	for(i=0;i<nv;i++) {
		if(adjust_vv[vv[i]] < 0) {
			adjust_vv[vv[i]] = kcnt;
			kcnt++;
		}
	}
	for(i=0;i<nv;i++) {
		vv[i] = adjust_vv[vv[i]];
	}
		

	for(i=0;i<nv;i++)
		c_vv[i] = i;


	while(1) {
		int pflag=0;
		for(i=0;i<nv;i++) {
			for(int j=csc_v[i];j<csc_v[i+1];j++) {
				if(c_vv[i] != c_vv[csc_e[j]]) {
					c_vv[i] = MIN(c_vv[i], c_vv[csc_e[j]]);
					c_vv[csc_e[j]] = MIN(c_vv[i], c_vv[csc_e[j]]);
					pflag=1;
				}
			}
		}
		if(pflag == 0) break;
	}
	
	memset(adjust_vv, -1, sizeof(int)*nv);

	kcnt= 0;
	for(i=0;i<nv;i++) {
		if(adjust_vv[c_vv[i]] < 0) {
			adjust_vv[c_vv[i]] = kcnt;
			kcnt++;
		}
	}
	for(i=0;i<nv;i++)
		c_vv[i] = adjust_vv[c_vv[i]];

//	memset(c_vv, -1, sizeof(int)*nv);
//	int kkcnt=0;
/*	while(1) {
		for(i=0;i<nv;i++) {
			if(c_vv[i] < 0) break;
		}
		if(i == nv) break;
	        int qhead=0, qtail=1;
	        c_vv[i] = kkcnt; c_q[0] = i;

	 	while(1) {
       	       		if(qhead == qtail) break;
       	         	i = c_q[qhead%nv];
       	         	qhead++;
                	for(int j=csc_v[i]; j<csc_v[i+1]; j++) {
                        	int k = csc_e[j];
                        	if(c_vv[k] < 0) {
                                	c_vv[k] = kkcnt;
                               		c_q[qtail%nv] = k;
                                	qtail++;
                        	}
                	}
        	}
		kkcnt++;
	}*/

	for(i=0;i<nv;i++) {
		if(vv[i] != c_vv[i]) break;
//		if(vv[i] != c_vv[i]) printf("%d %d %d %d\n", i, temp_vv[i], vv[i], c_vv[i]);
	}
	if(i == nv) fprintf(stdout, "validation : PASS\n");
	else fprintf(stdout, "validation : FAIL\n");

#endif

#if defined BFS || defined SSSP || defined BC
        for(i=0;i<nv;i++) {
            if(vv[itable[i]] != c_vv[i]) break;
//            if(vv[itable[i]] != c_vv[i]) printf("%d %d %d %d\n", i, itable[i], vv[itable[i]], c_vv[i]);
        }
        if(i == nv) fprintf(stdout, "validation : PASS\n");
        else fprintf(stdout, "validation : FAIL %d\n", i);
#endif



#ifdef PRINT_OUTPUT

	FILE *fp;
	fp = fopen("out.txt", "w");
	for(i=0;i<nv;i++) {
		fprintf(fp, "%d\t%f\n", i, vv[itable[i]]);
	} fclose(fp);

#endif
	

#else // VALIDATE
	printf("\n");
#endif // VALIDATE
}
