#include "hip/hip_runtime.h"
#include "user_function.h"
//#include "common.h"
#if defined BFS || defined CC || defined BC || defined SSSP

#define MODE_HYBRID
//#define TRACE

#if defined BFS || defined CC || defined BC 
	#define SAME_FRONTIER
#endif
#if defined SSSP
	#define POSITIVE_FRONTIER
#endif

/*
void initValue(struct HYB *m, int **vv) // int is dep
{
        hipMalloc((void **) vv, sizeof(int)*CEIL(m->nv,PSIZE)*PSIZE);
        hipMemset(*vv, UIN, sizeof(int)*CEIL(m->nv,PSIZE)*PSIZE);

	int *v = *vv;

        hipMemcpyAsync(&(m->start_point), &(m->_itable[0]), sizeof(int), hipMemcpyDeviceToHost);
        hipMemset(&v[m->start_point], 0, sizeof(int));
}

void initFrontier(struct HYB *m, int **curr_f, int **next_f, int **gl)
{
	hipMalloc((void **) curr_f, sizeof(int)*(m->nv)*SCATTER_FACTOR);
	hipMalloc((void **) next_f, sizeof(int)*(m->nv)*SCATTER_FACTOR);
	hipMalloc((void **) gl, sizeof(int)*SCATTER_FACTOR);

	int *p_curr_f = *curr_f;
	int *p_gl = *gl;
	int h_fs[SCATTER_FACTOR]={1}, htot_size = 1;
	
	int k = m->start_point;

//int *p_next_f=*next_f;
//hipMemset(p_curr_f, 0, sizeof(int)*(m->nv)*4);
//hipMemset(p_next_f, 0, sizeof(int)*(m->nv)*4);

//	hipMemcpyAsync(p_gl, h_fs, sizeof(int)*SCATTER_FACTOR, hipMemcpyHostToDevice);
	hipMemset(p_gl, 0, sizeof(int)*SCATTER_FACTOR);
	hipMemcpyAsync(p_curr_f, &k, sizeof(int), hipMemcpyHostToDevice);
}*/

__global__ void phase_sample(int *vv, float *sample_partial, int iter
#ifdef POSITIVE_FRONTIER
, int *vfc
#endif
)
{
	int index=blockIdx.x*blockDim.x+threadIdx.x;
#ifdef SAME_FRONTIER
	int val = vv[index];
#endif
#ifdef POSITIVE_FRONTIER
	int val = vfc[index];
#endif
	float fval;
	if(val == iter) fval = 1; else fval = 0;
        for(int offset = 16; offset > 0; offset = offset >> 1) {
                fval += __shfl_down(val, offset);
        }
        if(threadIdx.x == 0) sample_partial[blockIdx.x] = fval;
}

__global__ void phase_sample_reduction(int size, float *sample_partial)
{
	__shared__ float sval[32];

	float val;
	if(threadIdx.x < size) val = sample_partial[threadIdx.x];	
	else val = 0;
        for(int offset = 16; offset > 0; offset = offset >> 1) {
                val += __shfl_down(val, offset);
        }
	if((threadIdx.x&31) == 0) {
		sval[threadIdx.x>>5] = val;
	}
	__syncthreads();
	if(threadIdx.x < 32) {
		val = sval[threadIdx.x];
	        for(int offset = 16; offset > 0; offset = offset >> 1) {
                	val += __shfl_down(val, offset);
      		}
	}
	if(threadIdx.x == 0) {
		sample_partial[threadIdx.x] = val;
	}

}

__global__ void make_frontier(int nv, int iter, int *vv, int *f1, int *f2, int *f3, int *f4, int *pointer, int *vfc)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int warp_id = ((threadIdx.x>>5)&3);
	if(index < nv) {
#if defined SAME_FRONTIER
		if(vv[index] == iter) {
#endif
#if defined POSITIVE_FRONTIER
		if(vfc[index] == iter) {
#endif
			if(warp_id == 0) f1[atomicAggInc(&pointer[0])] = index;
			else if(warp_id == 1) f2[atomicAggInc(&pointer[1])] = index;
			else if(warp_id == 2) f3[atomicAggInc(&pointer[2])] = index;
			else f4[atomicAggInc(&pointer[3])] = index;
		}
	}
}

__global__ void pseudo_process_f1(int nv, int tf_size, int *csc_v, int *csc_e, int *gl, int nn1, int *f1, int *n_f1, int nn2, int *f2, int *n_f2, int nn3, int *f3, int *n_f3, int nn4, int *f4, int *n_f4, int *vv, int *nvv, int iter, int flag, int *vfc
#ifdef V2
, V2T *vv2, V2T *nvv2
#endif
#ifdef E1
, E1T *csc_ev
#endif
)
{
	__shared__ int buffer1[64], buffer2[64], buffer3[64], buffer4[64];
	__shared__ int buffer_p[2];

	int i, j;
	int base_addr = blockIdx.x*64 + (threadIdx.x>>2);
	int checker = (blockIdx.x & (ssample_factor-1));
	if(flag == 0 && checker > 0) return;
	else if(flag == 1 && checker == 0) return;

	int warp_id, index, index_size, bias;
	if(threadIdx.x < 2) {
		buffer_p[threadIdx.x] = 0;
	}
	__syncthreads();
		warp_id = ((threadIdx.x>>5)&3);
	if(base_addr < tf_size) {
		bias = 0;
		if(base_addr < nn1) index = f1[base_addr];
		else if(base_addr <nn1+nn2) index = f2[base_addr-nn1];
		else if(base_addr < nn1+nn2+nn3) index = f3[base_addr-nn1-nn2];
		else index = f4[base_addr-nn1-nn2-nn3];
			index_size = csc_v[index+1] - csc_v[index];
			if(index_size >= 32) {
			bias = index_size - (index_size&31);
			if((threadIdx.x&3) == 0) {
				int p = atomicAggInc(&buffer_p[0]);
				buffer1[p] = index;
				buffer2[p] = bias;
			}
			if(index_size >= 256) {
				if((threadIdx.x&3) == 0) {
					int p2 = atomicAggInc(&buffer_p[1]);
					buffer3[p2] = index;
					buffer4[p2] = index_size - (index_size&255);
				}
			}
		}
		for(i=bias+(threadIdx.x&3); i<index_size; i+=4) {
			int index_dst = csc_e[csc_v[index]+i];
			int tmp;
#ifdef V2
			V2T tmp2;
#endif
			get_partial_result(&tmp, vv[index]
#ifdef V2
			, &tmp2, vv2[index]
#endif
#ifdef E1
			, csc_ev[csc_v[index]+i]
#endif
			); // bfs can be optimized
			if(fused_update_condition(&vv[index_dst], &nvv[index_dst], tmp, iter, &vfc[index_dst]
#ifdef V2
			, &vv2[index_dst], &nvv2[index_dst], tmp2
#endif
			)) {
				if(warp_id == 0) n_f1[atomicAggInc(&gl[0])] = index_dst;
				else if(warp_id == 1) n_f2[atomicAggInc(&gl[1])] = index_dst;
				else if(warp_id == 2) n_f3[atomicAggInc(&gl[2])] = index_dst;
				else n_f4[atomicAggInc(&gl[3])] = index_dst;
			}
		}
	}
	__syncthreads();
	for(i=(threadIdx.x>>5);i<(buffer_p[0]>>1);i+=8) {
		index = buffer1[i];
		int bf2 = buffer2[i];
		int bf22 = bf2 - (bf2&255);
		for(j=bf22+(threadIdx.x&31);j<bf2;j+=32) {
			int index_dst = csc_e[csc_v[index]+j];
			int tmp;
#ifdef V2
			V2T tmp2;
#endif
			get_partial_result(&tmp, vv[index]
#ifdef V2
			, &tmp2, vv2[index]
#endif
#ifdef E1
			, csc_ev[csc_v[index]+j]
#endif
			); // bfs can be optimized
			if(fused_update_condition(&vv[index_dst], &nvv[index_dst], tmp, iter, &vfc[index_dst]
#ifdef V2
			, &vv2[index_dst], &nvv2[index_dst], tmp2
#endif
			)) {
				if(warp_id == 0) n_f1[atomicAggInc(&gl[0])] = index_dst;
				else if(warp_id == 1) n_f2[atomicAggInc(&gl[1])] = index_dst;
				else if(warp_id == 2) n_f3[atomicAggInc(&gl[2])] = index_dst;
				else n_f4[atomicAggInc(&gl[3])] = index_dst;
			}

		}
	}
	for(i=0;i<buffer_p[1];i++) {
		index = buffer3[i];
		for(j=threadIdx.x;j<(buffer4[i]>>1);j+=blockDim.x) {
			int index_dst = csc_e[csc_v[index]+j];
			int tmp;
#ifdef V2
			int tmp2;
#endif
			get_partial_result(&tmp, vv[index]
#ifdef V2
			, &tmp2, vv2[index]
#endif
#ifdef E1
			, csc_ev[csc_v[index]+j]
#endif
			); // bfs can be optimized
			if(fused_update_condition(&vv[index_dst], &nvv[index_dst], tmp, iter, &vfc[index_dst]
#ifdef V2
			, &vv2[index_dst], &nvv2[index_dst], tmp2
#endif
			)) {
				if(warp_id == 0) n_f1[atomicAggInc(&gl[0])] = index_dst;
				else if(warp_id == 1) n_f2[atomicAggInc(&gl[1])] = index_dst;
				else if(warp_id == 2) n_f3[atomicAggInc(&gl[2])] = index_dst;
				else n_f4[atomicAggInc(&gl[3])] = index_dst;
			}

		}
	}
}


// relaseValue can be added
__global__ void pseudo_phase1(int nv, int ne, int upper_ne, int np, short *sp1, int *vv, int *i1, int *ces, int *mapper
#ifdef V2
, V2T *vv2, V2T *i2
#endif
#ifdef E1
, E1T *sp1v
#endif
)
{
        __shared__ int sv[PSIZE];
#ifdef V2
	__shared__ V2T sv2[PSIZE];
#endif

	int sblock = blockIdx.x*dsample_factor+(dsample_factor>>1);
        int base_addr = blockIdx.x*PSIZE*dsample_factor;
        int i, index;
        short temp;

        for(i=threadIdx.x; i<PSIZE; i+=blockDim.x) {
                sv[i] = vv[i+base_addr];
#ifdef V2
		sv2[i] = vv2[i+base_addr];
#endif
        }
        __syncthreads();

        for(i=ces[sblock]+threadIdx.x; i<ces[sblock]+((ces[sblock+1]-ces[sblock])>>2); i+=blockDim.x) {
                index = mapper[i>>3]+(threadIdx.x&7);
                temp = sp1[i];
                if(temp < PSIZE) {
			////i1[index] = sv[temp]+1;
			get_partial_result(&i1[index], sv[temp]
#ifdef V2
			,&i2[index], sv2[temp]
#endif
#ifdef E1
			, sp1v[i]
#endif
			);
		}
        }
}

// DFACTOR, LOG_DFACTOR will be removed
__global__ void pseudo_phase11(int nv, int np, int *vv, int *i1, int *dx, int *dy, int *dindex, short *dp1, int *dmap, int iter
#ifdef V2
, V2T *vv2, V2T *i2
#endif
#ifdef E1
, E1T *dp1v
#endif
)
{
        __shared__ int sv[PSIZE];
#ifdef V2
	__shared__ V2T sv2[PSIZE];
#endif
	int sblock = blockIdx.x*dsample_factor+(dsample_factor>>1);

        int i;
        int x_base = dx[sblock];//, y_base = dy[blockIdx.x];
//if(x_base < 0 || x_base >= np || y_base < 0 || y_base >= np) printf("err0\n");
        short curr_index = SUNUSED;
        int curr_v, temp_v;
#ifdef V2
	V2T curr_v2, temp_v2;
#endif
	initialize(&curr_v, 0
#ifdef V2
	, &curr_v2, 0
#endif
	);

        int dmap_index = dmap[sblock];


        for(i=threadIdx.x;i<PSIZE;i+=blockDim.x) {
//if(x_base*PSIZE+i < 0 || x_base*PSIZE+i >= nv) printf("err1\n");
                sv[i] = vv[x_base*PSIZE+i];
#ifdef V2
		sv2[i] = vv2[x_base*PSIZE+i];
#endif
        }
        __syncthreads();

        for(i=dindex[sblock]+threadIdx.x;i<dindex[sblock]+((dindex[sblock+1]-dindex[sblock])>>2);i+=blockDim.x) {
                short edge_value = dp1[i];
                if(edge_value < 0) {
//if(edge_value < -1*PSIZE) printf("errxx\n");
                        if(curr_index != SUNUSED) {
				accumulate(&i1[dmap_index+curr_index], curr_v, iter
#ifdef V2
				, &i2[dmap_index+curr_index], curr_v2
#endif
				);
                        }
                        curr_index = -(edge_value+1);
			initialize(&curr_v, 0
#ifdef V2
			, &curr_v2, 0
#endif
			);
//                        curr_v = IUNUSED;
                } else if(edge_value != SUNUSED) {
			temp_v = sv[edge_value];
#ifdef V2
			temp_v2 = sv2[edge_value];
#endif
			int tmp;
#ifdef V2
			V2T tmp2;
#endif
			get_partial_result(&tmp, temp_v
#ifdef V2
			, &tmp2, temp_v2
#endif
#ifdef E1
			, dp1v[i]
#endif
			);
			accumulate_nonAtomics(&curr_v, tmp, iter
#ifdef V2
			, &curr_v2, tmp2
#endif
			);

                        //temp_v = sv[edge_value];
                        //if(temp_v + 1 < curr_v) curr_v = temp_v+1;
                }
        }
	if(curr_index != SUNUSED) {
		accumulate(&i1[dmap_index+curr_index], curr_v, iter
#ifdef V2
		, &i2[dmap_index+curr_index], curr_v2
#endif
		);
	}
        //if(curr_index != SUNUSED && curr_v < i1[dmap_index + curr_index])
        //        i1[dmap_index+curr_index] = curr_v;

}

//DFAFCTOR will be removed
__global__ void pseudo_phase2(int nv, int ne, int np, short *sp2, int *vv, int *dummy_vv, int *i1, int *ces, int *finished, int iter, int *vfc
#ifdef V2
, V2T *vv2, V2T *nvv2, V2T *i2 
#endif
)
{
        __shared__ int sv[PSIZE];
#ifdef V2
	__shared__ V2T sv2[PSIZE];
#endif

	int sblock = blockIdx.x*dsample_factor+(dsample_factor>>1);
        int base_addr = sblock*PSIZE;
        int i;

        for(i=threadIdx.x; i<PSIZE; i+=blockDim.x) {
		initialize(&sv[i], IUNUSED
#ifdef V2
		, &sv2[i], 0
#endif
		);
                ////sv[i] = IUNUSED;
        }
        __syncthreads();

//if(cpb2[fac*blockIdx.x] < 0 || cpb2[fac*(blockIdx.x+1)] > 1648304) printf("errr %d %d %d\n", blockIdx.x, cpb2[fac*blockIdx.x], cpb2[fac*(blockIdx.x+1)]);

        for(i=ces[sblock]+threadIdx.x;i<ces[sblock]+((ces[sblock+1]-ces[sblock])>>2); i+=blockDim.x) {
                int ii=sp2[i];
                ////if(i1[i] != IUNUSED && ii < PSIZE && sv[ii] > i1[i])
                ////     sv[ii] = i1[i];
		if(i1[i] != IUNUSED && ii < PSIZE) {
#ifdef BFS
	                if(i1[i] <= iter && i1[i] < sv[ii]) {
	                        sv[ii] = i1[i];
	                }
#endif
#ifdef SSSP
	                atomicMin(&sv[ii], i1[i]);
#endif
#ifdef BC
	                if(i1[i] <= iter && i1[i] < sv[ii]) {
	                        sv[ii] = i1[i];
	                }
	                if((i1[i] == iter) && (sv[ii] == BFS_INF || sv[ii] == iter)) {
	                        atomicAdd(&sv2[ii], i2[i]);
	                }
#endif

/*			accumulate(&sv[ii], i1[i], iter
#ifdef V2
		, &sv2[ii], i2[i]
#endif
			);*/
		}
        }
        __syncthreads();


        int flag = false;
        for(i=threadIdx.x; i<PSIZE; i+=blockDim.x) {
#ifdef BC
                if(sv[i] < vv[i+base_addr]) {
                        dummy_vv[i+base_addr] = sv[i];
//                        vv2[i+base_addr] = sv2[i];
                        flag = true;
                }
#endif
#ifdef SSSP
                if(sv[i] < vv[i+base_addr]) {
                        dummy_vv[i+base_addr] = sv[i];
                        flag = true;
                }
#endif
#ifdef BFS
                if(sv[i] < vv[i+base_addr]) {
                        dummy_vv[i+base_addr] = sv[i];
                        flag = true;
                }
#endif

/*		flag = update_condition(&vv[i+base_addr], &dummy_vv[i+base_addr], sv[i], iter, &vfc[i+base_addr]
#ifdef V2
		, &vv2[i+base_addr], &nvv2[i+base_addr], sv2[i]
#endif
		);*/


        }

        if(__syncthreads_or(flag)) {
                if(threadIdx.x == 0) {
                        (*finished) = 1;
                }
        }
}

__global__ void process_f1(int nv, int tf_size, int *csc_v, int *csc_e, int *gl, int nn1, int *f1, int *n_f1, int nn2, int *f2, int *n_f2, int nn3, int *f3, int *n_f3, int nn4, int *f4, int *n_f4, int *vv, int *nvv, int iter, int flag, int *vfc
#ifdef V2
, V2T *vv2, V2T *nvv2
#endif
#ifdef E1
, E1T *csc_ev
#endif
)
{
	__shared__ int buffer1[64], buffer2[64], buffer3[64], buffer4[64];
	__shared__ int buffer_p[2];

	int i, j;
	int base_addr = blockIdx.x*64 + (threadIdx.x>>2);
//	int checker = (blockIdx.x & (sample_factor-1));
//	if(flag == 0 && checker > 0) return;
//	else if(flag == 1 && checker == 0) return;

	int warp_id, index, index_size, bias;
	if(threadIdx.x < 2) {
		buffer_p[threadIdx.x] = 0;
	}
	__syncthreads();
		warp_id = ((threadIdx.x>>5)&3);
	if(base_addr < tf_size) {
		bias = 0;
		if(base_addr < nn1) index = f1[base_addr];
		else if(base_addr <nn1+nn2) index = f2[base_addr-nn1];
		else if(base_addr < nn1+nn2+nn3) index = f3[base_addr-nn1-nn2];
		else index = f4[base_addr-nn1-nn2-nn3];
			index_size = csc_v[index+1] - csc_v[index];
			if(index_size >= 32) {
			bias = index_size - (index_size&31);
			if((threadIdx.x&3) == 0) {
				int p = atomicAggInc(&buffer_p[0]);
				buffer1[p] = index;
				buffer2[p] = bias;
			}
			if(index_size >= 256) {
				if((threadIdx.x&3) == 0) {
					int p2 = atomicAggInc(&buffer_p[1]);
					buffer3[p2] = index;
					buffer4[p2] = index_size - (index_size&255);
				}
			}
		}
		for(i=bias+(threadIdx.x&3); i<index_size; i+=4) {
			int index_dst = csc_e[csc_v[index]+i];
			int tmp;
#ifdef V2
			V2T tmp2;
#endif
			get_partial_result(&tmp, vv[index]
#ifdef V2
			, &tmp2, vv2[index]
#endif
#ifdef E1
			, csc_ev[csc_v[index]+i]
#endif
			); // bfs can be optimized
			if(fused_update_condition(&vv[index_dst], &vv[index_dst], tmp, iter, &vfc[index_dst]
#ifdef V2
			, &vv2[index_dst], &vv2[index_dst], tmp2
#endif
			)) {
				if(warp_id == 0) n_f1[atomicAggInc(&gl[0])] = index_dst;
				else if(warp_id == 1) n_f2[atomicAggInc(&gl[1])] = index_dst;
				else if(warp_id == 2) n_f3[atomicAggInc(&gl[2])] = index_dst;
				else n_f4[atomicAggInc(&gl[3])] = index_dst;
			}

			//if(vv[index_dst] == IUNUSED) {
			//	nvv[index_dst] = iter;
			//	if(warp_id == 0) n_f1[atomicAggInc(&gl[0])] = index_dst;
			//	else if(warp_id == 1) n_f2[atomicAggInc(&gl[1])] = index_dst;
			//	else if(warp_id == 2) n_f3[atomicAggInc(&gl[2])] = index_dst;
			//	else n_f4[atomicAggInc(&gl[3])] = index_dst;
			//}
		}
	}
	__syncthreads();
	for(i=(threadIdx.x>>5);i<buffer_p[0];i+=8) {
		index = buffer1[i];
		int bf2 = buffer2[i];
		int bf22 = bf2 - (bf2&255);
		for(j=bf22+(threadIdx.x&31);j<bf2;j+=32) {
			int index_dst = csc_e[csc_v[index]+j];
//if(index_dst < 0 || index_dst >= nv) printf("err\n");
			int tmp;
#ifdef V2
			V2T tmp2;
#endif
			get_partial_result(&tmp, vv[index]
#ifdef V2
			, &tmp2, vv2[index]
#endif
#ifdef E1
			, csc_ev[csc_v[index]+j]
#endif
			); // bfs can be optimized
			if(fused_update_condition(&vv[index_dst], &vv[index_dst], tmp, iter, &vfc[index_dst]
#ifdef V2
			, &vv2[index_dst], &vv2[index_dst], tmp2
#endif
			)) {
				if(warp_id == 0) n_f1[atomicAggInc(&gl[0])] = index_dst;
				else if(warp_id == 1) n_f2[atomicAggInc(&gl[1])] = index_dst;
				else if(warp_id == 2) n_f3[atomicAggInc(&gl[2])] = index_dst;
				else n_f4[atomicAggInc(&gl[3])] = index_dst;
			}

			//if(vv[index_dst] == IUNUSED) {
			//	nvv[index_dst] = iter;
			//	if(warp_id == 0) n_f1[atomicAggInc(&gl[0])] = index_dst;
			//	else if(warp_id == 1) n_f2[atomicAggInc(&gl[1])] = index_dst;
			//	else if(warp_id == 2) n_f3[atomicAggInc(&gl[2])] = index_dst;
			//	else n_f4[atomicAggInc(&gl[3])] = index_dst;
			//}	
		}
	}
	for(i=0;i<buffer_p[1];i++) {
		index = buffer3[i];
		for(j=threadIdx.x;j<buffer4[i];j+=blockDim.x) {
			int index_dst = csc_e[csc_v[index]+j];
//if(index_dst < 0 || index_dst >= nv) printf("err\n");
			int tmp;
#ifdef V2
			V2T tmp2;
#endif
			get_partial_result(&tmp, vv[index]
#ifdef V2
			, &tmp2, vv2[index]
#endif
#ifdef E1
			, csc_ev[csc_v[index]+j]
#endif
			); // bfs can be optimized
			if(fused_update_condition(&vv[index_dst], &vv[index_dst], tmp, iter, &vfc[index_dst]
#ifdef V2
			, &vv2[index_dst], &vv2[index_dst], tmp2
#endif
			)) {
				if(warp_id == 0) n_f1[atomicAggInc(&gl[0])] = index_dst;
				else if(warp_id == 1) n_f2[atomicAggInc(&gl[1])] = index_dst;
				else if(warp_id == 2) n_f3[atomicAggInc(&gl[2])] = index_dst;
				else n_f4[atomicAggInc(&gl[3])] = index_dst;
			}

			//if(vv[index_dst] == IUNUSED) {
			//	nvv[index_dst] = iter;
			//	       if(warp_id == 0) n_f1[atomicAggInc(&gl[0])] = index_dst;
			//	       else if(warp_id == 1) n_f2[atomicAggInc(&gl[1])] = index_dst;
			//	       else if(warp_id == 2) n_f3[atomicAggInc(&gl[2])] = index_dst;
			//	       else n_f4[atomicAggInc(&gl[3])] = index_dst;
			//}
		}
	}
}


	// relaseValue can be added
__global__ void phase1(int nv, int ne, int upper_ne, int np, short *sp1, int *vv, int *i1, int *ces, int *mapper
#ifdef V2
, V2T *vv2, V2T *i2
#endif
#ifdef E1
, E1T *sp1v
#endif
)
{
	__shared__ int sv[PSIZE];
#ifdef V2
	__shared__ V2T sv2[PSIZE];
#endif

//	if(sampled == 1 && (blockIdx.x & (ssample-1)) == 0) return;

	int base_addr = blockIdx.x*PSIZE;
	int i, index;
	short temp;
	for(i=threadIdx.x; i<PSIZE; i+=blockDim.x) {
		sv[i] = vv[i+base_addr];
#ifdef V2
		sv2[i] = vv2[i+base_addr];
#endif
	}
	__syncthreads();
//printf("(%d %d)\n", ces[blockIdx.x], ces[blockIdx.x+1]);
	for(i=ces[blockIdx.x]+threadIdx.x; i<ces[blockIdx.x+1]; i+=blockDim.x) {
//printf("(%d)\n", i);
		index = mapper[i>>3]+(threadIdx.x&7);
		temp = sp1[i];
		if(temp < PSIZE) {
//#ifdef BFS
//			i1[index] = sv[temp]+1;
//#endif
			////i1[index] = sv[temp]+1;
			get_partial_result(&i1[index], sv[temp]
#ifdef V2
			, &i2[index], sv2[temp]
#endif
#ifdef E1
			, sp1v[i]
#endif
			);

		}
	}
}
	// DFACTOR, LOG_DFACTOR will be removed
__global__ void phase11(int nv, int np, int *vv, int *i1, int *dx, int *dy, int *dindex, short *dp1, int *dmap, int iter
#ifdef V2
, V2T *vv2, V2T *i2
#endif
#ifdef E1
, E1T *dp1v
#endif
)
{
	__shared__ int sv[PSIZE];
#ifdef V2
	__shared__ V2T sv2[PSIZE];
#endif

	int i;
	int x_base = dx[blockIdx.x];//, y_base = dy[blockIdx.x];

//	if(sampled == 1 && (x_base & (ssample-1)) == 0) return;

//if(x_base < 0 || x_base >= np || y_base < 0 || y_base >= np) printf("err0\n");
	short curr_index = SUNUSED;
	int curr_v, temp_v;
#ifdef V2
	V2T curr_v2, temp_v2;
#endif
	initialize(&curr_v, 0
#ifdef V2
	, &curr_v2, 0
#endif
	);

	int dmap_index = dmap[blockIdx.x];
	for(i=threadIdx.x;i<PSIZE;i+=blockDim.x) {
//if(x_base*PSIZE+i < 0 || x_base*PSIZE+i >= nv) printf("err1\n");
		sv[i] = vv[x_base*PSIZE+i];
#ifdef V2
		sv2[i] = vv2[x_base*PSIZE+i];
#endif
	}
	__syncthreads();

	for(i=dindex[blockIdx.x]+threadIdx.x;i<dindex[blockIdx.x+1];i+=blockDim.x) {
		short edge_value = dp1[i];
		if(edge_value < 0) {
//if(edge_value < -1*PSIZE) printf("errxx\n");
			if(curr_index != SUNUSED) {
/*
#ifdef BFS
				if(curr_v <= iter && curr_v < i1[dmap_index+curr_index]) {
					i1[dmap_index+curr_index] = curr_v;
				}
#endif
#ifdef SSSP
				atomicMin(&i1[dmap_index+curr_index], curr_v);
#endif
#ifdef BC
				if(curr_v <= iter && curr_v < i1[dmap_index+curr_index]) {
					i1[dmap_index+curr_index] = curr_v;
				}
				if((curr_v == iter && (i1[dmap_index+curr_index] == BFS_INF || i1[dmap_index+curr_index] == iter))) {
					atomicAdd(&i2[dmap_index+curr_index], curr_v2);
				}
#endif
*/

				accumulate(&i1[dmap_index+curr_index], curr_v, iter
#ifdef V2
				, &i2[dmap_index+curr_index], curr_v2
#endif
				);
			}
			curr_index = -(edge_value+1);
//if(curr_index < 0 || curr_index >= PSIZE) printf("erryy\n");
			initialize(&curr_v, 0
#ifdef V2
			, &curr_v2, 0
#endif
			);
//			curr_v = IUNUSED;
		} else if(edge_value != SUNUSED) {
			temp_v = sv[edge_value];
#ifdef V2
			temp_v2 = sv2[edge_value];
#endif
			int tmp;
#ifdef V2
			V2T tmp2;
#endif
			get_partial_result(&tmp, temp_v
#ifdef V2
			, &tmp2, temp_v2
#endif
#ifdef E1
			, dp1v[i]
#endif
			);
			accumulate_nonAtomics(&curr_v, tmp, iter
#ifdef V2
			, &curr_v2, tmp2
#endif
			);
			//if(temp_v + 1 < curr_v) curr_v = temp_v+1;
		}
	}
	if(curr_index != SUNUSED) {
		accumulate(&i1[dmap_index+curr_index], curr_v, iter
#ifdef V2
		, &i2[dmap_index+curr_index], curr_v2
#endif
		);
	}
	
	//if(curr_index != SUNUSED && curr_v < i1[dmap_index + curr_index] && curr_v <= iter)
	//	i1[dmap_index+curr_index] = curr_v;
}

	//DFAFCTOR will be removed
__global__ void phase2(int nv, int ne, int np, short *sp2, int *vv, int *i1, int *ces, int *finished, int iter, int *vfc
#ifdef V2
, V2T *vv2, V2T *i2
#endif
)
{
	__shared__ int sv[PSIZE];
#ifdef V2
	__shared__ V2T sv2[PSIZE];
#endif

	int base_addr = blockIdx.x*PSIZE;
	int i;

        for(i=threadIdx.x; i<PSIZE; i+=blockDim.x) {
                ////sv[i] = IUNUSED;
#ifdef BFS
//		sv[i] = BFS_INF;
#endif

		initialize(&sv[i], IUNUSED
#ifdef V2
		, &sv2[i], 0
#endif
		);

        }
        __syncthreads();

//if(cpb2[fac*blockIdx.x] < 0 || cpb2[fac*(blockIdx.x+1)] > 1648304) printf("errr %d %d %d\n", blockIdx.x, cpb2[fac*blockIdx.x], cpb2[fac*(blockIdx.x+1)]);

        for(i=ces[blockIdx.x]+threadIdx.x;i<ces[blockIdx.x+1]; i+=blockDim.x) {
                int ii=sp2[i];
/*
#ifdef BFS
		if(i1[i] < IUNUSED && ii < PSIZE) {
			sv[ii] = MIN(sv[ii], i1[i]);
		}
#endif
*/

		if(i1[i] != IUNUSED && ii < PSIZE) {
/*			accumulate(&sv[ii], i1[i], iter
#ifdef V2
			, &sv2[ii], i2[i]
#endif
			);
//printf("(%d %d)", i1[i], sv2[ii]);*/
#ifdef BFS
		if(i1[i] <= iter && i1[i] < sv[ii]) {
			sv[ii] = i1[i];
		}
#endif
#ifdef SSSP
		atomicMin(&sv[ii], i1[i]);
#endif
#ifdef BC
		if(i1[i] <= iter && i1[i] < sv[ii]) {
			sv[ii] = i1[i];
		}
		if((i1[i] == iter) && (sv[ii] == BFS_INF || sv[ii] == iter)) {
			atomicAdd(&sv2[ii], i2[i]);
		}
#endif
		}

        }
        __syncthreads();


        int flag = false;
        for(i=threadIdx.x; i<PSIZE; i+=blockDim.x) {

#ifdef BC
		if(sv[i] < vv[i+base_addr]) {
			vv[i+base_addr] = sv[i];
			vv2[i+base_addr] = sv2[i];
			flag = true;
		}
#endif
#ifdef SSSP
		if(sv[i] < vv[i+base_addr]) {
			vv[i+base_addr] = sv[i];
			flag = true;
#ifndef DENSE_MODE
			vfc[i+base_addr] = iter;
#endif
		}
#endif
#ifdef BFS
		if(sv[i] < vv[i+base_addr]) {
			vv[i+base_addr] = sv[i];
			flag = true;
		}
#endif

/*
		flag = update_condition(&vv[i+base_addr], &vv[i+base_addr], sv[i], iter, &vfc[i+base_addr]
#ifdef V2
		, &vv2[i+base_addr], &vv2[i+base_addr], sv2[i]
#endif
		);
*/
                ////if(vv[i+base_addr] > sv[i]) {
                ////        vv[i+base_addr] = sv[i];
                ////        flag = true;
                ////}
        }
//if(flag == true) printf("true %d\n", threadIdx.x);
        if(__syncthreads_or(flag)) {
                if(threadIdx.x == 0) {
                        (*finished) = 1;
                }
        }

}


//__global__ vv_kernel<<<(nv+255)>>8, 256>>>(m->nv, m->_vv, m->_finished); 


__global__ void vv_kernel(int nv, int htot_size, int nn1, int *f1, int *n_f1, int nn2, int *f2, int *n_f2, int nn3, int *f3, int *n_f3, int nn4, int *f4, int *n_f4, int *vv, int *nvv, int iter, int *gl)
{
	int base_addr = blockIdx.x*256 + threadIdx.x;
	int warp_id = ((threadIdx.x>>5)&3);
	int index;
	int dummy;
	
	if(base_addr < htot_size) {
		if(base_addr < nn1) index = f1[base_addr];
		else if(base_addr < nn1+nn2) index = f2[base_addr-nn1];
		else if(base_addr < nn1+nn2+nn3) index = f3[base_addr-nn1-nn2];
		else index = f4[base_addr-nn1-nn2-nn3];

//		if(index < 0 || index >= nv) printf("err1 : %d\n", index);
//		if(vv[index] < 0 || vv[index] >= nv) printf("err2 : %d\n", vv[index]);

		if(update_condition(&vv[index], &vv[index], dummy, dummy, &dummy
#ifdef V2
		,&vv[vv[index]], &vv[vv[index]], dummy
#endif
		)) {
	                if(warp_id == 0) n_f1[atomicAggInc(&gl[0])] = index;
                        else if(warp_id == 1) n_f2[atomicAggInc(&gl[1])] = index;
                        else if(warp_id == 2) n_f3[atomicAggInc(&gl[2])] = index;
                        else n_f4[atomicAggInc(&gl[3])] = index;
		}
	}
}

__global__ void ee_kernel(int nv, int ne, int htot_size, int *v1, int *v2, int nn1, int *f1, int *n_f1, int nn2, int *f2, int *n_f2, int nn3, int *f3, int *n_f3, int nn4, int *f4, int *n_f4, int *vv, int *nvv, int iter, int *gl)
{
	int i,j, index;
	int base_addr = blockIdx.x*256 + threadIdx.x;
	int warp_id = ((threadIdx.x>>5)&3);
	int *dummy;

	if(base_addr < htot_size) {
		if(base_addr < nn1) index = f1[base_addr];
		else if(base_addr < nn1+nn2) index = f2[base_addr-nn1];
		else if(base_addr < nn1+nn2+nn3) index = f3[base_addr-nn1-nn2];
		else index = f4[base_addr-nn1-nn2-nn3];

		int src_value, dst_value;
		int index1=v1[index], index2=v2[index];
//		if(index1 < 0 || index1 >= nv || index2 < 0 || index2 >= nv) printf("err01 : %d %d\n", index1, index2);
		int val1 = vv[index1], val2 = vv[index2];
//		if(val1 < 0 || val1 >= nv || val2 < 0 || val2 >= nv) printf("err02 : %d %d\n", index1, index2);

		if(fused_update_condition(&vv[val1], &nvv[val1], val1, iter, dummy
#ifdef V2
		,&vv[val2], &nvv[val2], val2
#endif
		)) {
	                if(warp_id == 0) n_f1[atomicAggInc(&gl[0])] = index;
                        else if(warp_id == 1) n_f2[atomicAggInc(&gl[1])] = index;
                        else if(warp_id == 2) n_f3[atomicAggInc(&gl[2])] = index;
                        else n_f4[atomicAggInc(&gl[3])] = index;
		}
	}
}

__global__ void cudaMemcpydd(int htot_size, int b_p, int *accum_fr, int nn1, int *f1, int nn2, int *f2, int nn3, int *f3, int nn4, int *f4)
{
        int base_addr = blockIdx.x*blockDim.x+threadIdx.x;

        if(base_addr < nn1) accum_fr[b_p+base_addr] = f1[base_addr]; // 4 atomics
        else if(base_addr < nn1+nn2) accum_fr[b_p+base_addr] = f2[base_addr-nn1];
        else if(base_addr < nn1+nn2+nn3) accum_fr[b_p+base_addr] = f3[base_addr-nn1-nn2];
        else if(base_addr < htot_size) accum_fr[b_p+base_addr] = f4[base_addr-nn1-nn2-nn3];
}


int MultiGraph_V_V(struct MULTI_SPARSE *m, struct vector_data *vd0, struct vector_data *vd, struct OUTPUT *out, int *g_iter)
{
	int iter=*g_iter;
	int ne = m->ne, nv = m->nv;
	int *_temp_f;

//	int finished = 0;
//	cudaMemcpyAync(m->_finished, &finished, sizeof(int), hipMemcpyHostToDevice);
//	vv_kernel<<<(nv+255)>>8, 256>>>(m->nv, m->_vv, m->_finished); 


	hipMemset(vd->_gl, 0, sizeof(int)*SCATTER_FACTOR);

//int *tt;
//INT_PRINT(vd->_curr_f, 20);

	vv_kernel<<<(vd->htot_size+255)>>8, 256>>>(m->nv, vd->htot_size, vd->h_fs[0], &(vd->_curr_f[0]), &(vd->_next_f[0]),
	vd->h_fs[1], &(vd->_curr_f[nv]), &(vd->_next_f[nv]), vd->h_fs[2], &(vd->_curr_f[nv*2]), &(vd->_next_f[nv*2]),
	vd->h_fs[3], &(vd->_curr_f[nv*3]), &(vd->_next_f[nv*3]), vd0->_vv, vd0->_vv, iter, vd->_gl);


//INT_PRINT(vd->_curr_f, 20);

	hipMemcpyAsync(vd->h_fs, vd->_gl, sizeof(int)*SCATTER_FACTOR, hipMemcpyDeviceToHost);
        vd->htot_size = (vd->h_fs[0])+(vd->h_fs[1])+(vd->h_fs[2])+(vd->h_fs[3]);
	if ((vd->htot_size) == 0) { out->r = (vd->_vv); return false; }

//printf("%d %d %d %d %d\n", vd->h_fs[0], vd->h_fs[1], vd->h_fs[2], vd->h_fs[3], vd->htot_size);

	*g_iter = iter+1;
	_temp_f = (vd->_curr_f); (vd->_curr_f) = (vd->_next_f); (vd->_next_f) = _temp_f;



	out->r = (vd->_vv); //nvv (swap)
	return true;
}


int MultiGraph_E_E(struct MULTI_SPARSE *m, struct vector_data *vd, struct OUTPUT *out, int *g_iter)
{
	int iter=*g_iter;
	int ne = m->ne;
	int *_temp_f;	
	int *tt;

	hipMemset(vd->_gl, 0, sizeof(int)*SCATTER_FACTOR);

//INT_PRINT(vd->_vv, 50);

	ee_kernel<<<(vd->htot_size+255)>>8, 256>>>(m->nv, m->ne, vd->htot_size, m->_p1, m->_p2, vd->h_fs[0], &(vd->_curr_f[0]), &(vd->_next_f[0]),
	vd->h_fs[1], &(vd->_curr_f[ne]), &(vd->_next_f[ne]), vd->h_fs[2], &(vd->_curr_f[ne+(ne>>1)]), &(vd->_next_f[ne+(ne>>1)]),
	vd->h_fs[3], &(vd->_curr_f[ne*2]), &(vd->_next_f[ne*2]), vd->_vv, vd->_vv, iter, vd->_gl);

	hipMemcpyAsync(vd->h_fs, vd->_gl, sizeof(int)*SCATTER_FACTOR, hipMemcpyDeviceToHost);
        vd->htot_size = (vd->h_fs[0])+(vd->h_fs[1])+(vd->h_fs[2])+(vd->h_fs[3]);
	if ((vd->htot_size) == 0) { out->r = (vd->_vv); return false; }

//INT_PRINT(vd->_vv, 50);

	*g_iter = iter+1;
	_temp_f = (vd->_curr_f); (vd->_curr_f) = (vd->_next_f); (vd->_next_f) = _temp_f;

	out->r = (vd->_vv); //nvv (swap)
	return true;
}

int MultiGraph_V_E_V(struct HYB *m, struct csc_package *inp, struct vector_data *vd, struct OUTPUT *out, int *g_iter) // result : dep
{

	int iter=*g_iter;
	int finished=0;
	int palgo = vd->algo;

//	int *_vv=vd->_vv;
//	int *_curr_f=vd->_curr_f, *_next_f=vd->_next_f, *_gl=vd->_gl;
	int *_temp_f;
//	int *h_fs=vd->h_fs;

//	initValue(m, &_vv);

//	initFrontier(m, &(vd->_curr_f), &_next_f, &_gl);

//	hipMalloc((void **) &_finished, sizeof(int));

	// sampling
//        hipStream_t stream1=m->stream1, stream2=m->stream2;

#ifdef MODE_HYBRID

//	int *_dummy_vv=vd->_nvv;

	int d_threshold=IUNUSED, sampled = 0; // if added

	// add variables
	float sample_partial;
	//int *_temp_front;
	//hipMalloc((void **) &_temp_front, sizeof(int)*SCATTER_FACTOR);
	//hipMemset(_temp_front, 0, sizeof(int)*SCATTER_FACTOR);
#if defined SPARSE_MODE || defined DENSE_MODE
m->d_time = 100;
#endif

	if(m->d_time < 0) {
		hipDeviceSynchronize();
		double sample_start = rtclock(); // start time

		pseudo_phase1<<<m->dsample_size,BSIZE,0,m->stream1>>>((m->nv), (m->ne), (m->upper_ne), (m->np), (m->_sp1), (vd->_vv), (m->_i1), (m->_ces), (m->_mapper)
#ifdef V2
		, (vd->_vv2), (m->_i2)
#endif
#ifdef E1
		, (m->_sp1v)
#endif
		);
		pseudo_phase11<<<m->dsample_dcnt,BSIZE,0,m->stream2>>>((m->nv), (m->np), (vd->_vv), (m->_i1), (m->_dx), (m->_dy), (m->_dccindex), (m->_dp1), (m->_dmap), iter
#ifdef V2
		, (vd->_vv2), (m->_i2)
#endif
#ifdef E1
		, (m->_dp1v)
#endif
		);
		pseudo_phase2<<<m->dsample_size,BSIZE>>>((m->nv), (m->ne), (m->np), (m->_sp2), (vd->_vv), vd->_nvv, (m->_i1), (m->_cey), m->_finished, iter, (vd->_vfc)
#ifdef V2
		, (vd->_vv2), (vd->_nvv2), (m->_i2)
#endif
		);

		hipDeviceSynchronize();
		double sample_end = rtclock();
		(m->d_time) = (sample_end - sample_start) * dsample_factor * 4;
#ifdef TRACE
		fprintf(stdout, "sample : %f\n", 1000* ( m->d_time));
#endif
	}

#ifdef SPARSE_MODE
m->fflag = 2;
vd->algo = SPARSE;
#endif
#ifdef DENSE_MODE
m->fflag = 2;
vd->algo = DENSE;
#endif

//m->fflag = 2;
////m->fflag = 0;vd->algo = DENSE;

	if(m->fflag < 2) {
//	while(1) {
		if(vd->fhave == 0) { // don't have frontiter
//printf("should not happen\n");
			phase_sample<<<(m->mp)<<4,32>>>((vd->_vv), m->_sample_partial, iter-1
#ifdef POSITIVE_FRONTIER
			, (vd->_vfc)
#endif
			); //assuem dataset is not very small
			phase_sample_reduction<<<1,1024>>>((m->mp)<<4, m->_sample_partial);
			hipMemcpyAsync(&sample_partial, m->_sample_partial, sizeof(float), hipMemcpyDeviceToHost);	
			(vd->htot_size) = (int)(sample_partial * (m->nv) / ((m->mp)<<9));
			if((vd->htot_size) < ((m->nv)>>DENSE_THRESHOLD)) vd->algo = SPARSE;
//			else if((vd->htot_size) > d_threshold) algo = DENSE;
			else vd->algo = NOT_DETERMINED;
		} else {
			if((vd->htot_size) < ((m->nv)>>DENSE_THRESHOLD)) vd->algo = SPARSE;
////			else if((vd->htot_size) > d_threshold) algo = DENSE;
			else vd->algo = NOT_DETERMINED;
		}

//if(algo == NOT_DETERMINED) algo = DENSE;
//vd->algo = NOT_DETERMINED; // will be removed
//vd->algo = SPARSE;
//vd->algo = DENSE;
//vd->algo = DENSE;
		if(vd->algo == NOT_DETERMINED) {
			sampled = 1;
			// accumulate results
			if(vd->fhave == 0) {
				hipMemset(m->_temp_front, 0, sizeof(int)*SCATTER_FACTOR);

				vd->fhave = 1;
				int grid_size = (((m->nv)+127)>>7); // sampling can be do(m->ne)
				make_frontier<<<grid_size, 128>>>((m->nv), iter-1, (vd->_vv), &(vd->_curr_f[0]), &(vd->_curr_f[(m->nv)]), &(vd->_curr_f[(m->nv)+((m->nv)>>1)]), &(vd->_curr_f[(m->nv)*2]), m->_temp_front, (vd->_vfc));
				hipMemcpyAsync(vd->h_fs, m->_temp_front, sizeof(int)*SCATTER_FACTOR, hipMemcpyDeviceToHost);
				(vd->htot_size) = (vd->h_fs[0])+(vd->h_fs[1])+(vd->h_fs[2])+(vd->h_fs[3]);
			}

#if defined BFS || defined SSSP || defined CC || defined BC 
			hipMemset(vd->_gl, 0, sizeof(int)*SCATTER_FACTOR);
#endif
			hipDeviceSynchronize();
			double sample_start2 = rtclock();
			pseudo_process_f1<<<((vd->htot_size)+255)>>6, 256>>>((m->nv), (vd->htot_size), (m->_ncsc_v), (m->_ncsc_e), vd->_gl, vd->h_fs[0], &(vd->_curr_f[0]), &(vd->_next_f[0]),
			vd->h_fs[1], &(vd->_curr_f[(m->nv)]), &(vd->_next_f[(m->nv)]), vd->h_fs[2], &(vd->_curr_f[(m->nv)+((m->nv)>>1)]), &(vd->_next_f[(m->nv)+((m->nv)>>1)]), vd->h_fs[3], &(vd->_curr_f[(m->nv)*2]),
#if defined BFS || defined CC
			&(vd->_next_f[(m->nv)*2]), (vd->_vv), (vd->_vv), iter, 0, (vd->_vfc)
#endif
#if defined BC
			&(vd->_next_f[(m->nv)*2]), (vd->_vv), (vd->_nvv), iter, 0, (vd->_vfc)
#endif
#if defined SSSP
			&(vd->_next_f[(m->nv)*2]), (vd->_vv), (vd->_nvv), iter, 0, (vd->_vfc)
#endif
#ifdef V2
			, vd->_vv2, vd->_nvv2
#endif
#ifdef E1
			, (m->_ncsc_ev)
#endif
			);
#if defined BC 
			hipMemset(vd->_gl, 0, sizeof(int)*SCATTER_FACTOR);
#endif

			hipDeviceSynchronize();
			double sample_end2 = rtclock();
			double s_time = (sample_end2 - sample_start2) * ssample_factor * 1;
			if((m->d_time) > s_time) vd->algo = SPARSE;
			else vd->algo = DENSE;

#ifdef TRACE
printf("%f %f ", s_time*1000, (m->d_time)*1000);
#endif
//sampled = 2;
		} else {
			sampled = 2;	
		} 

//printf("%d %d\n", iter, algo);
//if(d_threshold < IUNUSED) algo = DENSE;
//int kp = rand()%2;

//vd->algo = DENSE;
	}

//int kp;
//if((kp&1) == 0) vd->algo = SPARSE;
//else vd->algo = DENSE;
//vd->algo = SPARSE;
//m->fflag = 0;


		if(palgo != vd->algo) m->fflag++;

		if(vd->algo == DENSE) {
#ifdef TRACE
			hipDeviceSynchronize();
			double local_start = rtclock();
#endif
			if(d_threshold == IUNUSED) d_threshold = MIN(d_threshold, (vd->htot_size));
//printf("((%d))\n", (vd->htot_size));
			vd->fhave = 0;
			finished = 0;
			hipMemcpyAsync(m->_finished, &finished, sizeof(int), hipMemcpyHostToDevice);
	
			phase1<<<(m->np),BSIZE,0,m->stream1>>>((m->nv), (m->ne), (m->upper_ne), (m->np), (m->_sp1), (vd->_vv), (m->_i1), (m->_ces), (m->_mapper)
#ifdef V2
			, (vd->_vv2), (m->_i2)
#endif
#ifdef E1
			, (m->_sp1v)
#endif
			);
			phase11<<<(m->dcnt),BSIZE,0,m->stream2>>>((m->nv), (m->np), (vd->_vv), (m->_i1), (m->_dx), (m->_dy), (m->_dccindex), (m->_dp1), (m->_dmap), iter
#ifdef V2
			, (vd->_vv2), (m->_i2)
#endif
#ifdef E1
			, (m->_dp1v)
#endif
			);

			phase2<<<(m->np),BSIZE>>>((m->nv), (m->ne), (m->np), (m->_sp2), (vd->_vv), (m->_i1), (m->_cey), m->_finished, iter, vd->_vfc
#ifdef V2
		, (vd->_vv2), (m->_i2)
#endif
			);
			hipMemcpyAsync(&finished, m->_finished, sizeof(int), hipMemcpyDeviceToHost);
//			if(finished == 0) break;
			if(finished == 0) {
				if(sampled == 2) { out->r = (vd->_vv); return false; }
				else {
					hipMemcpyAsync(vd->h_fs, vd->_gl, sizeof(int)*SCATTER_FACTOR, hipMemcpyDeviceToHost);
					if((vd->h_fs[0])+(vd->h_fs[1])+(vd->h_fs[2])+(vd->h_fs[3]) == 0) { out->r = (vd->_vv); return false; }
				}			
			}
			*g_iter = iter+1;
#ifdef TRACE
			hipDeviceSynchronize();
			double local_end = rtclock();
			printf("0 %d %f\n", iter-1, (local_end-local_start)*1000);
//			printf("0 %d %f %d\n", iter-1, (local_end-local_start)*1000, d_threshold);
#endif
		} else {
			if(vd->fhave == 0) {
//printf("should not happen2\n");
				hipMemset(m->_temp_front, 0, sizeof(int)*SCATTER_FACTOR);
				hipMemset(vd->_gl, 0, sizeof(int)*SCATTER_FACTOR);
				vd->fhave = 1;
				int grid_size = (((m->nv)+127)>>7);
				make_frontier<<<grid_size, 128>>>((m->nv), iter-1, (vd->_vv), &(vd->_curr_f[0]), &(vd->_curr_f[(m->nv)]), &(vd->_curr_f[(m->nv)+((m->nv)>>1)]), &(vd->_curr_f[(m->nv)*2]), m->_temp_front, vd->_vfc);
				hipMemcpyAsync(vd->h_fs, m->_temp_front, sizeof(int)*SCATTER_FACTOR, hipMemcpyDeviceToHost);
				(vd->htot_size) = (vd->h_fs[0])+(vd->h_fs[1])+(vd->h_fs[2])+(vd->h_fs[3]);
			}
#if defined BC
                        vd->accum_frpoint[iter] = (vd->last_accum)+(vd->htot_size);
                        cudaMemcpydd<<<((vd->htot_size+127)>>7), 128>>>(vd->htot_size, vd->last_accum, vd->accum_fr, vd->h_fs[0], &(vd->_curr_f[0]), vd->h_fs[1], &(vd->_curr_f[m->nv]), vd->h_fs[2],
			&(vd->_curr_f[(m->nv)+((m->nv)>>1)]), vd->h_fs[3], &(vd->_curr_f[(m->nv)*2]));
                        vd->last_accum = vd->accum_frpoint[iter];
//			printf("(%d %d)\n", vd->last_accum, vd->accum_frpoint[iter]);
#endif

#ifdef TRACE
			hipDeviceSynchronize();
			double local_start = rtclock();
#endif


			process_f1<<<((vd->htot_size)+255)>>6, 256>>>((m->nv), (vd->htot_size), (m->_ncsc_v), (m->_ncsc_e), vd->_gl, vd->h_fs[0], &(vd->_curr_f[0]), &(vd->_next_f[0]),
			vd->h_fs[1], &(vd->_curr_f[m->nv]), &(vd->_next_f[(m->nv)]), vd->h_fs[2], &(vd->_curr_f[(m->nv)+((m->nv)>>1)]), &(vd->_next_f[(m->nv)+((m->nv)>>1)]), vd->h_fs[3], &(vd->_curr_f[(m->nv)*2]), &(vd->_next_f[(m->nv)*2]), (vd->_vv), (vd->_vv), iter, sampled, (vd->_vfc)
#ifdef V2
			, vd->_vv2, vd->_vv2
#endif
#ifdef E1
, (m->_ncsc_ev)
#endif
);
			
			hipMemcpy(vd->h_fs, vd->_gl, sizeof(int)*SCATTER_FACTOR, hipMemcpyDeviceToHost);
			(vd->htot_size) = (vd->h_fs[0])+(vd->h_fs[1])+(vd->h_fs[2])+(vd->h_fs[3]);

			if((vd->htot_size) == 0) { out->r = (vd->_vv); return false; }

			hipMemset(vd->_gl, 0, sizeof(int)*SCATTER_FACTOR);			
			*g_iter = iter+1;
			_temp_f = (vd->_curr_f); (vd->_curr_f) = (vd->_next_f); (vd->_next_f) = _temp_f;
#ifdef TRACE
			hipDeviceSynchronize();
			double local_end = rtclock();
			printf("1 %d %f\n", iter-1, (local_end-local_start)*1000);
#endif
		}		
//	}
//	hipDeviceSynchronize();
//	double total_end = rtclock();
//	fprintf(stdout, "%d %f\n", iter, (total_end - sample_start)*1000);
#endif
	out->r = (vd->_vv);
#ifdef V2
	out->r2 = (vd->_vv2);
#endif
#ifdef V3
	out->r3 = (vd->_vv3);
#endif
	return true;
}
#endif

